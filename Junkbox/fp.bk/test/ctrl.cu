#include "hip/hip_runtime.h"
# This is an input file for Cu, with comments documenting the input.
# Lines beginning with '#' are comment lines.

# ... Version control
VERS    LMF-6 LMASA-6 LM:7 FP:7
IO      SHOW=f HELP=F VERBOS=31,30 WKP=F IACTIV=0
TESTLMF lmfa --no-iactiv cu -vnk=8 -vbigbas=f
        lmf  --no-iactiv cu -vnk=8 -vbigbas=f
        rm mixm.cu
        lmf  --no-iactiv cu -vnk=8 -vbigbas=t -vpwmode=0 -voveps=0d-7
        lmf  --no-iactiv cu -vnk=8 -vbigbas=t -vpwmode=0 -voveps=0d-7 --band:fn=syml
TEST3P  lmfa --no-iactiv cu -vnk=8 -vbigbas=t -vcupval=1
        lmf  --no-iactiv cu -vnk=8 -vbigbas=t -vcupval=1 --rs=0
TESTGW  echo -1 | lmfgw cu -vnk=8 -vbigbas=t 
        echo  0 | lmfgw cu -vnk=8 -vbigbas=t 
        echo  1 | lmfgw cu -vnk=8 -vbigbas=t 
        echo cu | lmf2gw 
CLEAN   rm -f atm.cu dos.cu mixm.cu out.lmf.cu save.cu ctrl.cu log.cu moms.cu rst.cu wkp.cu
# ... Preprocessor variable declarations:
#   bigbas   bigbas=1 uses a larger basis
#   cupval   Puts cu p state in the valence as a local orbital
#            uses foca=0
% const bigbas=0 cupval=0

# ... Variable declarations used as input in other categories
#   a     the lattice constant, in a.u.
#   da    change in lattice constant: a+da is lattice constant
#   nit   maximum number of band passes for LDA self-consistency
#   nk    number k-points divisions for BZ integrations
#   dist  parameter defining lattice shear; see SHEAR below
#   bzj   0=> k-points include gamma; 1=> k-points offset from gamma; see BZJOB
#   vol   cell volume
#   avw   average WS radius
#   rmt   MT radius.  lmf can handle up to about
#                     10% overlaps with negligible loss in accuracy.
#   gmax  energy cutoff for specifying FT mesh
% const a=6.798
CONST   a={a} da=0 nit=12
        nk=12 dist=0 bzj=1
        vol=a^3/4 avw=(3/4/pi*vol)^(1/3) rmt=.87*avw
        gmax={cupval?12:9} nkgw=8
STRUC   NBAS=1 NSPEC=1 NL=5
        ALAT=a PLAT=  .0 .5 .5  .5  .0 .5  .5 .5  .0
        DALAT=da
# Use one the two following line with dist<>0 for a volume-conserving shear
# SHEAR=0 0 1 ... => tetragonal  SHEAR=1 1 1 ... => trigonal
        SHEAR=0 0 1 1+dist
#       SHEAR=1 1 1 1+dist
SITE    ATOM=A POS= 0 0 0
% const pwmode=0 pwemin=1 pwemax=3 oveps=0
HAM     NSPIN=1 REL=t XCFUN=2 
        FORCES=0 ELIND=-.5 TOL=1e-6
        GMAX=gmax
        FTMESH=10 10 10
        PWMODE={pwmode} PWEMIN={pwemin} PWEMAX={pwemax} OVEPS={oveps}
GW      NKABC=nkgw GCUTB=2.7 GCUTX=2.2
% const hf=f
OPTIONS NSPIN=1 REL=t XCFUN=2 HF={hf}
        PFLOAT=0
BZ      NKABC=nk BZJOB=bzj W=.002 NPTS=1001 SAVDOS=t
# Because bigbas=t is really large, use a more cautious metal treatment
% ifdef bigbas
        METAL=3
% endif
        EF0=0 DELEF=.1 TETRA=t DOS=0-1 0+.5 METAL=2
% ifdef hf
        NEVMX=-1
% endif
EWALD   AS=2.0 TOL=1D-12 ALAT0=a NKRMX=600 NKDMX=600
# Because bigbas=t is really large, use a smaller mixing to help convergence
%ifdef bigbas&f
# for version 7
ITER    MIX=A1,b=.5,n=1;A0,b=.5,n=2 CONV=1e-5 CONVC=1e-5 NIT=nit
MIX     MODE=A1,b=.5,n=1;A0,b=.5,n=2 CONV=1e-5 CONVC=1e-5
%endif
# for version 7
ITER    MIX=A3 CONV=1e-5 CONVC=1e-5 NIT=nit
MIX     MODE=A3 CONV=1e-5 CONVC=1e-5
START   NIT=nit

#  ... Tokens for SPEC category
#  KMXA  defines the cutoff in the polynomial expansion of augmented basis functions.
#        (If not specified, a default is chosen.) KMXA=4 is a rather strict cutoff.
#  A=    parameter defining radial mesh for tabulation of augmented w.f. and density
#  EREF= reference energy, subtracted from the total energy.
#  RSMG= smoothing radius used in electrostatics
#  RFOCA=smoothing radius used in fitting core tails
#  LFOCA=1=> frozen core with tails expanded into the interstitial
#        2=> frozen core with xc pot from tails treated in perturbation theory
#  LMXA= l-cutoff for the basis function in augmentation spheres.
#
#  RSMH,EH, RSMH2, EH2 below define the basis set.
#  NB: the %const construct defines variables in a manner similar to the
#  CONST category above.  But variables defined with the %const or %var
#  are defined for the preprocessor stages, and are cleared once the
#  preprocessor is complete.  See doc/input-file-style.txt
SPEC    ATOM=A Z=29 R=rmt IDMOD=0,0,0,1,1
        P=4.65,4.34,3.87,4.11

%ifdef cupval&bigbas
        PZ=5.5,3.9,4.5
        LMXA=4 LMX=4
        LFOCA=0
%elseifd cupval
%stop 1 cupval must be used in conjunction with bigbas
%elseifd bigbas
        PZ=5.5,5.5,4.5
        LMXA=4 LMX=4
%endif
        KMXA=4

        A=.025
        EREF=-3304.4345
# The following line is not needed since these are the defaults.
        RSMG=.25*rmt RFOCA=0.4*rmt LFOCA=1 LMX=3 LMXA=3

% const rsm1=2.5 rsmd1=1 ed1=-.01
        RSMH={rsm1},{rsm1},{rsmd1} EH=-.01,-.01,{ed1},-.01,-.01
        PZ=5.5,5.5,4.5

%ifdef cupval&bigbas
% const rsmp=0.7 ep=-5 rsm2=1.3
        RSMH2={rsm2},{rsmp},{rsmd1},{rsm2},{rsm2*0} EH2=-1,{ep},-1,-.01,-.01
%elseifd bigbas
% const rsm2=1.3
        RSMH2={rsm2},0,{rsmd1},{rsm2},{rsm2*0} EH2=-1,-1,-1,-.01,-.01
%endif
