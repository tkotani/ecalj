#include "hip/hip_runtime.h"
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 9e69d7e280d2306a6dd11d9e49d6dc9b875e733f
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Feb 14 23:33:57 2023 +0900
INFO: linked at Wed Feb 15 13:01:38 JST 2023
=== START LFMA ===
 mpisize=           1
m_lmfinit: LMFA
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 2.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 3 1.000000  1.000000 beta elin wc killj=  -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  2
freats:

conf:------------------------------------------------------
conf:SPEC_ATOM= A : --- Table for atomic configuration ---
conf:  isp  l  int(P) int(P)z    Qval     Qcore   CoreConf
conf:    1  0       4  5         1.000    6.000 => 1,2,3,
conf:    1  1       4  5         0.000   12.000 => 2,3,
conf:    1  2       3  4        10.000    0.000 => 
conf:    1  3       4  0         0.000    0.000 => 
usedQ=     1.000     0.000    10.000     0.000
conf: Species  A        Z=  29.00 Qc=  18.000 R=  2.311271 Q=  0.000000 nsp= 1 mom=  0.000000
conf: rmt rmax a=  2.311271  48.805862  0.025000 nrmt nr= 393 515
 goto atomc xxx
 atomsc nmcore=           0

 end of atomsc xxxxx
 vsum=  -130.79144076069792                1
sumev= -4.333254 etot= -3304.416258 eref=  0.000000 etot-eref= -3304.416258

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)       pnu
   4s      -0.36411         0.890       2.256       3.582     0.643062       4.761  0
   5s      -0.00028         3.669      10.794      19.873     0.990448       5.848  1
   4p      -0.06295         0.975       3.484       7.414     0.901829       4.561  0
   5p       0.00796         6.760      30.414      48.806*    0.999240       5.593  1
   3d      -0.39691         0.000       0.600       3.429     0.056076       3.888  0
   4d       0.01308         1.868      33.290      48.806*    0.999995       4.148  1
   4f       0.01948         0.000      35.393      48.806*    1.000000       4.137  0

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000141
   3p      -5.29682         0.260       0.619       1.078     0.000727
 tailsm: init

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.31127, rsm= 1.15564
  ---E:energies of smHankels. C:fitting coeeficient for core tail. ---
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000   -15.00000
 C:    -0.07160    10.75053  -187.49213  1222.02349 -4717.78530 21166.80769
        r          rho         fit         diff
    2.311271    0.017797    0.017766    0.000031
    2.967767    0.005662    0.005658    0.000005
    3.810725    0.001517    0.001518   -0.000001
    4.893104    0.000305    0.000305   -0.000000
    6.282906    0.000041    0.000041   -0.000001
    8.067448    0.000003    0.000003    0.000000
    q(fit):     1.203836    rms diff:   0.000016
    fit: r>rmt  1.203836   r<rmt  3.442816   qtot  4.646652
    rho: r>rmt  1.203836   r<rmt  9.796164   qtot 11.000000
 tailsm:  fit tails to        6 functions with

 rsm=  0.11556D+01 rms error=  0.16285D-04
conf: Core rhoc(rmt)= 0.003922 spillout= 0.004646
 Fit with Hankel e=-24.082483 coeff=764.352513
      r            rhoc          fit
    2.311271    0.02095279    0.02095279
    2.429779    0.01229068    0.01231367
    2.753317    0.00285262    0.00285190
    3.119934    0.00054243    0.00053465
    3.535366    0.00008235    0.00007888
    4.006112    0.00000969    0.00000887
    4.539536    0.00000085    0.00000073
    5.143985    0.00000005    0.00000004
 end of freats: spid nmcore=A                  0
Sum of reference energies:                      0.000000000000
OK! end of LMFA ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 9e69d7e280d2306a6dd11d9e49d6dc9b875e733f
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Feb 14 23:33:57 2023 +0900
INFO: linked at Wed Feb 15 13:01:38 JST 2023
===START LMF with   ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=f<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
=== SITE =1
=== SITE =1
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
=== SITE =1
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 2.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 3 1.000000  1.000000 beta elin wc killj=  -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol=   78.538660

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
mksym: found  48  space group operations
splcls:  ibas iclass ispec label(ispec)
          1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
 m_qplistinit:start

 iors  : read rst restart file (binary mesh density)
 iors  : empty file ... nothing read

rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected A,       read A        with rmt=  2.3113  mesh   393  0.025
  ovlpfa: overlap smooth part of FA densities
 site 1 spec 1 pos 0.0000  0.0000  0.0000 Qsmooth 4.6466523386126539 mom 4.6466523386126539
 total smooth Q =  4.6466523386126539

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.796164    3.442816   10.275300    3.921952    6.353348

 Smooth charge on mesh:            4.646652
 Sum of local charges:             6.353348
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:       -0.000000

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.555104
   smooth rhoves     11.022231   charge     4.646652
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.843799 -5.010453 -0.851784
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:   -0.000099
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff            -12.157495      -177.337532      -189.495027
   rhoval*ves            -46.690633      -115.324376      -162.015010
   psnuc*ves              68.735095    -12976.662436    -12907.927341
   utot                   11.022231     -6545.993406     -6534.971175
   rho*exc                -3.843799      -126.414298      -130.258096
   rho*vxc                -5.010453      -167.409316      -172.419769
   valence chg             4.646652         6.353348        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0447

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144466;  11.000000 electrons
         Sum occ. bands:   -0.854464, incl. Bloechl correction: -0.006586
 bndfp:Generating TDOS: efermi=  0.144466  dos window emin emax=  -0.672856  3.084388


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1    9.927753    3.113496    6.814257
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -0.854464 Vin*nin=    -189.495027 Ek=Eb-Vin*nin=     188.640563
 Ek(core)=    3171.756639 Exc=    -130.258096 Ees=   -6534.971175 Eharris=   -3304.832069

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.361301   -168.223682   -174.584983
    E_B(band energy sum)=   -0.854464  E_B-nout*Vin=  173.730519

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178923   charge     4.185743
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054120 -3.974966 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030715      -175.114087      -182.144802
   rhoval*ves            -50.183969      -106.215150      -156.399119
   psnuc*ves              76.541815    -12962.871088    -12886.329272
   utot                   13.178923     -6534.543119     -6521.364196
   rho*exc                -3.054120      -125.587137      -128.641257
   rho*vxc                -3.974966      -166.302309      -170.277275
   valence chg             4.185743         6.814257        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.730519 Ekcore=      3171.756639 Ektot    =     3345.487159
 Exc=    -128.641257 Ees   =     -6521.364196 EKohnSham=    -3304.518294
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 0 RMS DQ= 4.34E-2
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.34D-02
 mixrho: add corrections to qcell smrho = -0.23414D-07 -0.29812D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.832069   ehk=   -3304.518294
h ehf(eV)=-44964.884163 ehk(eV)=-44960.615006 sev(eV)=-11.625667

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178923   charge     4.185743
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054120 -3.974966 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000553
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030715      -175.114087      -182.144801
   rhoval*ves            -50.183969      -106.215150      -156.399119
   psnuc*ves              76.541815    -12962.871088    -12886.329272
   utot                   13.178923     -6534.543119     -6521.364196
   rho*exc                -3.054120      -125.587137      -128.641257
   rho*vxc                -3.974966      -166.302309      -170.277275
   valence chg             4.185743         6.814257        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0560

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.257389;  11.000000 electrons
         Sum occ. bands:   -9.368175, incl. Bloechl correction: -0.013546
 bndfp:Generating TDOS: efermi= -0.257389  dos window emin emax=  -0.960038  2.682534


 mkrout:  Qtrue      sm,loc       local
   1   10.453470    1.685884    8.767586
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -9.368175 Vin*nin=    -182.144801 Ek=Eb-Vin*nin=     172.776627
 Ek(core)=    3171.756639 Exc=    -128.641257 Ees=   -6521.364196 Eharris=   -3305.472187

 mkekin:
   nout*Vin = smpart,onsite,total=:     -3.908741   -233.344935   -237.253675
    E_B(band energy sum)=   -9.368175  E_B-nout*Vin=  227.885500

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.359167
   smooth rhoves      3.889627   charge     2.232413
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.459173 -1.896418 -0.723230
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -2.683250      -210.677503      -213.360752
   rhoval*ves            -33.661409      -148.825746      -182.487155
   psnuc*ves              41.440663    -12997.718880    -12956.278216
   utot                    3.889627     -6573.272313     -6569.382686
   rho*exc                -1.459173      -131.947804      -133.406976
   rho*vxc                -1.896418      -174.700211      -176.596629
   valence chg             2.232413         8.767586        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      227.885500 Ekcore=      3171.756639 Ektot    =     3399.642140
 Exc=    -133.406976 Ees   =     -6569.382686 EKohnSham=    -3303.147522
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 1 RMS DQ= 1.23E-1  last it= 4.34E-2
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.23D-01
   tj: 0.82062
 mixrho: add corrections to qcell smrho = -0.96492D-08 -0.12286D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3305.472187   ehk=   -3303.147522
 From last iter    ehf=   -3304.832069   ehk=   -3304.518294
 diffe(q)= -0.640118 (0.123166)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44973.593479 ehk(eV)=-44941.964555 sev(eV)=-127.461515

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.620291
   smooth rhoves     11.107548   charge     3.835349
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.748140 -3.575952 -0.844926
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000503
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.127022      -182.314718      -188.441740
   rhoval*ves            -48.030181      -113.790712      -161.820893
   psnuc*ves              70.245276    -12969.122178    -12898.876902
   utot                   11.107548     -6541.456445     -6530.348897
   rho*exc                -2.748140      -126.718973      -129.467113
   rho*vxc                -3.575952      -167.796505      -171.372457
   valence chg             3.835349         7.164651        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0574

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.148169;  11.000000 electrons
         Sum occ. bands:   -5.251649, incl. Bloechl correction: -0.012268
 bndfp:Generating TDOS: efermi= -0.148169  dos window emin emax=  -0.774071  2.791753


 mkrout:  Qtrue      sm,loc       local
   1   10.297519    2.200427    8.097092
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -5.251649 Vin*nin=    -188.441740 Ek=Eb-Vin*nin=     183.190091
 Ek(core)=    3171.756639 Exc=    -129.467113 Ees=   -6530.348897 Eharris=   -3304.869280

 mkekin:
   nout*Vin = smpart,onsite,total=:     -4.778849   -205.317019   -210.095869
    E_B(band energy sum)=   -5.251649  E_B-nout*Vin=  204.844220

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.463277
   smooth rhoves      6.414066   charge     2.902909
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.980933 -2.576057 -0.778176
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.013607      -196.297023      -200.310630
   rhoval*ves            -40.418130      -131.103281      -171.521411
   psnuc*ves              53.246261    -12980.824930    -12927.578668
   utot                    6.414066     -6555.964106     -6549.550040
   rho*exc                -1.980933      -129.503698      -131.484631
   rho*vxc                -2.576057      -171.471048      -174.047106
   valence chg             2.902909         8.097092        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      204.844220 Ekcore=      3171.756639 Ektot    =     3376.600859
 Exc=    -131.484631 Ees   =     -6549.550040 EKohnSham=    -3304.433811
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 2 RMS DQ= 5.12E-2  last it= 1.23E-1
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 5.12D-02
   tj:-0.76791  -0.08165
 mixrho: add corrections to qcell smrho = -0.31521D-06 -0.40134D-08

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.869280   ehk=   -3304.433811
 From last iter    ehf=   -3305.472187   ehk=   -3303.147522
 diffe(q)=  0.602907 (0.051213)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44965.390444 ehk(eV)=-44959.465546 sev(eV)=-71.452882

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.525743
   smooth rhoves      8.256180   charge     3.313051
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.319647 -3.017521 -0.807475
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000461
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.941480      -185.839184      -190.780664
   rhoval*ves            -43.897574      -119.382195      -163.279769
   psnuc*ves              60.409935    -12969.317726    -12908.907791
   utot                    8.256180     -6544.349961     -6536.093780
   rho*exc                -2.319647      -127.961918      -130.281565
   rho*vxc                -3.017521      -169.433789      -172.451310
   valence chg             3.313051         7.686949        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0589

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.132346;  11.000000 electrons
         Sum occ. bands:   -0.993219, incl. Bloechl correction: -0.006627
 bndfp:Generating TDOS: efermi=  0.132346  dos window emin emax=  -0.684035  3.072269


 mkrout:  Qtrue      sm,loc       local
   1    9.930892    3.113709    6.817183
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -0.993219 Vin*nin=    -190.780664 Ek=Eb-Vin*nin=     189.787444
 Ek(core)=    3171.756639 Exc=    -130.281565 Ees=   -6536.093780 Eharris=   -3304.831261

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.042093   -168.378044   -174.420138
    E_B(band energy sum)=   -0.993219  E_B-nout*Vin=  173.426918

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.676372
   smooth rhoves     13.152472   charge     4.182817
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.052152 -3.972407 -0.866429
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.029083      -174.837177      -181.866260
   rhoval*ves            -50.154645      -105.981094      -156.135739
   psnuc*ves              76.459589    -12962.467179    -12886.007589
   utot                   13.152472     -6534.224136     -6521.071664
   rho*exc                -3.052152      -125.574326      -128.626478
   rho*vxc                -3.972407      -166.285214      -170.257621
   valence chg             4.182817         6.817183        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.426918 Ekcore=      3171.756639 Ektot    =     3345.183558
 Exc=    -128.626478 Ees   =     -6521.071664 EKohnSham=    -3304.514584
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 3 RMS DQ= 4.22E-2  last it= 5.12E-2
 AMIX: nmix=3 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.22D-02
   tj: 0.74166  -0.16547   0.00023
 mixrho: add corrections to qcell smrho = -0.17530D-06 -0.22320D-08

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.831261   ehk=   -3304.514584
 From last iter    ehf=   -3304.869280   ehk=   -3304.433811
 diffe(q)=  0.038018 (0.042227)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44964.873174 ehk(eV)=-44960.564532 sev(eV)=-13.513546

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570816
   smooth rhoves      9.541427   charge     3.556294
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515298 -3.272452 -0.825708
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.474529      -185.001328      -190.475858
   rhoval*ves            -45.954938      -117.383196      -163.338134
   psnuc*ves              65.037792    -12970.249397    -12905.211605
   utot                    9.541427     -6543.816296     -6534.274870
   rho*exc                -2.515298      -127.432934      -129.948232
   rho*vxc                -3.272452      -168.737527      -172.009979
   valence chg             3.556294         7.443706        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0592

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014879;  11.000000 electrons
         Sum occ. bands:   -2.770224, incl. Bloechl correction: -0.009077
 bndfp:Generating TDOS: efermi= -0.014879  dos window emin emax=  -0.725887  2.925044


 mkrout:  Qtrue      sm,loc       local
   1   10.105683    2.685743    7.419940
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.770224 Vin*nin=    -190.475858 Ek=Eb-Vin*nin=     187.705634
 Ek(core)=    3171.756639 Exc=    -129.948232 Ees=   -6534.274870 Eharris=   -3304.760829

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.495500   -184.335642   -189.831142
    E_B(band energy sum)=   -2.770224  E_B-nout*Vin=  187.060918

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575486
   smooth rhoves      9.678781   charge     3.580060
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.534251 -3.297145 -0.827512
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.522086      -184.602389      -190.124476
   rhoval*ves            -46.157297      -116.896962      -163.054259
   psnuc*ves              65.514859    -12969.845858    -12904.330999
   utot                    9.678781     -6543.371410     -6533.692629
   rho*exc                -2.534251      -127.351062      -129.885313
   rho*vxc                -3.297145      -168.629433      -171.926578
   valence chg             3.580060         7.419940        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.060918 Ekcore=      3171.756639 Ektot    =     3358.817557
 Exc=    -129.885313 Ees   =     -6533.692629 EKohnSham=    -3304.760384
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.53E-3  last it= 4.22E-2
 AMIX: nmix=3 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.53D-03
   tj: 0.41658   0.73067  -0.16309
 mixrho: add corrections to qcell smrho = -0.17255D-06 -0.21970D-08

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.760829   ehk=   -3304.760384
 From last iter    ehf=   -3304.831261   ehk=   -3304.514584
 diffe(q)=  0.070433 (0.001529)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914882 ehk(eV)=-44963.908839 sev(eV)=-37.691115

--- BNDFP:  begin iteration 6 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570805
   smooth rhoves      9.540941   charge     3.556172
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515182 -3.272301 -0.825702
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.474117      -185.002056      -190.476173
   rhoval*ves            -45.954331      -117.384093      -163.338425
   psnuc*ves              65.036213    -12970.248272    -12905.212058
   utot                    9.540941     -6543.816183     -6534.275241
   rho*exc                -2.515182      -127.433074      -129.948256
   rho*vxc                -3.272301      -168.737710      -172.010011
   valence chg             3.556172         7.443828        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0601

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014873;  11.000000 electrons
         Sum occ. bands:   -2.770144, incl. Bloechl correction: -0.009077
 bndfp:Generating TDOS: efermi= -0.014873  dos window emin emax=  -0.725886  2.925050


 mkrout:  Qtrue      sm,loc       local
   1   10.108507    2.682717    7.425790
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.770144 Vin*nin=    -190.476173 Ek=Eb-Vin*nin=     187.706029
 Ek(core)=    3171.756639 Exc=    -129.948256 Ees=   -6534.275241 Eharris=   -3304.760829

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.491101   -184.470259   -189.961360
    E_B(band energy sum)=   -2.770144  E_B-nout*Vin=  187.191216

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.574220
   smooth rhoves      9.644115   charge     3.574210
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.529655 -3.291158 -0.827059
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.510756      -184.677149      -190.187905
   rhoval*ves            -46.105475      -116.997545      -163.103020
   psnuc*ves              65.393706    -12969.909603    -12904.515898
   utot                    9.644115     -6543.453574     -6533.809459
   rho*exc                -2.529655      -127.369301      -129.898956
   rho*vxc                -3.291158      -168.653500      -171.944658
   valence chg             3.574210         7.425790        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.191216 Ekcore=      3171.756639 Ektot    =     3358.947855
 Exc=    -129.898956 Ees   =     -6533.809459 EKohnSham=    -3304.760559
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.21E-3  last it= 1.53E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.21D-03
   tj:-3.79423
 mixrho: add corrections to qcell smrho = -0.22055D-06 -0.28082D-08

 iors  : write rst restart file (binary mesh density)

   it  6  of 12    ehf=   -3304.760829   ehk=   -3304.760559
 From last iter    ehf=   -3304.760829   ehk=   -3304.760384
 diffe(q)= -0.000001 (0.001215)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914889 ehk(eV)=-44963.911219 sev(eV)=-37.690025

--- BNDFP:  begin iteration 7 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569419
   smooth rhoves      9.513166   charge     3.552013
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.512250 -3.268486 -0.825334
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000478
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.467808      -184.959939      -190.427747
   rhoval*ves            -45.907688      -117.379450      -163.287139
   psnuc*ves              64.934021    -12970.151466    -12905.217445
   utot                    9.513166     -6543.765458     -6534.252292
   rho*exc                -2.512250      -127.438521      -129.950771
   rho*vxc                -3.268486      -168.744835      -172.013321
   valence chg             3.552013         7.447987        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0602

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012753;  11.000000 electrons
         Sum occ. bands:   -2.742206, incl. Bloechl correction: -0.009047
 bndfp:Generating TDOS: efermi= -0.012753  dos window emin emax=  -0.724725  2.927170


 mkrout:  Qtrue      sm,loc       local
   1   10.106736    2.686891    7.419846
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.742206 Vin*nin=    -190.427747 Ek=Eb-Vin*nin=     187.685541
 Ek(core)=    3171.756639 Exc=    -129.950771 Ees=   -6534.252292 Eharris=   -3304.760882

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.492377   -184.319413   -189.811790
    E_B(band energy sum)=   -2.742206  E_B-nout*Vin=  187.069584

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575225
   smooth rhoves      9.675847   charge     3.580155
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.534631 -3.297644 -0.827467
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.524762      -184.596620      -190.121382
   rhoval*ves            -46.150489      -116.898724      -163.049212
   psnuc*ves              65.502182    -12969.852056    -12904.349873
   utot                    9.675847     -6543.375390     -6533.699543
   rho*exc                -2.534631      -127.352462      -129.887093
   rho*vxc                -3.297644      -168.631288      -171.928932
   valence chg             3.580155         7.419846        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.069584 Ekcore=      3171.756639 Ektot    =     3358.826223
 Exc=    -129.887093 Ees   =     -6533.699543 EKohnSham=    -3304.760412
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.51E-3  last it= 1.21E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.51D-03
   tj: 4.76989  -4.67925
 mixrho: add corrections to qcell smrho = -0.26268D-06 -0.33445D-08

 iors  : write rst restart file (binary mesh density)

   it  7  of 12    ehf=   -3304.760882   ehk=   -3304.760412
 From last iter    ehf=   -3304.760829   ehk=   -3304.760559
 diffe(q)= -0.000053 (0.001512)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.915611 ehk(eV)=-44963.909217 sev(eV)=-37.309903

--- BNDFP:  begin iteration 8 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569213
   smooth rhoves      9.511363   charge     3.552241
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.512712 -3.269092 -0.825303
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000479
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.470491      -184.952846      -190.423337
   rhoval*ves            -45.902714      -117.378587      -163.281302
   psnuc*ves              64.925441    -12970.155550    -12905.230109
   utot                    9.511363     -6543.767069     -6534.255705
   rho*exc                -2.512712      -127.439352      -129.952064
   rho*vxc                -3.269092      -168.745938      -172.015030
   valence chg             3.552241         7.447759        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0603

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012086;  11.000000 electrons
         Sum occ. bands:   -2.733117, incl. Bloechl correction: -0.009035
 bndfp:Generating TDOS: efermi= -0.012086  dos window emin emax=  -0.724447  2.927837


 mkrout:  Qtrue      sm,loc       local
   1   10.105987    2.688659    7.417329
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.733117 Vin*nin=    -190.423337 Ek=Eb-Vin*nin=     187.690220
 Ek(core)=    3171.756639 Exc=    -129.952064 Ees=   -6534.255705 Eharris=   -3304.760910

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.494911   -184.251051   -189.745963
    E_B(band energy sum)=   -2.733117  E_B-nout*Vin=  187.012845

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575649
   smooth rhoves      9.689300   charge     3.582671
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.536738 -3.300392 -0.827640
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.530692      -184.558258      -190.088950
   rhoval*ves            -46.169513      -116.852972      -163.022485
   psnuc*ves              65.548112    -12969.821742    -12904.273629
   utot                    9.689300     -6543.337357     -6533.648057
   rho*exc                -2.536738      -127.345025      -129.881763
   rho*vxc                -3.300392      -168.621475      -171.921866
   valence chg             3.582671         7.417329        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.012845 Ekcore=      3171.756639 Ektot    =     3358.769485
 Exc=    -129.881763 Ees   =     -6533.648057 EKohnSham=    -3304.760335
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.66E-3  last it= 1.51E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: Reducing nmix to  1: t_j exceeds tm: tj=  9.81781   0.51738
 AMIX: Reducing nmix to  0: t_j exceeds tm: tj= 11.40537
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.66D-03
 mixrho: add corrections to qcell smrho = -0.44011D-07 -0.56037D-09

 iors  : write rst restart file (binary mesh density)

   it  8  of 12    ehf=   -3304.760910   ehk=   -3304.760335
 From last iter    ehf=   -3304.760882   ehk=   -3304.760412
 diffe(q)= -0.000028 (0.001655)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.915985 ehk(eV)=-44963.908167 sev(eV)=-37.186248

--- BNDFP:  begin iteration 9 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575649
   smooth rhoves      9.689300   charge     3.582671
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.536738 -3.300391 -0.827640
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.530692      -184.558258      -190.088950
   rhoval*ves            -46.169513      -116.852972      -163.022485
   psnuc*ves              65.548112    -12969.821742    -12904.273629
   utot                    9.689300     -6543.337357     -6533.648057
   rho*exc                -2.536738      -127.345025      -129.881763
   rho*vxc                -3.300391      -168.621475      -171.921866
   valence chg             3.582671         7.417329        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0605

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.035747;  11.000000 electrons
         Sum occ. bands:   -3.077823, incl. Bloechl correction: -0.009542
 bndfp:Generating TDOS: efermi= -0.035747  dos window emin emax=  -0.731399  2.904176


 mkrout:  Qtrue      sm,loc       local
   1   10.137094    2.610458    7.526636
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -3.077823 Vin*nin=    -190.088950 Ek=Eb-Vin*nin=     187.011127
 Ek(core)=    3171.756639 Exc=    -129.881763 Ees=   -6533.648057 Eharris=   -3304.762053

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.375770   -187.428657   -192.804427
    E_B(band energy sum)=   -3.077823  E_B-nout*Vin=  189.726604

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.557495
   smooth rhoves      9.117168   charge     3.473364
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.445215 -3.181090 -0.820136
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.273022      -186.420755      -191.693778
   rhoval*ves            -45.334500      -119.030003      -164.364503
   psnuc*ves              63.568837    -12971.427099    -12907.858263
   utot                    9.117168     -6545.228551     -6536.111383
   rho*exc                -2.445215      -127.682223      -130.127438
   rho*vxc                -3.181090      -169.066510      -172.247601
   valence chg             3.473364         7.526636        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      189.726604 Ekcore=      3171.756639 Ektot    =     3361.483243
 Exc=    -130.127438 Ees   =     -6536.111383 EKohnSham=    -3304.755578
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 6.35E-3  last it= 1.66E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 6.35D-03
   tj: 0.79421
 mixrho: add corrections to qcell smrho = -0.43104D-07 -0.54883D-09

 iors  : write rst restart file (binary mesh density)

   it  9  of 12    ehf=   -3304.762053   ehk=   -3304.755578
 From last iter    ehf=   -3304.760910   ehk=   -3304.760335
 diffe(q)= -0.001143 (0.006352)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.931539 ehk(eV)=-44963.843438 sev(eV)=-41.876240

--- BNDFP:  begin iteration 10 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571913
   smooth rhoves      9.570134   charge     3.560176
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.517836 -3.275752 -0.826108
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.477246      -184.944128      -190.421375
   rhoval*ves            -46.000522      -117.300446      -163.300968
   psnuc*ves              65.140790    -12970.152114    -12905.011325
   utot                    9.570134     -6543.726280     -6534.156146
   rho*exc                -2.517836      -127.414392      -129.932228
   rho*vxc                -3.275752      -168.713025      -171.988777
   valence chg             3.560176         7.439824        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0610

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020462;  11.000000 electrons
         Sum occ. bands:   -2.850377, incl. Bloechl correction: -0.009198
 bndfp:Generating TDOS: efermi= -0.020462  dos window emin emax=  -0.727413  2.919460


 mkrout:  Qtrue      sm,loc       local
   1   10.116285    2.663939    7.452347
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.850377 Vin*nin=    -190.421375 Ek=Eb-Vin*nin=     187.570998
 Ek(core)=    3171.756639 Exc=    -129.932228 Ees=   -6534.156146 Eharris=   -3304.760737

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.460618   -185.224306   -190.684924
    E_B(band energy sum)=   -2.850377  E_B-nout*Vin=  187.834547

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569759
   smooth rhoves      9.503154   charge     3.547653
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.507420 -3.262174 -0.825237
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.448283      -185.117030      -190.565313
   rhoval*ves            -45.903673      -117.513756      -163.417429
   psnuc*ves              64.909982    -12970.280547    -12905.370566
   utot                    9.503154     -6543.897152     -6534.393997
   rho*exc                -2.507420      -127.450433      -129.957852
   rho*vxc                -3.262174      -168.760569      -172.022744
   valence chg             3.547653         7.452347        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.834547 Ekcore=      3171.756639 Ektot    =     3359.591186
 Exc=    -129.957852 Ees   =     -6534.393997 EKohnSham=    -3304.760663
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 6.49E-4  last it= 6.35E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 6.49D-04
   tj:-0.01272   0.25026
 mixrho: add corrections to qcell smrho = -0.49861D-07 -0.63486D-09

 iors  : write rst restart file (binary mesh density)

   it 10  of 12    ehf=   -3304.760737   ehk=   -3304.760663
 From last iter    ehf=   -3304.762053   ehk=   -3304.755578
 diffe(q)=  0.001316 (0.000649)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.913630 ehk(eV)=-44963.912635 sev(eV)=-38.781659

--- BNDFP:  begin iteration 11 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571389
   smooth rhoves      9.554541   charge     3.557362
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515543 -3.272763 -0.825904
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.471106      -184.960785      -190.431891
   rhoval*ves            -45.977652      -117.329047      -163.306699
   psnuc*ves              65.086735    -12970.151146    -12905.064411
   utot                    9.554541     -6543.740097     -6534.185555
   rho*exc                -2.515543      -127.421104      -129.936647
   rho*vxc                -3.272763      -168.721867      -171.994630
   valence chg             3.557362         7.442638        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0611

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018833;  11.000000 electrons
         Sum occ. bands:   -2.827063, incl. Bloechl correction: -0.009164
 bndfp:Generating TDOS: efermi= -0.018833  dos window emin emax=  -0.726911  2.921089


 mkrout:  Qtrue      sm,loc       local
   1   10.113814    2.669418    7.444397
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.827063 Vin*nin=    -190.431891 Ek=Eb-Vin*nin=     187.604828
 Ek(core)=    3171.756639 Exc=    -129.936647 Ees=   -6534.185555 Eharris=   -3304.760735

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.468221   -185.002190   -190.470411
    E_B(band energy sum)=   -2.827063  E_B-nout*Vin=  187.643348

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571124
   smooth rhoves      9.545634   charge     3.555603
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.514033 -3.270794 -0.825790
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.466665      -184.988182      -190.454847
   rhoval*ves            -45.965158      -117.360761      -163.325919
   psnuc*ves              65.056427    -12970.171664    -12905.115237
   utot                    9.545634     -6543.766212     -6534.220578
   rho*exc                -2.514033      -127.426110      -129.940143
   rho*vxc                -3.270794      -168.728471      -171.999265
   valence chg             3.555603         7.444397        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.643348 Ekcore=      3171.756639 Ektot    =     3359.399988
 Exc=    -129.940143 Ees   =     -6534.220578 EKohnSham=    -3304.760733
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 9.32E-5  last it= 6.49E-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 9.32D-05
   tj:-0.08030  -0.00778
 mixrho: add corrections to qcell smrho = -0.45349D-07 -0.57741D-09

 iors  : write rst restart file (binary mesh density)

   it 11  of 12    ehf=   -3304.760735   ehk=   -3304.760733
 From last iter    ehf=   -3304.760737   ehk=   -3304.760663
 diffe(q)=  0.000002 (0.000093)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.913604 ehk(eV)=-44963.913584 sev(eV)=-38.464452

--- BNDFP:  begin iteration 12 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571339
   smooth rhoves      9.552421   charge     3.556881
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515101 -3.272187 -0.825878
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469660      -184.966613      -190.436273
   rhoval*ves            -45.974914      -117.335507      -163.310421
   psnuc*ves              65.079757    -12970.153156    -12905.073399
   utot                    9.552421     -6543.744331     -6534.191910
   rho*exc                -2.515101      -127.422165      -129.937267
   rho*vxc                -3.272187      -168.723265      -171.995452
   valence chg             3.556881         7.443119        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0607

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018655;  11.000000 electrons
         Sum occ. bands:   -2.824472, incl. Bloechl correction: -0.009160
 bndfp:Generating TDOS: efermi= -0.018655  dos window emin emax=  -0.726872  2.921267


 mkrout:  Qtrue      sm,loc       local
   1   10.113520    2.670098    7.443422
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.824472 Vin*nin=    -190.436273 Ek=Eb-Vin*nin=     187.611801
 Ek(core)=    3171.756639 Exc=    -129.937267 Ees=   -6534.191910 Eharris=   -3304.760736

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.469227   -184.973775   -190.443002
    E_B(band energy sum)=   -2.824472  E_B-nout*Vin=  187.618530

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571289
   smooth rhoves      9.550826   charge     3.556578
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.514846 -3.271855 -0.825858
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.468938      -184.971185      -190.440123
   rhoval*ves            -45.972632      -117.340965      -163.313596
   psnuc*ves              65.074283    -12970.156704    -12905.082421
   utot                    9.550826     -6543.748834     -6534.198009
   rho*exc                -2.514846      -127.423051      -129.937897
   rho*vxc                -3.271855      -168.724433      -171.996288
   valence chg             3.556578         7.443422        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.618530 Ekcore=      3171.756639 Ektot    =     3359.375169
 Exc=    -129.937897 Ees   =     -6534.198009 EKohnSham=    -3304.760736
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.62E-5  last it= 9.32E-5
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.62D-05
   tj:-0.21091
 mixrho: add corrections to qcell smrho = -0.44959D-07 -0.57244D-09

 iors  : write rst restart file (binary mesh density)

   it 12  of 12    ehf=   -3304.760736   ehk=   -3304.760736
 From last iter    ehf=   -3304.760735   ehk=   -3304.760733
 diffe(q)= -0.000002 (0.000016)    tol= 0.000010 (0.000010)   more=F
x ehf(eV)=-44963.913628 ehk(eV)=-44963.913626 sev(eV)=-38.429203
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 9e69d7e280d2306a6dd11d9e49d6dc9b875e733f
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Feb 14 23:33:57 2023 +0900
INFO: linked at Wed Feb 15 13:01:38 JST 2023
===START LMF with   ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
=== SITE =1
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
=== SITE =1
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
=== SITE =1
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 3 1.000000  1.000000 beta elin wc killj=  -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110  5.100
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000  0.000
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto rsmh2    1  1.30  0.00  1.00  1.30
mto  eh2     1 -1.00 -1.00 -1.00 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  3  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol=   78.538660

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
mksym: found  48  space group operations
splcls:  ibas iclass ispec label(ispec)
          1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 m_qplistinit:start
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.39  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00
         site   1, species A       : augmentation lmax changed from 3 to 4
         site   1, species A       : inflate local density from nlm= 16 to 25

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571324
   smooth rhoves      9.551921   charge     3.556784
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515018 -3.272078 -0.825872
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469417      -184.967600      -190.437017
   rhoval*ves            -45.974208      -117.336789      -163.310997
   psnuc*ves              65.078050    -12970.153549    -12905.075499
   utot                    9.551921     -6543.745169     -6534.193248
   rho*exc                -2.515018      -127.422406      -129.937424
   rho*vxc                -3.272078      -168.723581      -171.995660
   valence chg             3.556784         7.443216        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1887

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018882;  11.000000 electrons
         Sum occ. bands:   -2.826227, incl. Bloechl correction: -0.009153
 bndfp:Generating TDOS: efermi= -0.018882  dos window emin emax=  -0.727728  2.921040


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128172    2.833460    7.294712
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.826227 Vin*nin=    -190.437017 Ek=Eb-Vin*nin=     187.610790
 Ek(core)=    3171.756639 Exc=    -129.937424 Ees=   -6534.193248 Eharris=   -3304.763242

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.930175   -184.355150   -190.285326
    E_B(band energy sum)=   -2.826227  E_B-nout*Vin=  187.459098

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581330
   smooth rhoves     10.109478   charge     3.705288
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657899 -3.458574 -0.832968
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010283      -184.303210      -190.313494
   rhoval*ves            -46.616857      -116.580881      -163.197739
   psnuc*ves              66.835813    -12971.741073    -12904.905260
   utot                   10.109478     -6544.160977     -6534.051499
   rho*exc                -2.657899      -127.269510      -129.927409
   rho*vxc                -3.458574      -168.523773      -171.982348
   valence chg             3.705288         7.294712        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.459098 Ekcore=      3171.756639 Ektot    =     3359.215738
 Exc=    -129.927409 Ees   =     -6534.051499 EKohnSham=    -3304.763170
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 0 RMS DQ= 4.32E-3
 AMIX: nmix=0 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 4.32D-03
 mixrho: add corrections to qcell smrho = -0.28236D-07 -0.35952D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.763242   ehk=   -3304.763170
i ehf(eV)=-44963.947720 ehk(eV)=-44963.946745 sev(eV)=-38.453084

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581330
   smooth rhoves     10.109478   charge     3.705288
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657899 -3.458574 -0.832968
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000042
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010283      -184.303210      -190.313494
   rhoval*ves            -46.616857      -116.580881      -163.197739
   psnuc*ves              66.835813    -12971.741073    -12904.905260
   utot                   10.109478     -6544.160977     -6534.051499
   rho*exc                -2.657899      -127.269510      -129.927409
   rho*vxc                -3.458574      -168.523773      -171.982348
   valence chg             3.705288         7.294712        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.2003

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020804;  11.000000 electrons
         Sum occ. bands:   -2.854402, incl. Bloechl correction: -0.009202
 bndfp:Generating TDOS: efermi= -0.020804  dos window emin emax=  -0.727848  2.919119


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.131652    2.824824    7.306828
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.854402 Vin*nin=    -190.313494 Ek=Eb-Vin*nin=     187.459091
 Ek(core)=    3171.756639 Exc=    -129.927409 Ees=   -6534.051499 Eharris=   -3304.763177

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.992925   -184.685915   -190.678840
    E_B(band energy sum)=   -2.854402  E_B-nout*Vin=  187.824438

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579287
   smooth rhoves     10.043910   charge     3.693172
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.647646 -3.445208 -0.832150
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.980081      -184.560852      -190.540933
   rhoval*ves            -46.528467      -116.863058      -163.391525
   psnuc*ves              66.616286    -12971.995161    -12905.378875
   utot                   10.043910     -6544.429110     -6534.385200
   rho*exc                -2.647646      -127.311303      -129.958949
   rho*vxc                -3.445208      -168.578965      -172.024173
   valence chg             3.693172         7.306828        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.824438 Ekcore=      3171.756639 Ektot    =     3359.581077
 Exc=    -129.958949 Ees   =     -6534.385200 EKohnSham=    -3304.763072
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 1 RMS DQ= 8.02E-4  last it= 4.32E-3
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 8.02D-04
   tj: 0.08304
 mixrho: add corrections to qcell smrho = -0.26732D-07 -0.34037D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3304.763177   ehk=   -3304.763072
 From last iter    ehf=   -3304.763242   ehk=   -3304.763170
 diffe(q)=  0.000065 (0.000802)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946840 ehk(eV)=-44963.945404 sev(eV)=-38.836424

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579457
   smooth rhoves     10.049347   charge     3.694179
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.648498 -3.446317 -0.832218
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.982587      -184.539475      -190.522062
   rhoval*ves            -46.535824      -116.839627      -163.375450
   psnuc*ves              66.634517    -12971.974061    -12905.339544
   utot                   10.049347     -6544.406844     -6534.357497
   rho*exc                -2.648498      -127.307832      -129.956329
   rho*vxc                -3.446317      -168.574381      -172.020698
   valence chg             3.694179         7.305821        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.2193

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.011984;  11.000000 electrons
         Sum occ. bands:   -2.728304, incl. Bloechl correction: -0.009007
 bndfp:Generating TDOS: efermi= -0.011984  dos window emin emax=  -0.725679  2.927939


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.118663    2.849909    7.268755
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.728304 Vin*nin=    -190.522062 Ek=Eb-Vin*nin=     187.793758
 Ek(core)=    3171.756639 Exc=    -129.956329 Ees=   -6534.357497 Eharris=   -3304.763429

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.025990   -183.425391   -189.451381
    E_B(band energy sum)=   -2.728304  E_B-nout*Vin=  186.723077

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.586229
   smooth rhoves     10.259585   charge     3.731245
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.679154 -3.486274 -0.834838
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.068382      -183.815393      -189.883775
   rhoval*ves            -46.821476      -116.018934      -162.840410
   psnuc*ves              67.340646    -12971.265048    -12903.924402
   utot                   10.259585     -6543.641991     -6533.382406
   rho*exc                -2.679154      -127.180581      -129.859735
   rho*vxc                -3.486274      -168.406353      -171.892627
   valence chg             3.731245         7.268755        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      186.723077 Ekcore=      3171.756639 Ektot    =     3358.479716
 Exc=    -129.859735 Ees   =     -6533.382406 EKohnSham=    -3304.762424
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 2 RMS DQ= 2.36E-3  last it= 8.02E-4
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 2.36D-03
   tj: 0.74718  -0.00311
 mixrho: add corrections to qcell smrho = -0.27292D-07 -0.34750D-09

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.763429   ehk=   -3304.762424
 From last iter    ehf=   -3304.763177   ehk=   -3304.763072
 diffe(q)= -0.000251 (0.002358)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.950261 ehk(eV)=-44963.936593 sev(eV)=-37.120760

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581057
   smooth rhoves     10.098682   charge     3.702879
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655668 -3.455663 -0.832837
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.002540      -184.371330      -190.373869
   rhoval*ves            -46.603621      -116.647901      -163.251522
   psnuc*ves              66.800985    -12971.809097    -12905.008112
   utot                   10.098682     -6544.228499     -6534.129817
   rho*exc                -2.655668      -127.277971      -129.933639
   rho*vxc                -3.455663      -168.534952      -171.990614
   valence chg             3.702879         7.297121        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.2216

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.019147;  11.000000 electrons
         Sum occ. bands:   -2.830219, incl. Bloechl correction: -0.009164
 bndfp:Generating TDOS: efermi= -0.019147  dos window emin emax=  -0.727515  2.920776


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.129011    2.830104    7.298907
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.830219 Vin*nin=    -190.373869 Ek=Eb-Vin*nin=     187.543650
 Ek(core)=    3171.756639 Exc=    -129.933639 Ees=   -6534.129817 Eharris=   -3304.763166

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.000745   -184.420040   -190.420784
    E_B(band energy sum)=   -2.830219  E_B-nout*Vin=  187.590566

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580714
   smooth rhoves     10.088310   charge     3.701093
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.654215 -3.453769 -0.832707
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.998622      -184.401382      -190.400003
   rhoval*ves            -46.589274      -116.683573      -163.272847
   psnuc*ves              66.765893    -12971.837530    -12905.071637
   utot                   10.088310     -6544.260552     -6534.172242
   rho*exc                -2.654215      -127.283912      -129.938126
   rho*vxc                -3.453769      -168.542795      -171.996563
   valence chg             3.701093         7.298907        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.590566 Ekcore=      3171.756639 Ektot    =     3359.347205
 Exc=    -129.938126 Ees   =     -6534.172242 EKohnSham=    -3304.763163
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 3 RMS DQ= 1.06E-4  last it= 2.36E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 1.06D-04
   tj: 0.08939   0.16483
 mixrho: add corrections to qcell smrho = -0.24057D-07 -0.30631D-09

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.763166   ehk=   -3304.763163
 From last iter    ehf=   -3304.763429   ehk=   -3304.762424
 diffe(q)=  0.000263 (0.000106)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946681 ehk(eV)=-44963.946648 sev(eV)=-38.507392

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580972
   smooth rhoves     10.096230   charge     3.702483
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655358 -3.455259 -0.832807
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001787      -184.375434      -190.377221
   rhoval*ves            -46.600151      -116.653745      -163.253896
   psnuc*ves              66.792610    -12971.812338    -12905.019728
   utot                   10.096230     -6544.233042     -6534.136812
   rho*exc                -2.655358      -127.279187      -129.934545
   rho*vxc                -3.455259      -168.536557      -171.991816
   valence chg             3.702483         7.297517        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.2050

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018829;  11.000000 electrons
         Sum occ. bands:   -2.825670, incl. Bloechl correction: -0.009157
 bndfp:Generating TDOS: efermi= -0.018829  dos window emin emax=  -0.727421  2.921093


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128558    2.830941    7.297617
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.825670 Vin*nin=    -190.377221 Ek=Eb-Vin*nin=     187.551551
 Ek(core)=    3171.756639 Exc=    -129.934545 Ees=   -6534.136812 Eharris=   -3304.763167

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.001692   -184.378525   -190.380218
    E_B(band energy sum)=   -2.825670  E_B-nout*Vin=  187.554548

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580952
   smooth rhoves     10.095643   charge     3.702383
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655277 -3.455153 -0.832799
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001572      -184.377386      -190.378958
   rhoval*ves            -46.599338      -116.655998      -163.255337
   psnuc*ves              66.790625    -12971.814350    -12905.023726
   utot                   10.095643     -6544.235174     -6534.139531
   rho*exc                -2.655277      -127.279546      -129.934822
   rho*vxc                -3.455153      -168.537030      -171.992183
   valence chg             3.702383         7.297617        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.554548 Ekcore=      3171.756639 Ektot    =     3359.311187
 Exc=    -129.934822 Ees   =     -6534.139531 EKohnSham=    -3304.763167
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 6.58E-6  last it= 1.06E-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 6.58D-06
   tj:-0.06605
 mixrho: add corrections to qcell smrho = -0.26041D-07 -0.33156D-09

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.763167   ehk=   -3304.763167
 From last iter    ehf=   -3304.763166   ehk=   -3304.763163
 diffe(q)= -0.000001 (0.000007)    tol= 0.000010 (0.000010)   more=F
c ehf(eV)=-44963.946694 ehk(eV)=-44963.946691 sev(eV)=-38.445498
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 9e69d7e280d2306a6dd11d9e49d6dc9b875e733f
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Feb 14 23:33:57 2023 +0900
INFO: linked at Wed Feb 15 13:01:38 JST 2023
===START LMF with   ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7 --band:fn=syml<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
=== SITE =1
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
=== SITE =1
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
=== SITE =1
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 3 1.000000  1.000000 beta elin wc killj=  -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0
pnu list       ibas isp  pnu(0:lmxa) 
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110  5.100
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto rsmh2    1  1.30  0.00  1.00  1.30
mto  eh2     1 -1.00 -1.00 -1.00 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  3  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol=   78.538660

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
mksym: found  48  space group operations
splcls:  ibas iclass ispec label(ispec)
          1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 m_qplistinit:start
  --- Readin syml file --- 
   41   0.5000   0.5000   0.5000    0.0000   0.0000   0.0000 L Gamma
   41   0.0000   0.0000   0.0000    1.0000   0.0000   0.0000 Gamma X
   21   1.0000   0.0000   0.0000    1.0000   0.5000   0.0000 X W
   41   1.0000   0.5000   0.0000    0.0000   0.0000   0.0000 W Gamma
nsyml nkp=    4  144
 -------- qplist --------           4
    1   0.500   0.500   0.500  <-- isyml= 001
    2   0.487   0.487   0.487 
    3   0.475   0.475   0.475 
    4   0.463   0.463   0.463 
    5   0.450   0.450   0.450 
    6   0.438   0.438   0.438 
    7   0.425   0.425   0.425 
    8   0.412   0.412   0.412 
    9   0.400   0.400   0.400 
   10   0.388   0.388   0.388 
   11   0.375   0.375   0.375 
   12   0.362   0.362   0.362 
   13   0.350   0.350   0.350 
   14   0.338   0.338   0.338 
   15   0.325   0.325   0.325 
   16   0.312   0.312   0.312 
   17   0.300   0.300   0.300 
   18   0.287   0.287   0.287 
   19   0.275   0.275   0.275 
   20   0.263   0.263   0.263 
   21   0.250   0.250   0.250 
   22   0.237   0.237   0.237 
   23   0.225   0.225   0.225 
   24   0.213   0.213   0.213 
   25   0.200   0.200   0.200 
   26   0.188   0.188   0.188 
   27   0.175   0.175   0.175 
   28   0.162   0.162   0.162 
   29   0.150   0.150   0.150 
   30   0.138   0.138   0.138 
   31   0.125   0.125   0.125 
   32   0.112   0.112   0.112 
   33   0.100   0.100   0.100 
   34   0.088   0.088   0.088 
   35   0.075   0.075   0.075 
   36   0.062   0.062   0.062 
   37   0.050   0.050   0.050 
   38   0.037   0.037   0.037 
   39   0.025   0.025   0.025 
   40   0.013   0.013   0.013 
   41   0.000   0.000   0.000 
   42   0.000   0.000   0.000  <-- isyml= 002
   43   0.025   0.000   0.000 
   44   0.050   0.000   0.000 
   45   0.075   0.000   0.000 
   46   0.100   0.000   0.000 
   47   0.125   0.000   0.000 
   48   0.150   0.000   0.000 
   49   0.175   0.000   0.000 
   50   0.200   0.000   0.000 
   51   0.225   0.000   0.000 
   52   0.250   0.000   0.000 
   53   0.275   0.000   0.000 
   54   0.300   0.000   0.000 
   55   0.325   0.000   0.000 
   56   0.350   0.000   0.000 
   57   0.375   0.000   0.000 
   58   0.400   0.000   0.000 
   59   0.425   0.000   0.000 
   60   0.450   0.000   0.000 
   61   0.475   0.000   0.000 
   62   0.500   0.000   0.000 
   63   0.525   0.000   0.000 
   64   0.550   0.000   0.000 
   65   0.575   0.000   0.000 
   66   0.600   0.000   0.000 
   67   0.625   0.000   0.000 
   68   0.650   0.000   0.000 
   69   0.675   0.000   0.000 
   70   0.700   0.000   0.000 
   71   0.725   0.000   0.000 
   72   0.750   0.000   0.000 
   73   0.775   0.000   0.000 
   74   0.800   0.000   0.000 
   75   0.825   0.000   0.000 
   76   0.850   0.000   0.000 
   77   0.875   0.000   0.000 
   78   0.900   0.000   0.000 
   79   0.925   0.000   0.000 
   80   0.950   0.000   0.000 
   81   0.975   0.000   0.000 
   82   1.000   0.000   0.000 
   83   1.000   0.000   0.000  <-- isyml= 003
   84   1.000   0.025   0.000 
   85   1.000   0.050   0.000 
   86   1.000   0.075   0.000 
   87   1.000   0.100   0.000 
   88   1.000   0.125   0.000 
   89   1.000   0.150   0.000 
   90   1.000   0.175   0.000 
   91   1.000   0.200   0.000 
   92   1.000   0.225   0.000 
   93   1.000   0.250   0.000 
   94   1.000   0.275   0.000 
   95   1.000   0.300   0.000 
   96   1.000   0.325   0.000 
   97   1.000   0.350   0.000 
   98   1.000   0.375   0.000 
   99   1.000   0.400   0.000 
  100   1.000   0.425   0.000 
  101   1.000   0.450   0.000 
  102   1.000   0.475   0.000 
  103   1.000   0.500   0.000 
  104   1.000   0.500   0.000  <-- isyml= 004
  105   0.975   0.487   0.000 
  106   0.950   0.475   0.000 
  107   0.925   0.463   0.000 
  108   0.900   0.450   0.000 
  109   0.875   0.438   0.000 
  110   0.850   0.425   0.000 
  111   0.825   0.412   0.000 
  112   0.800   0.400   0.000 
  113   0.775   0.388   0.000 
  114   0.750   0.375   0.000 
  115   0.725   0.362   0.000 
  116   0.700   0.350   0.000 
  117   0.675   0.338   0.000 
  118   0.650   0.325   0.000 
  119   0.625   0.312   0.000 
  120   0.600   0.300   0.000 
  121   0.575   0.287   0.000 
  122   0.550   0.275   0.000 
  123   0.525   0.263   0.000 
  124   0.500   0.250   0.000 
  125   0.475   0.237   0.000 
  126   0.450   0.225   0.000 
  127   0.425   0.213   0.000 
  128   0.400   0.200   0.000 
  129   0.375   0.188   0.000 
  130   0.350   0.175   0.000 
  131   0.325   0.162   0.000 
  132   0.300   0.150   0.000 
  133   0.275   0.138   0.000 
  134   0.250   0.125   0.000 
  135   0.225   0.112   0.000 
  136   0.200   0.100   0.000 
  137   0.175   0.088   0.000 
  138   0.150   0.075   0.000 
  139   0.125   0.062   0.000 
  140   0.100   0.050   0.000 
  141   0.075   0.037   0.000 
  142   0.050   0.025   0.000 
  143   0.025   0.013   0.000 
  144   0.000   0.000   0.000 
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.38  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 1
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580968
   smooth rhoves     10.096127   charge     3.702468
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655347 -3.455244 -0.832805
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001767      -184.375801      -190.377568
   rhoval*ves            -46.600003      -116.654177      -163.254180
   psnuc*ves              66.792258    -12971.812819    -12905.020561
   utot                   10.096127     -6544.233498     -6534.137371
   rho*exc                -2.655347      -127.279257      -129.934604
   rho*vxc                -3.455244      -168.536649      -171.991894
   valence chg             3.702468         7.297532        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of   144 k=  0.5000  0.5000  0.5000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of   144 k=  0.4875  0.4875  0.4875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of   144 k=  0.4750  0.4750  0.4750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of   144 k=  0.4625  0.4625  0.4625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of   144 k=  0.4500  0.4500  0.4500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of   144 k=  0.4375  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of   144 k=  0.4250  0.4250  0.4250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of   144 k=  0.4125  0.4125  0.4125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of   144 k=  0.4000  0.4000  0.4000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of   144 k=  0.3875  0.3875  0.3875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of   144 k=  0.3750  0.3750  0.3750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of   144 k=  0.3625  0.3625  0.3625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of   144 k=  0.3500  0.3500  0.3500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of   144 k=  0.3375  0.3375  0.3375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of   144 k=  0.3250  0.3250  0.3250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    16 of   144 k=  0.3125  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    17 of   144 k=  0.3000  0.3000  0.3000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    18 of   144 k=  0.2875  0.2875  0.2875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    19 of   144 k=  0.2750  0.2750  0.2750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    20 of   144 k=  0.2625  0.2625  0.2625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    21 of   144 k=  0.2500  0.2500  0.2500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    22 of   144 k=  0.2375  0.2375  0.2375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    23 of   144 k=  0.2250  0.2250  0.2250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    24 of   144 k=  0.2125  0.2125  0.2125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    25 of   144 k=  0.2000  0.2000  0.2000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    26 of   144 k=  0.1875  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    27 of   144 k=  0.1750  0.1750  0.1750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    28 of   144 k=  0.1625  0.1625  0.1625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    29 of   144 k=  0.1500  0.1500  0.1500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    30 of   144 k=  0.1375  0.1375  0.1375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    31 of   144 k=  0.1250  0.1250  0.1250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    32 of   144 k=  0.1125  0.1125  0.1125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    33 of   144 k=  0.1000  0.1000  0.1000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    34 of   144 k=  0.0875  0.0875  0.0875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    35 of   144 k=  0.0750  0.0750  0.0750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    36 of   144 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.4821
  Writing bands to bands file for gnuplot ...
 bndfp: kpt    1 of  144 k jsp=  0.50000  0.50000  0.50000 1 nev=   31
 bndfp: kpt    2 of  144 k jsp=  0.48750  0.48750  0.48750 1 nev=   31
 bndfp: kpt    3 of  144 k jsp=  0.47500  0.47500  0.47500 1 nev=   31
 bndfp: kpt    4 of  144 k jsp=  0.46250  0.46250  0.46250 1 nev=   31
 bndfp: kpt    5 of  144 k jsp=  0.45000  0.45000  0.45000 1 nev=   31
 bndfp: kpt    6 of  144 k jsp=  0.43750  0.43750  0.43750 1 nev=   31
 bndfp: kpt    7 of  144 k jsp=  0.42500  0.42500  0.42500 1 nev=   31
 bndfp: kpt    8 of  144 k jsp=  0.41250  0.41250  0.41250 1 nev=   31
 bndfp: kpt    9 of  144 k jsp=  0.40000  0.40000  0.40000 1 nev=   31
 bndfp: kpt   10 of  144 k jsp=  0.38750  0.38750  0.38750 1 nev=   31
 bndfp: kpt   11 of  144 k jsp=  0.37500  0.37500  0.37500 1 nev=   31
 bndfp: kpt   12 of  144 k jsp=  0.36250  0.36250  0.36250 1 nev=   31
 bndfp: kpt   13 of  144 k jsp=  0.35000  0.35000  0.35000 1 nev=   31
 bndfp: kpt   14 of  144 k jsp=  0.33750  0.33750  0.33750 1 nev=   31
 bndfp: kpt   15 of  144 k jsp=  0.32500  0.32500  0.32500 1 nev=   31
 bndfp: kpt   16 of  144 k jsp=  0.31250  0.31250  0.31250 1 nev=   31
 bndfp: kpt   17 of  144 k jsp=  0.30000  0.30000  0.30000 1 nev=   31
 bndfp: kpt   18 of  144 k jsp=  0.28750  0.28750  0.28750 1 nev=   31
 bndfp: kpt   19 of  144 k jsp=  0.27500  0.27500  0.27500 1 nev=   31
 bndfp: kpt   20 of  144 k jsp=  0.26250  0.26250  0.26250 1 nev=   31
 bndfp: kpt   21 of  144 k jsp=  0.25000  0.25000  0.25000 1 nev=   31
 bndfp: kpt   22 of  144 k jsp=  0.23750  0.23750  0.23750 1 nev=   31
 bndfp: kpt   23 of  144 k jsp=  0.22500  0.22500  0.22500 1 nev=   31
 bndfp: kpt   24 of  144 k jsp=  0.21250  0.21250  0.21250 1 nev=   31
 bndfp: kpt   25 of  144 k jsp=  0.20000  0.20000  0.20000 1 nev=   31
 bndfp: kpt   26 of  144 k jsp=  0.18750  0.18750  0.18750 1 nev=   31
 bndfp: kpt   27 of  144 k jsp=  0.17500  0.17500  0.17500 1 nev=   31
 bndfp: kpt   28 of  144 k jsp=  0.16250  0.16250  0.16250 1 nev=   31
 bndfp: kpt   29 of  144 k jsp=  0.15000  0.15000  0.15000 1 nev=   31
 bndfp: kpt   30 of  144 k jsp=  0.13750  0.13750  0.13750 1 nev=   31
 bndfp: kpt   31 of  144 k jsp=  0.12500  0.12500  0.12500 1 nev=   31
 bndfp: kpt   32 of  144 k jsp=  0.11250  0.11250  0.11250 1 nev=   31
 bndfp: kpt   33 of  144 k jsp=  0.10000  0.10000  0.10000 1 nev=   31
 bndfp: kpt   34 of  144 k jsp=  0.08750  0.08750  0.08750 1 nev=   31
 bndfp: kpt   35 of  144 k jsp=  0.07500  0.07500  0.07500 1 nev=   31
 bndfp: kpt   36 of  144 k jsp=  0.06250  0.06250  0.06250 1 nev=   31
 bndfp: kpt   37 of  144 k jsp=  0.05000  0.05000  0.05000 1 nev=   31
 bndfp: kpt   38 of  144 k jsp=  0.03750  0.03750  0.03750 1 nev=   31
 bndfp: kpt   39 of  144 k jsp=  0.02500  0.02500  0.02500 1 nev=   31
 bndfp: kpt   40 of  144 k jsp=  0.01250  0.01250  0.01250 1 nev=   31
 bndfp: kpt   41 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   42 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   43 of  144 k jsp=  0.02500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   44 of  144 k jsp=  0.05000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   45 of  144 k jsp=  0.07500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   46 of  144 k jsp=  0.10000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   47 of  144 k jsp=  0.12500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   48 of  144 k jsp=  0.15000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   49 of  144 k jsp=  0.17500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   50 of  144 k jsp=  0.20000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   51 of  144 k jsp=  0.22500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   52 of  144 k jsp=  0.25000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   53 of  144 k jsp=  0.27500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   54 of  144 k jsp=  0.30000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   55 of  144 k jsp=  0.32500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   56 of  144 k jsp=  0.35000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   57 of  144 k jsp=  0.37500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   58 of  144 k jsp=  0.40000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   59 of  144 k jsp=  0.42500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   60 of  144 k jsp=  0.45000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   61 of  144 k jsp=  0.47500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   62 of  144 k jsp=  0.50000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   63 of  144 k jsp=  0.52500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   64 of  144 k jsp=  0.55000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   65 of  144 k jsp=  0.57500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   66 of  144 k jsp=  0.60000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   67 of  144 k jsp=  0.62500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   68 of  144 k jsp=  0.65000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   69 of  144 k jsp=  0.67500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   70 of  144 k jsp=  0.70000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   71 of  144 k jsp=  0.72500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   72 of  144 k jsp=  0.75000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   73 of  144 k jsp=  0.77500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   74 of  144 k jsp=  0.80000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   75 of  144 k jsp=  0.82500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   76 of  144 k jsp=  0.85000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   77 of  144 k jsp=  0.87500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   78 of  144 k jsp=  0.90000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   79 of  144 k jsp=  0.92500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   80 of  144 k jsp=  0.95000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   81 of  144 k jsp=  0.97500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   82 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   83 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   84 of  144 k jsp=  1.00000  0.02500  0.00000 1 nev=   31
 bndfp: kpt   85 of  144 k jsp=  1.00000  0.05000  0.00000 1 nev=   31
 bndfp: kpt   86 of  144 k jsp=  1.00000  0.07500  0.00000 1 nev=   31
 bndfp: kpt   87 of  144 k jsp=  1.00000  0.10000  0.00000 1 nev=   31
 bndfp: kpt   88 of  144 k jsp=  1.00000  0.12500  0.00000 1 nev=   31
 bndfp: kpt   89 of  144 k jsp=  1.00000  0.15000  0.00000 1 nev=   31
 bndfp: kpt   90 of  144 k jsp=  1.00000  0.17500  0.00000 1 nev=   31
 bndfp: kpt   91 of  144 k jsp=  1.00000  0.20000  0.00000 1 nev=   31
 bndfp: kpt   92 of  144 k jsp=  1.00000  0.22500  0.00000 1 nev=   31
 bndfp: kpt   93 of  144 k jsp=  1.00000  0.25000  0.00000 1 nev=   31
 bndfp: kpt   94 of  144 k jsp=  1.00000  0.27500  0.00000 1 nev=   31
 bndfp: kpt   95 of  144 k jsp=  1.00000  0.30000  0.00000 1 nev=   31
 bndfp: kpt   96 of  144 k jsp=  1.00000  0.32500  0.00000 1 nev=   31
 bndfp: kpt   97 of  144 k jsp=  1.00000  0.35000  0.00000 1 nev=   31
 bndfp: kpt   98 of  144 k jsp=  1.00000  0.37500  0.00000 1 nev=   31
 bndfp: kpt   99 of  144 k jsp=  1.00000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  100 of  144 k jsp=  1.00000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  101 of  144 k jsp=  1.00000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  102 of  144 k jsp=  1.00000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  103 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  104 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  105 of  144 k jsp=  0.97500  0.48750  0.00000 1 nev=   31
 bndfp: kpt  106 of  144 k jsp=  0.95000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  107 of  144 k jsp=  0.92500  0.46250  0.00000 1 nev=   31
 bndfp: kpt  108 of  144 k jsp=  0.90000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  109 of  144 k jsp=  0.87500  0.43750  0.00000 1 nev=   31
 bndfp: kpt  110 of  144 k jsp=  0.85000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  111 of  144 k jsp=  0.82500  0.41250  0.00000 1 nev=   31
 bndfp: kpt  112 of  144 k jsp=  0.80000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  113 of  144 k jsp=  0.77500  0.38750  0.00000 1 nev=   31
 bndfp: kpt  114 of  144 k jsp=  0.75000  0.37500  0.00000 1 nev=   31
 bndfp: kpt  115 of  144 k jsp=  0.72500  0.36250  0.00000 1 nev=   31
 bndfp: kpt  116 of  144 k jsp=  0.70000  0.35000  0.00000 1 nev=   31
 bndfp: kpt  117 of  144 k jsp=  0.67500  0.33750  0.00000 1 nev=   31
 bndfp: kpt  118 of  144 k jsp=  0.65000  0.32500  0.00000 1 nev=   31
 bndfp: kpt  119 of  144 k jsp=  0.62500  0.31250  0.00000 1 nev=   31
 bndfp: kpt  120 of  144 k jsp=  0.60000  0.30000  0.00000 1 nev=   31
 bndfp: kpt  121 of  144 k jsp=  0.57500  0.28750  0.00000 1 nev=   31
 bndfp: kpt  122 of  144 k jsp=  0.55000  0.27500  0.00000 1 nev=   31
 bndfp: kpt  123 of  144 k jsp=  0.52500  0.26250  0.00000 1 nev=   31
 bndfp: kpt  124 of  144 k jsp=  0.50000  0.25000  0.00000 1 nev=   31
 bndfp: kpt  125 of  144 k jsp=  0.47500  0.23750  0.00000 1 nev=   31
 bndfp: kpt  126 of  144 k jsp=  0.45000  0.22500  0.00000 1 nev=   31
 bndfp: kpt  127 of  144 k jsp=  0.42500  0.21250  0.00000 1 nev=   31
 bndfp: kpt  128 of  144 k jsp=  0.40000  0.20000  0.00000 1 nev=   31
 bndfp: kpt  129 of  144 k jsp=  0.37500  0.18750  0.00000 1 nev=   31
 bndfp: kpt  130 of  144 k jsp=  0.35000  0.17500  0.00000 1 nev=   31
 bndfp: kpt  131 of  144 k jsp=  0.32500  0.16250  0.00000 1 nev=   31
 bndfp: kpt  132 of  144 k jsp=  0.30000  0.15000  0.00000 1 nev=   31
 bndfp: kpt  133 of  144 k jsp=  0.27500  0.13750  0.00000 1 nev=   31
 bndfp: kpt  134 of  144 k jsp=  0.25000  0.12500  0.00000 1 nev=   31
 bndfp: kpt  135 of  144 k jsp=  0.22500  0.11250  0.00000 1 nev=   31
 bndfp: kpt  136 of  144 k jsp=  0.20000  0.10000  0.00000 1 nev=   31
 bndfp: kpt  137 of  144 k jsp=  0.17500  0.08750  0.00000 1 nev=   31
 bndfp: kpt  138 of  144 k jsp=  0.15000  0.07500  0.00000 1 nev=   31
 bndfp: kpt  139 of  144 k jsp=  0.12500  0.06250  0.00000 1 nev=   31
 bndfp: kpt  140 of  144 k jsp=  0.10000  0.05000  0.00000 1 nev=   31
 bndfp: kpt  141 of  144 k jsp=  0.07500  0.03750  0.00000 1 nev=   31
 bndfp: kpt  142 of  144 k jsp=  0.05000  0.02500  0.00000 1 nev=   31
 bndfp: kpt  143 of  144 k jsp=  0.02500  0.01250  0.00000 1 nev=   31
 bndfp: kpt  144 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
Exit 0 procid= 0 plot band mode done
