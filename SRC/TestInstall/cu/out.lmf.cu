#include "hip/hip_runtime.h"
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 913e769c0a5a77a2254ce7ce7011c5bc7fb5168a
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Mon Feb 13 19:43:59 2023 +0900
INFO: linked at Tue Feb 14 12:44:43 JST 2023
=== START LFMA ===
 mpisize=           1
 m_lmfinit:program LMFA
 cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 2.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0

 ... Species  1
  bndfp (warning): no sigm file found ... LDA calculation only
pnuall: j isp pnu= 1 1 4.650000  4.340000  3.870000  4.110000
pnzall: j isp  pz= 1 1 5.500000  5.500000  4.500000  0.000000


mmm === MTO setting ===
mmm ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mmm rsmh1    1  2.50  2.50  1.00
mmm   eh1    1 -0.01 -0.01 -0.01
mmm pz       1  5.50  5.50  4.50
mmm lh       2  2
xxx isp pz= 1 5.500000  5.500000  4.500000  0.000000
 goto freats

conf:------------------------------------------------------
conf:SPEC_ATOM= A : --- Table for atomic configuration ---
conf:  isp  l  int(P) int(P)z    Qval     Qcore   CoreConf
conf:    1  0       4  5         1.000    6.000 => 1,2,3,
conf:    1  1       4  5         0.000   12.000 => 2,3,
conf:    1  2       3  4        10.000    0.000 => 
conf:    1  3       4  0         0.000    0.000 => 
usedQ=     1.000     0.000    10.000     0.000
conf: Species  A        Z=  29.00 Qc=  18.000 R=  2.311271 Q=  0.000000 nsp= 1 mom=  0.000000
conf: rmt rmax a=  2.311271  48.805862  0.025000 nrmt nr= 393 515
 goto atomc xxx
 atomsc nmcore=           0

 end of atomsc xxxxx
 vsum=  -130.79144076069792                1
sumev= -4.333254 etot= -3304.416258 eref=  0.000000 etot-eref= -3304.416258

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)       pnu
   4s      -0.36411         0.890       2.256       3.582     0.643062       4.761  0
   5s      -0.00028         3.669      10.794      19.873     0.990448       5.848  1
   4p      -0.06295         0.975       3.484       7.414     0.901829       4.561  0
   5p       0.00796         6.760      30.414      48.806*    0.999240       5.593  1
   3d      -0.39691         0.000       0.600       3.429     0.056076       3.888  0
   4d       0.01308         1.868      33.290      48.806*    0.999995       4.148  1
   4f       0.01948         0.000      35.393      48.806*    1.000000       4.137  0

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000141
   3p      -5.29682         0.260       0.619       1.078     0.000727
 tailsm: init

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.31127, rsm= 1.15564
  ---E:energies of smHankels. C:fitting coeeficient for core tail. ---
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000   -15.00000
 C:    -0.07160    10.75053  -187.49213  1222.02349 -4717.78530 21166.80769
        r          rho         fit         diff
    2.311271    0.017797    0.017766    0.000031
    2.967767    0.005662    0.005658    0.000005
    3.810725    0.001517    0.001518   -0.000001
    4.893104    0.000305    0.000305   -0.000000
    6.282906    0.000041    0.000041   -0.000001
    8.067448    0.000003    0.000003    0.000000
    q(fit):     1.203836    rms diff:   0.000016
    fit: r>rmt  1.203836   r<rmt  3.442816   qtot  4.646652
    rho: r>rmt  1.203836   r<rmt  9.796164   qtot 11.000000
 tailsm:  fit tails to        6 functions with

 rsm=  0.11556D+01 rms error=  0.16285D-04
conf: Core rhoc(rmt)= 0.003922 spillout= 0.004646
 Fit with Hankel e=-24.082483 coeff=764.352513
      r            rhoc          fit
    2.311271    0.02095279    0.02095279
    2.429779    0.01229068    0.01231367
    2.753317    0.00285262    0.00285190
    3.119934    0.00054243    0.00053465
    3.535366    0.00008235    0.00007888
    4.006112    0.00000969    0.00000887
    4.539536    0.00000085    0.00000073
    5.143985    0.00000005    0.00000004
 end of freats: spid nmcore=A                  0
Sum of reference energies:                      0.000000000000
OK! end of LMFA ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 913e769c0a5a77a2254ce7ce7011c5bc7fb5168a
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Mon Feb 13 19:43:59 2023 +0900
INFO: linked at Tue Feb 14 12:44:43 JST 2023
 m_lmfinit:program LMF
===START LMF with   ===
 mpisize=           4
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=f<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1

 ... Species  1
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1

 ... Species  1

 ... Species  1
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 2.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0

 ... Species  1
  bndfp (warning): no sigm file found ... LDA calculation only
pnuall: j isp pnu= 1 1 4.650000  4.340000  3.870000  4.110000
pnzall: j isp  pz= 1 1 5.500000  5.500000  4.500000  0.000000
 imx=           3           3           3
 imx=           3           3           3


mmm === MTO setting ===
mmm ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mmm rsmh1    1  2.50  2.50  1.00
mmm   eh1    1 -0.01 -0.01 -0.01
mmm pz       1  5.50  5.50  4.50
mmm lh       2  2
 imx=           3           3           3
 imx=           4           4           4
 imx=           4           4           4

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
 imx=           4           4           4
  Cell vol=   78.538660
 imx=           3           3           3
 imx=           4           4           4

 LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
 nnnnnn         729         889
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
MKSYM: found  48  space group operations
SPLCLS: ibas iclass ispec label(ispec)
 SPLCLS     1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
 nnnnnn         729         889
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
 m_qplistinit:start
 nnnnnn         729         889
 nnnnnn         729         889

 iors  : read rst restart file (binary mesh density)
 iors  : empty file ... nothing read

rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected A,       read A        with rmt=  2.3113  mesh   393  0.025
  ovlpfa: overlap smooth part of FA densities
 site 1 spec 1 pos 0.0000  0.0000  0.0000 Qsmooth 4.6466523386126539 mom 4.6466523386126539
 total smooth Q =  4.6466523386126539

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.796164    3.442816   10.275300    3.921952    6.353348

 Smooth charge on mesh:            4.646652
 Sum of local charges:             6.353348
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:       -0.000000

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.555104
   smooth rhoves     11.022231   charge     4.646652
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.843799 -5.010453 -0.851784
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:   -0.000099
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff            -12.157495      -177.337532      -189.495027
   rhoval*ves            -46.690633      -115.324376      -162.015010
   psnuc*ves              68.735095    -12976.662436    -12907.927341
   utot                   11.022231     -6545.993406     -6534.971175
   rho*exc                -3.843799      -126.414298      -130.258096
   rho*vxc                -5.010453      -167.409316      -172.419769
   valence chg             4.646652         6.353348        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0382

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144466;  11.000000 electrons
         Sum occ. bands:   -0.854464, incl. Bloechl correction: -0.006586
 bndfp:Generating TDOS: efermi=  0.144466  dos window emin emax=  -0.672856  3.084388


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1    9.927753    3.113496    6.814257
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -0.854464 Vin*nin=    -189.495027 Ek=Eb-Vin*nin=     188.640563
 Ek(core)=    3171.756639 Exc=    -130.258096 Ees=   -6534.971175 Eharris=   -3304.832069

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.361301   -168.223682   -174.584983
    E_B(band energy sum)=   -0.854464  E_B-nout*Vin=  173.730519

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178923   charge     4.185743
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054120 -3.974966 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030715      -175.114087      -182.144802
   rhoval*ves            -50.183969      -106.215150      -156.399119
   psnuc*ves              76.541815    -12962.871088    -12886.329272
   utot                   13.178923     -6534.543119     -6521.364196
   rho*exc                -3.054120      -125.587137      -128.641257
   rho*vxc                -3.974966      -166.302309      -170.277275
   valence chg             4.185743         6.814257        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.730519 Ekcore=      3171.756639 Ektot    =     3345.487159
 Exc=    -128.641257 Ees   =     -6521.364196 EKohnSham=    -3304.518294
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 0 RMS DQ= 4.34E-2
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.34D-02
 mixrho: add corrections to qcell smrho = -0.23414D-07 -0.29812D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.832069   ehk=   -3304.518294
h ehf(eV)=-44964.884163 ehk(eV)=-44960.615006 sev(eV)=-11.625667

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178923   charge     4.185743
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054120 -3.974966 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000553
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030715      -175.114087      -182.144801
   rhoval*ves            -50.183969      -106.215150      -156.399119
   psnuc*ves              76.541815    -12962.871088    -12886.329272
   utot                   13.178923     -6534.543119     -6521.364196
   rho*exc                -3.054120      -125.587137      -128.641257
   rho*vxc                -3.974966      -166.302309      -170.277275
   valence chg             4.185743         6.814257        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0588

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.257389;  11.000000 electrons
         Sum occ. bands:   -9.368175, incl. Bloechl correction: -0.013546
 bndfp:Generating TDOS: efermi= -0.257389  dos window emin emax=  -0.960038  2.682534


 mkrout:  Qtrue      sm,loc       local
   1   10.453470    1.685884    8.767586
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -9.368175 Vin*nin=    -182.144801 Ek=Eb-Vin*nin=     172.776627
 Ek(core)=    3171.756639 Exc=    -128.641257 Ees=   -6521.364196 Eharris=   -3305.472187

 mkekin:
   nout*Vin = smpart,onsite,total=:     -3.908741   -233.344935   -237.253675
    E_B(band energy sum)=   -9.368175  E_B-nout*Vin=  227.885500

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.359167
   smooth rhoves      3.889627   charge     2.232413
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.459173 -1.896418 -0.723230
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -2.683250      -210.677503      -213.360752
   rhoval*ves            -33.661409      -148.825746      -182.487155
   psnuc*ves              41.440663    -12997.718880    -12956.278216
   utot                    3.889627     -6573.272313     -6569.382686
   rho*exc                -1.459173      -131.947804      -133.406976
   rho*vxc                -1.896418      -174.700211      -176.596629
   valence chg             2.232413         8.767586        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      227.885500 Ekcore=      3171.756639 Ektot    =     3399.642140
 Exc=    -133.406976 Ees   =     -6569.382686 EKohnSham=    -3303.147522
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 1 RMS DQ= 1.23E-1  last it= 4.34E-2
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.23D-01
   tj: 0.82062
 mixrho: add corrections to qcell smrho = -0.96492D-08 -0.12286D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3305.472187   ehk=   -3303.147522
 From last iter    ehf=   -3304.832069   ehk=   -3304.518294
 diffe(q)= -0.640118 (0.123166)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44973.593479 ehk(eV)=-44941.964555 sev(eV)=-127.461515

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.620291
   smooth rhoves     11.107548   charge     3.835349
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.748140 -3.575952 -0.844926
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000503
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.127022      -182.314718      -188.441740
   rhoval*ves            -48.030181      -113.790712      -161.820893
   psnuc*ves              70.245276    -12969.122178    -12898.876902
   utot                   11.107548     -6541.456445     -6530.348897
   rho*exc                -2.748140      -126.718973      -129.467113
   rho*vxc                -3.575952      -167.796505      -171.372457
   valence chg             3.835349         7.164651        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0534

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.148169;  11.000000 electrons
         Sum occ. bands:   -5.251649, incl. Bloechl correction: -0.012268
 bndfp:Generating TDOS: efermi= -0.148169  dos window emin emax=  -0.774071  2.791753


 mkrout:  Qtrue      sm,loc       local
   1   10.297519    2.200427    8.097092
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -5.251649 Vin*nin=    -188.441740 Ek=Eb-Vin*nin=     183.190091
 Ek(core)=    3171.756639 Exc=    -129.467113 Ees=   -6530.348897 Eharris=   -3304.869280

 mkekin:
   nout*Vin = smpart,onsite,total=:     -4.778849   -205.317019   -210.095869
    E_B(band energy sum)=   -5.251649  E_B-nout*Vin=  204.844220

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.463277
   smooth rhoves      6.414066   charge     2.902909
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.980933 -2.576057 -0.778176
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.013607      -196.297023      -200.310630
   rhoval*ves            -40.418130      -131.103281      -171.521411
   psnuc*ves              53.246261    -12980.824930    -12927.578668
   utot                    6.414066     -6555.964106     -6549.550040
   rho*exc                -1.980933      -129.503698      -131.484631
   rho*vxc                -2.576057      -171.471048      -174.047106
   valence chg             2.902909         8.097092        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      204.844220 Ekcore=      3171.756639 Ektot    =     3376.600859
 Exc=    -131.484631 Ees   =     -6549.550040 EKohnSham=    -3304.433811
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 2 RMS DQ= 5.12E-2  last it= 1.23E-1
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 5.12D-02
   tj:-0.76791  -0.08165
 mixrho: add corrections to qcell smrho = -0.31521D-06 -0.40134D-08

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.869280   ehk=   -3304.433811
 From last iter    ehf=   -3305.472187   ehk=   -3303.147522
 diffe(q)=  0.602907 (0.051213)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44965.390444 ehk(eV)=-44959.465546 sev(eV)=-71.452882

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.525743
   smooth rhoves      8.256180   charge     3.313051
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.319647 -3.017521 -0.807475
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000461
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.941480      -185.839184      -190.780664
   rhoval*ves            -43.897574      -119.382195      -163.279769
   psnuc*ves              60.409935    -12969.317726    -12908.907791
   utot                    8.256180     -6544.349961     -6536.093780
   rho*exc                -2.319647      -127.961918      -130.281565
   rho*vxc                -3.017521      -169.433789      -172.451310
   valence chg             3.313051         7.686949        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0537

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.132346;  11.000000 electrons
         Sum occ. bands:   -0.993219, incl. Bloechl correction: -0.006627
 bndfp:Generating TDOS: efermi=  0.132346  dos window emin emax=  -0.684035  3.072269


 mkrout:  Qtrue      sm,loc       local
   1    9.930892    3.113709    6.817183
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -0.993219 Vin*nin=    -190.780664 Ek=Eb-Vin*nin=     189.787444
 Ek(core)=    3171.756639 Exc=    -130.281565 Ees=   -6536.093780 Eharris=   -3304.831261

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.042093   -168.378044   -174.420138
    E_B(band energy sum)=   -0.993219  E_B-nout*Vin=  173.426918

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.676372
   smooth rhoves     13.152472   charge     4.182817
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.052152 -3.972407 -0.866429
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.029083      -174.837177      -181.866260
   rhoval*ves            -50.154645      -105.981094      -156.135739
   psnuc*ves              76.459589    -12962.467179    -12886.007589
   utot                   13.152472     -6534.224136     -6521.071664
   rho*exc                -3.052152      -125.574326      -128.626478
   rho*vxc                -3.972407      -166.285214      -170.257621
   valence chg             4.182817         6.817183        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.426918 Ekcore=      3171.756639 Ektot    =     3345.183558
 Exc=    -128.626478 Ees   =     -6521.071664 EKohnSham=    -3304.514584
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 3 RMS DQ= 4.22E-2  last it= 5.12E-2
 AMIX: nmix=3 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.22D-02
   tj: 0.74166  -0.16547   0.00023
 mixrho: add corrections to qcell smrho = -0.17530D-06 -0.22320D-08

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.831261   ehk=   -3304.514584
 From last iter    ehf=   -3304.869280   ehk=   -3304.433811
 diffe(q)=  0.038018 (0.042227)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44964.873174 ehk(eV)=-44960.564532 sev(eV)=-13.513546

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570816
   smooth rhoves      9.541427   charge     3.556294
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515298 -3.272452 -0.825708
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.474529      -185.001328      -190.475858
   rhoval*ves            -45.954938      -117.383196      -163.338134
   psnuc*ves              65.037792    -12970.249397    -12905.211605
   utot                    9.541427     -6543.816296     -6534.274870
   rho*exc                -2.515298      -127.432934      -129.948232
   rho*vxc                -3.272452      -168.737527      -172.009979
   valence chg             3.556294         7.443706        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0535

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014879;  11.000000 electrons
         Sum occ. bands:   -2.770224, incl. Bloechl correction: -0.009077
 bndfp:Generating TDOS: efermi= -0.014879  dos window emin emax=  -0.725887  2.925044


 mkrout:  Qtrue      sm,loc       local
   1   10.105683    2.685743    7.419940
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.770224 Vin*nin=    -190.475858 Ek=Eb-Vin*nin=     187.705634
 Ek(core)=    3171.756639 Exc=    -129.948232 Ees=   -6534.274870 Eharris=   -3304.760829

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.495500   -184.335642   -189.831142
    E_B(band energy sum)=   -2.770224  E_B-nout*Vin=  187.060918

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575486
   smooth rhoves      9.678781   charge     3.580060
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.534251 -3.297145 -0.827512
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.522086      -184.602389      -190.124476
   rhoval*ves            -46.157297      -116.896962      -163.054259
   psnuc*ves              65.514859    -12969.845858    -12904.330999
   utot                    9.678781     -6543.371410     -6533.692629
   rho*exc                -2.534251      -127.351062      -129.885313
   rho*vxc                -3.297145      -168.629433      -171.926578
   valence chg             3.580060         7.419940        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.060918 Ekcore=      3171.756639 Ektot    =     3358.817557
 Exc=    -129.885313 Ees   =     -6533.692629 EKohnSham=    -3304.760384
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 4 RMS DQ= 1.53E-3  last it= 4.22E-2
 AMIX: nmix=4 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.53D-03
   tj: 0.42366   0.74179  -0.16550   0.00023
 mixrho: add corrections to qcell smrho = -0.17533D-06 -0.22324D-08

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.760829   ehk=   -3304.760384
 From last iter    ehf=   -3304.831261   ehk=   -3304.514584
 diffe(q)=  0.070433 (0.001529)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914882 ehk(eV)=-44963.908839 sev(eV)=-37.691115

--- BNDFP:  begin iteration 6 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570815
   smooth rhoves      9.541421   charge     3.556293
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515298 -3.272452 -0.825708
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.474530      -185.001335      -190.475864
   rhoval*ves            -45.954929      -117.383208      -163.338137
   psnuc*ves              65.037772    -12970.249413    -12905.211641
   utot                    9.541421     -6543.816310     -6534.274889
   rho*exc                -2.515298      -127.432937      -129.948235
   rho*vxc                -3.272452      -168.737531      -172.009983
   valence chg             3.556293         7.443707        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0534

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014878;  11.000000 electrons
         Sum occ. bands:   -2.770209, incl. Bloechl correction: -0.009077
 bndfp:Generating TDOS: efermi= -0.014878  dos window emin emax=  -0.725888  2.925045


 mkrout:  Qtrue      sm,loc       local
   1   10.108512    2.682701    7.425811
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.770209 Vin*nin=    -190.475864 Ek=Eb-Vin*nin=     187.705656
 Ek(core)=    3171.756639 Exc=    -129.948235 Ees=   -6534.274889 Eharris=   -3304.760829

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.491139   -184.470980   -189.962118
    E_B(band energy sum)=   -2.770209  E_B-nout*Vin=  187.191909

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.574217
   smooth rhoves      9.644003   charge     3.574189
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.529637 -3.291135 -0.827058
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.510705      -184.677648      -190.188353
   rhoval*ves            -46.105318      -116.998090      -163.103408
   psnuc*ves              65.393325    -12969.910108    -12904.516783
   utot                    9.644003     -6543.454099     -6533.810095
   rho*exc                -2.529637      -127.369377      -129.899013
   rho*vxc                -3.291135      -168.653599      -171.944734
   valence chg             3.574189         7.425811        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.191909 Ekcore=      3171.756639 Ektot    =     3358.948549
 Exc=    -129.899013 Ees   =     -6533.810095 EKohnSham=    -3304.760560
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 5 RMS DQ= 1.21E-3  last it= 1.53E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.21D-03
   tj:-3.74145
 mixrho: add corrections to qcell smrho = -0.21809D-06 -0.27768D-08

 iors  : write rst restart file (binary mesh density)

   it  6  of 12    ehf=   -3304.760829   ehk=   -3304.760560
 From last iter    ehf=   -3304.760829   ehk=   -3304.760384
 diffe(q)= -0.000000 (0.001212)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914886 ehk(eV)=-44963.911226 sev(eV)=-37.690905

--- BNDFP:  begin iteration 7 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569469
   smooth rhoves      9.514455   charge     3.552221
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.512407 -3.268691 -0.825351
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000478
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.468164      -184.958377      -190.426541
   rhoval*ves            -45.909698      -117.376720      -163.286417
   psnuc*ves              64.938608    -12970.150494    -12905.211886
   utot                    9.514455     -6543.763607     -6534.249152
   rho*exc                -2.512407      -127.437916      -129.950323
   rho*vxc                -3.268691      -168.744037      -172.012728
   valence chg             3.552221         7.447779        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0563

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012923;  11.000000 electrons
         Sum occ. bands:   -2.744580, incl. Bloechl correction: -0.009050
 bndfp:Generating TDOS: efermi= -0.012923  dos window emin emax=  -0.724778  2.927000


 mkrout:  Qtrue      sm,loc       local
   1   10.106951    2.686361    7.420590
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.744580 Vin*nin=    -190.426541 Ek=Eb-Vin*nin=     187.681961
 Ek(core)=    3171.756639 Exc=    -129.950323 Ees=   -6534.249152 Eharris=   -3304.760874

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.491609   -184.340431   -189.832040
    E_B(band energy sum)=   -2.744580  E_B-nout*Vin=  187.087460

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575100
   smooth rhoves      9.671887   charge     3.579410
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.534005 -3.296829 -0.827416
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.522995      -184.608906      -190.131901
   rhoval*ves            -46.144898      -116.913084      -163.057981
   psnuc*ves              65.488671    -12969.862299    -12904.373628
   utot                    9.671887     -6543.387692     -6533.715805
   rho*exc                -2.534005      -127.354724      -129.888729
   rho*vxc                -3.296829      -168.634273      -171.931102
   valence chg             3.579410         7.420590        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.087460 Ekcore=      3171.756639 Ektot    =     3358.844100
 Exc=    -129.888729 Ees   =     -6533.715805 EKohnSham=    -3304.760434
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 6 RMS DQ= 1.46E-3  last it= 1.21E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.46D-03
   tj: 4.56404  -4.53114
 mixrho: add corrections to qcell smrho = -0.25579D-06 -0.32569D-08

 iors  : write rst restart file (binary mesh density)

   it  7  of 12    ehf=   -3304.760874   ehk=   -3304.760434
 From last iter    ehf=   -3304.760829   ehk=   -3304.760560
 diffe(q)= -0.000045 (0.001460)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.915500 ehk(eV)=-44963.909510 sev(eV)=-37.342205

--- BNDFP:  begin iteration 8 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569322
   smooth rhoves      9.513971   charge     3.552634
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.512992 -3.269456 -0.825339
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000478
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.471039      -184.951308      -190.422347
   rhoval*ves            -45.906901      -117.374348      -163.281249
   psnuc*ves              64.934844    -12970.154995    -12905.220151
   utot                    9.513971     -6543.764671     -6534.250700
   rho*exc                -2.512992      -127.438209      -129.951201
   rho*vxc                -3.269456      -168.744431      -172.013888
   valence chg             3.552634         7.447366        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0534

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012436;  11.000000 electrons
         Sum occ. bands:   -2.737978, incl. Bloechl correction: -0.009042
 bndfp:Generating TDOS: efermi= -0.012436  dos window emin emax=  -0.724564  2.927487


 mkrout:  Qtrue      sm,loc       local
   1   10.106420    2.687604    7.418816
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.737978 Vin*nin=    -190.422347 Ek=Eb-Vin*nin=     187.684369
 Ek(core)=    3171.756639 Exc=    -129.951201 Ees=   -6534.250700 Eharris=   -3304.760892

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.493426   -184.292480   -189.785906
    E_B(band energy sum)=   -2.737978  E_B-nout*Vin=  187.047928

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575401
   smooth rhoves      9.681379   charge     3.581184
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.535490 -3.298765 -0.827538
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.527168      -184.582274      -190.109442
   rhoval*ves            -46.158337      -116.881180      -163.039518
   psnuc*ves              65.521095    -12969.841484    -12904.320389
   utot                    9.681379     -6543.361332     -6533.679953
   rho*exc                -2.535490      -127.349506      -129.884997
   rho*vxc                -3.298765      -168.627389      -171.926154
   valence chg             3.581184         7.418816        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.047928 Ekcore=      3171.756639 Ektot    =     3358.804568
 Exc=    -129.884997 Ees   =     -6533.679953 EKohnSham=    -3304.760382
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 7 RMS DQ= 1.55E-3  last it= 1.46E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: Reducing nmix to  1: t_j exceeds tm: tj= 13.04540   0.80700
 AMIX: Reducing nmix to  0: t_j exceeds tm: tj= 15.86880
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.55D-03
 mixrho: add corrections to qcell smrho = -0.44015D-07 -0.56043D-09

 iors  : write rst restart file (binary mesh density)

   it  8  of 12    ehf=   -3304.760892   ehk=   -3304.760382
 From last iter    ehf=   -3304.760874   ehk=   -3304.760434
 diffe(q)= -0.000018 (0.001555)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.915751 ehk(eV)=-44963.908806 sev(eV)=-37.252375

--- BNDFP:  begin iteration 9 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.575401
   smooth rhoves      9.681379   charge     3.581184
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.535490 -3.298765 -0.827538
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000477
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.527168      -184.582274      -190.109442
   rhoval*ves            -46.158337      -116.881180      -163.039518
   psnuc*ves              65.521095    -12969.841484    -12904.320389
   utot                    9.681379     -6543.361332     -6533.679953
   rho*exc                -2.535490      -127.349506      -129.884997
   rho*vxc                -3.298765      -168.627389      -171.926154
   valence chg             3.581184         7.418816        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0539

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.034777;  11.000000 electrons
         Sum occ. bands:   -3.063027, incl. Bloechl correction: -0.009520
 bndfp:Generating TDOS: efermi= -0.034777  dos window emin emax=  -0.731137  2.905146


 mkrout:  Qtrue      sm,loc       local
   1   10.135745    2.613886    7.521859
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -3.063027 Vin*nin=    -190.109442 Ek=Eb-Vin*nin=     187.046415
 Ek(core)=    3171.756639 Exc=    -129.884997 Ees=   -6533.679953 Eharris=   -3304.761895

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.381167   -187.286421   -192.667589
    E_B(band energy sum)=   -3.063027  E_B-nout*Vin=  189.604562

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.558286
   smooth rhoves      9.141783   charge     3.478141
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.449201 -3.186286 -0.820466
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.284195      -186.337251      -191.621446
   rhoval*ves            -45.371656      -118.932242      -164.303898
   psnuc*ves              63.655221    -12971.353052    -12907.697831
   utot                    9.141783     -6545.142647     -6536.000864
   rho*exc                -2.449201      -127.667281      -130.116483
   rho*vxc                -3.186286      -169.046789      -172.233074
   valence chg             3.478141         7.521859        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      189.604562 Ekcore=      3171.756639 Ektot    =     3361.361202
 Exc=    -130.116483 Ees   =     -6536.000864 EKohnSham=    -3304.756145
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 8 RMS DQ= 5.99E-3  last it= 1.55E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 5.99D-03
   tj: 0.79470
 mixrho: add corrections to qcell smrho = -0.43174D-07 -0.54971D-09

 iors  : write rst restart file (binary mesh density)

   it  9  of 12    ehf=   -3304.761895   ehk=   -3304.756145
 From last iter    ehf=   -3304.760892   ehk=   -3304.760382
 diffe(q)= -0.001003 (0.005985)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.929396 ehk(eV)=-44963.851160 sev(eV)=-41.674926

--- BNDFP:  begin iteration 10 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571887
   smooth rhoves      9.569334   charge     3.560029
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.517715 -3.275594 -0.826098
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.476913      -184.944863      -190.421777
   rhoval*ves            -45.999357      -117.301784      -163.301141
   psnuc*ves              65.138025    -12970.151813    -12905.013789
   utot                    9.569334     -6543.726799     -6534.157465
   rho*exc                -2.517715      -127.414723      -129.932438
   rho*vxc                -3.275594      -168.713462      -171.989056
   valence chg             3.560029         7.439971        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0538

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020384;  11.000000 electrons
         Sum occ. bands:   -2.849249, incl. Bloechl correction: -0.009197
 bndfp:Generating TDOS: efermi= -0.020384  dos window emin emax=  -0.727388  2.919539


 mkrout:  Qtrue      sm,loc       local
   1   10.116160    2.664206    7.451954
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.849249 Vin*nin=    -190.421777 Ek=Eb-Vin*nin=     187.572528
 Ek(core)=    3171.756639 Exc=    -129.932438 Ees=   -6534.157465 Eharris=   -3304.760736

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.460979   -185.213400   -190.674380
    E_B(band energy sum)=   -2.849249  E_B-nout*Vin=  187.825131

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.569827
   smooth rhoves      9.505257   charge     3.548046
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.507745 -3.262598 -0.825264
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.449183      -185.110725      -190.559908
   rhoval*ves            -45.906731      -117.506230      -163.412962
   psnuc*ves              64.917246    -12970.275212    -12905.357965
   utot                    9.505257     -6543.890721     -6534.385463
   rho*exc                -2.507745      -127.449231      -129.956976
   rho*vxc                -3.262598      -168.758984      -172.021582
   valence chg             3.548046         7.451954        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.825131 Ekcore=      3171.756639 Ektot    =     3359.581771
 Exc=    -129.956976 Ees   =     -6534.385463 EKohnSham=    -3304.760669
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 8 RMS DQ= 6.21E-4  last it= 5.99E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 6.21D-04
   tj:-0.01447   0.25005
 mixrho: add corrections to qcell smrho = -0.49584D-07 -0.63133D-09

 iors  : write rst restart file (binary mesh density)

   it 10  of 12    ehf=   -3304.760736   ehk=   -3304.760669
 From last iter    ehf=   -3304.761895   ehk=   -3304.756145
 diffe(q)=  0.001159 (0.000621)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.913622 ehk(eV)=-44963.912709 sev(eV)=-38.766309

--- BNDFP:  begin iteration 11 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571388
   smooth rhoves      9.554467   charge     3.557343
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515525 -3.272740 -0.825903
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.471046      -184.960983      -190.432028
   rhoval*ves            -45.977561      -117.329257      -163.306818
   psnuc*ves              65.086496    -12970.151166    -12905.064670
   utot                    9.554467     -6543.740211     -6534.185744
   rho*exc                -2.515525      -127.421139      -129.936665
   rho*vxc                -3.272740      -168.721913      -171.994654
   valence chg             3.557343         7.442657        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0534

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018829;  11.000000 electrons
         Sum occ. bands:   -2.826994, incl. Bloechl correction: -0.009164
 bndfp:Generating TDOS: efermi= -0.018829  dos window emin emax=  -0.726910  2.921094


 mkrout:  Qtrue      sm,loc       local
   1   10.113805    2.669437    7.444368
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.826994 Vin*nin=    -190.432028 Ek=Eb-Vin*nin=     187.605034
 Ek(core)=    3171.756639 Exc=    -129.936665 Ees=   -6534.185744 Eharris=   -3304.760735

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.468248   -185.001362   -190.469610
    E_B(band energy sum)=   -2.826994  E_B-nout*Vin=  187.642616

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571128
   smooth rhoves      9.545786   charge     3.555632
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.514056 -3.270825 -0.825792
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.466730      -184.987683      -190.454414
   rhoval*ves            -45.965378      -117.360179      -163.325557
   psnuc*ves              65.056950    -12970.171219    -12905.114269
   utot                    9.545786     -6543.765699     -6534.219913
   rho*exc                -2.514056      -127.426020      -129.940076
   rho*vxc                -3.270825      -168.728352      -171.999177
   valence chg             3.555632         7.444368        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.642616 Ekcore=      3171.756639 Ektot    =     3359.399256
 Exc=    -129.940076 Ees   =     -6534.219913 EKohnSham=    -3304.760733
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 8 RMS DQ= 9.08E-5  last it= 6.21E-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 9.08D-05
   tj:-0.08146  -0.00810
 mixrho: add corrections to qcell smrho = -0.45336D-07 -0.57724D-09

 iors  : write rst restart file (binary mesh density)

   it 11  of 12    ehf=   -3304.760735   ehk=   -3304.760733
 From last iter    ehf=   -3304.760736   ehk=   -3304.760669
 diffe(q)=  0.000001 (0.000091)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.913605 ehk(eV)=-44963.913586 sev(eV)=-38.463515

--- BNDFP:  begin iteration 12 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571338
   smooth rhoves      9.552399   charge     3.556877
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515097 -3.272182 -0.825878
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469649      -184.966658      -190.436308
   rhoval*ves            -45.974883      -117.335565      -163.310448
   psnuc*ves              65.079682    -12970.153177    -12905.073496
   utot                    9.552399     -6543.744371     -6534.191972
   rho*exc                -2.515097      -127.422176      -129.937274
   rho*vxc                -3.272182      -168.723279      -171.995461
   valence chg             3.556877         7.443123        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0547

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018653;  11.000000 electrons
         Sum occ. bands:   -2.824438, incl. Bloechl correction: -0.009160
 bndfp:Generating TDOS: efermi= -0.018653  dos window emin emax=  -0.726872  2.921270


 mkrout:  Qtrue      sm,loc       local
   1   10.113517    2.670106    7.443410
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.824438 Vin*nin=    -190.436308 Ek=Eb-Vin*nin=     187.611870
 Ek(core)=    3171.756639 Exc=    -129.937274 Ees=   -6534.191972 Eharris=   -3304.760736

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.469239   -184.973444   -190.442683
    E_B(band energy sum)=   -2.824438  E_B-nout*Vin=  187.618245

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571291
   smooth rhoves      9.550887   charge     3.556590
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.514856 -3.271867 -0.825858
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.468965      -184.970990      -190.439955
   rhoval*ves            -45.972719      -117.340737      -163.313456
   psnuc*ves              65.074493    -12970.156537    -12905.082043
   utot                    9.550887     -6543.748637     -6534.197750
   rho*exc                -2.514856      -127.423015      -129.937871
   rho*vxc                -3.271867      -168.724386      -171.996253
   valence chg             3.556590         7.443410        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.618245 Ekcore=      3171.756639 Ektot    =     3359.374885
 Exc=    -129.937871 Ees   =     -6534.197750 EKohnSham=    -3304.760736
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 8 RMS DQ= 1.54E-5  last it= 9.08E-5
 AMIX: condition of normal eqns >100000. Reducing nmix to 7
 AMIX: condition of normal eqns >100000. Reducing nmix to 6
 AMIX: condition of normal eqns >100000. Reducing nmix to 5
 AMIX: condition of normal eqns >100000. Reducing nmix to 4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.54D-05
   tj:-0.20380
 mixrho: add corrections to qcell smrho = -0.44968D-07 -0.57256D-09

 iors  : write rst restart file (binary mesh density)

   it 12  of 12    ehf=   -3304.760736   ehk=   -3304.760736
 From last iter    ehf=   -3304.760735   ehk=   -3304.760733
 diffe(q)= -0.000002 (0.000015)    tol= 0.000010 (0.000010)   more=F
x ehf(eV)=-44963.913629 ehk(eV)=-44963.913626 sev(eV)=-38.428734
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 913e769c0a5a77a2254ce7ce7011c5bc7fb5168a
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Mon Feb 13 19:43:59 2023 +0900
INFO: linked at Tue Feb 14 12:44:43 JST 2023
===START LMF with   ===
 mpisize=           4
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
rval2: HAM_FORCES              defa n= 1 val= 0.00000000

 ... Species  1
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1

 ... Species  1
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1

 ... Species  1
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0

 ... Species  1
  bndfp (warning): no sigm file found ... LDA calculation only
pnuall: j isp pnu= 1 1 4.650000  4.340000  3.870000  4.110000  5.100000
pnzall: j isp  pz= 1 1 5.500000  5.500000  4.500000  0.000000  0.000000


mmm === MTO setting ===
mmm ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mmm rsmh1    1  2.50  2.50  1.00
mmm   eh1    1 -0.01 -0.01 -0.01
mmm rsmh2    1  1.30  0.00  1.00  1.30
 imx=           3           3           3
 imx=           3           3           3
mmm  eh2     1 -1.00 -1.00 -1.00 -0.01
 imx=           3           3           3
mmm pz       1  5.50  5.50  4.50
mmm lh       2  3  2
 imx=           4           4           4
 imx=           4           4           4

                Plat                                  Qlat
 imx=           4           4           4
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol=   78.538660
 imx=           3           3           3
 imx=           4           4           4

 LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
 nnnnnn         729         889
 nnnnnn         729         889
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
MKSYM: found  48  space group operations
SPLCLS: ibas iclass ispec label(ispec)
 SPLCLS     1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
 nnnnnn         729         889
     264 inequivalent tetrahedron=
 nnnnnn         729         889
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 m_qplistinit:start
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.39  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00
         site   1, species A       : augmentation lmax changed from 3 to 4
         site   1, species A       : inflate local density from nlm= 16 to 25

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571324
   smooth rhoves      9.551927   charge     3.556785
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515019 -3.272080 -0.825872
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469421      -184.967587      -190.437008
   rhoval*ves            -45.974215      -117.336774      -163.310989
   psnuc*ves              65.078069    -12970.153544    -12905.075476
   utot                    9.551927     -6543.745159     -6534.193233
   rho*exc                -2.515019      -127.422403      -129.937422
   rho*vxc                -3.272080      -168.723578      -171.995658
   valence chg             3.556785         7.443215        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1900

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018883;  11.000000 electrons
         Sum occ. bands:   -2.826235, incl. Bloechl correction: -0.009153
 bndfp:Generating TDOS: efermi= -0.018883  dos window emin emax=  -0.727728  2.921040


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128173    2.833459    7.294714
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.826235 Vin*nin=    -190.437008 Ek=Eb-Vin*nin=     187.610773
 Ek(core)=    3171.756639 Exc=    -129.937422 Ees=   -6534.193233 Eharris=   -3304.763242

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.930174   -184.355225   -190.285399
    E_B(band energy sum)=   -2.826235  E_B-nout*Vin=  187.459164

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581329
   smooth rhoves     10.109465   charge     3.705286
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657898 -3.458572 -0.832968
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010278      -184.303254      -190.313532
   rhoval*ves            -46.616840      -116.580931      -163.197771
   psnuc*ves              66.835769    -12971.741116    -12904.905347
   utot                   10.109465     -6544.161023     -6534.051559
   rho*exc                -2.657898      -127.269517      -129.927415
   rho*vxc                -3.458572      -168.523784      -171.982356
   valence chg             3.705286         7.294714        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.459164 Ekcore=      3171.756639 Ektot    =     3359.215803
 Exc=    -129.927415 Ees   =     -6534.051559 EKohnSham=    -3304.763170
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 0 RMS DQ= 4.32E-3
 AMIX: nmix=0 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 4.32D-03
 mixrho: add corrections to qcell smrho = -0.28235D-07 -0.35950D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.763242   ehk=   -3304.763170
i ehf(eV)=-44963.947720 ehk(eV)=-44963.946745 sev(eV)=-38.453188

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581329
   smooth rhoves     10.109465   charge     3.705286
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657898 -3.458572 -0.832968
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000042
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010278      -184.303254      -190.313532
   rhoval*ves            -46.616840      -116.580931      -163.197771
   psnuc*ves              66.835769    -12971.741116    -12904.905347
   utot                   10.109465     -6544.161023     -6534.051559
   rho*exc                -2.657898      -127.269517      -129.927415
   rho*vxc                -3.458572      -168.523784      -171.982356
   valence chg             3.705286         7.294714        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1866

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020802;  11.000000 electrons
         Sum occ. bands:   -2.854375, incl. Bloechl correction: -0.009202
 bndfp:Generating TDOS: efermi= -0.020802  dos window emin emax=  -0.727847  2.919121


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.131649    2.824829    7.306820
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.854375 Vin*nin=    -190.313532 Ek=Eb-Vin*nin=     187.459157
 Ek(core)=    3171.756639 Exc=    -129.927415 Ees=   -6534.051559 Eharris=   -3304.763177

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.992931   -184.685653   -190.678585
    E_B(band energy sum)=   -2.854375  E_B-nout*Vin=  187.824209

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579288
   smooth rhoves     10.043955   charge     3.693180
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.647653 -3.445216 -0.832150
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.980099      -184.560698      -190.540798
   rhoval*ves            -46.528529      -116.862883      -163.391412
   psnuc*ves              66.616438    -12971.995011    -12905.378572
   utot                   10.043955     -6544.428947     -6534.384992
   rho*exc                -2.647653      -127.311275      -129.958928
   rho*vxc                -3.445216      -168.578929      -172.024145
   valence chg             3.693180         7.306820        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.824209 Ekcore=      3171.756639 Ektot    =     3359.580849
 Exc=    -129.958928 Ees   =     -6534.384992 EKohnSham=    -3304.763072
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 1 RMS DQ= 8.01E-4  last it= 4.32E-3
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 8.01D-04
   tj: 0.08296
 mixrho: add corrections to qcell smrho = -0.26733D-07 -0.34038D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3304.763177   ehk=   -3304.763072
 From last iter    ehf=   -3304.763242   ehk=   -3304.763170
 diffe(q)=  0.000065 (0.000801)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946839 ehk(eV)=-44963.945406 sev(eV)=-38.836061

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579458
   smooth rhoves     10.049381   charge     3.694185
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.648503 -3.446324 -0.832218
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.982600      -184.539360      -190.521960
   rhoval*ves            -46.535871      -116.839494      -163.375366
   psnuc*ves              66.634633    -12971.973948    -12905.339314
   utot                   10.049381     -6544.406721     -6534.357340
   rho*exc                -2.648503      -127.307811      -129.956313
   rho*vxc                -3.446324      -168.574354      -172.020678
   valence chg             3.694185         7.305815        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1792

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.011989;  11.000000 electrons
         Sum occ. bands:   -2.728375, incl. Bloechl correction: -0.009007
 bndfp:Generating TDOS: efermi= -0.011989  dos window emin emax=  -0.725680  2.927933


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.118671    2.849894    7.268776
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.728375 Vin*nin=    -190.521960 Ek=Eb-Vin*nin=     187.793585
 Ek(core)=    3171.756639 Exc=    -129.956313 Ees=   -6534.357340 Eharris=   -3304.763429

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.025972   -183.426093   -189.452065
    E_B(band energy sum)=   -2.728375  E_B-nout*Vin=  186.723690

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.586225
   smooth rhoves     10.259462   charge     3.731224
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.679136 -3.486250 -0.834837
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.068332      -183.815807      -189.884140
   rhoval*ves            -46.821311      -116.019404      -162.840715
   psnuc*ves              67.340236    -12971.265450    -12903.925214
   utot                   10.259462     -6543.642427     -6533.382964
   rho*exc                -2.679136      -127.180654      -129.859790
   rho*vxc                -3.486250      -168.406450      -171.892700
   valence chg             3.731224         7.268776        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      186.723690 Ekcore=      3171.756639 Ektot    =     3358.480329
 Exc=    -129.859790 Ees   =     -6533.382964 EKohnSham=    -3304.762425
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 2 RMS DQ= 2.36E-3  last it= 8.01E-4
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 2.36D-03
   tj: 0.74720  -0.00311
 mixrho: add corrections to qcell smrho = -0.27292D-07 -0.34750D-09

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.763429   ehk=   -3304.762425
 From last iter    ehf=   -3304.763177   ehk=   -3304.763072
 diffe(q)= -0.000251 (0.002357)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.950256 ehk(eV)=-44963.936608 sev(eV)=-37.121727

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581057
   smooth rhoves     10.098682   charge     3.702879
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655668 -3.455663 -0.832837
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.002540      -184.371330      -190.373869
   rhoval*ves            -46.603620      -116.647902      -163.251522
   psnuc*ves              66.800984    -12971.809097    -12905.008113
   utot                   10.098682     -6544.228499     -6534.129817
   rho*exc                -2.655668      -127.277971      -129.933639
   rho*vxc                -3.455663      -168.534952      -171.990614
   valence chg             3.702879         7.297121        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1831

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.019147;  11.000000 electrons
         Sum occ. bands:   -2.830219, incl. Bloechl correction: -0.009164
 bndfp:Generating TDOS: efermi= -0.019147  dos window emin emax=  -0.727515  2.920776


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.129011    2.830104    7.298907
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.830219 Vin*nin=    -190.373869 Ek=Eb-Vin*nin=     187.543651
 Ek(core)=    3171.756639 Exc=    -129.933639 Ees=   -6534.129817 Eharris=   -3304.763166

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.000745   -184.420037   -190.420782
    E_B(band energy sum)=   -2.830219  E_B-nout*Vin=  187.590564

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580714
   smooth rhoves     10.088310   charge     3.701093
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.654215 -3.453769 -0.832707
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.998622      -184.401380      -190.400002
   rhoval*ves            -46.589274      -116.683572      -163.272846
   psnuc*ves              66.765894    -12971.837529    -12905.071635
   utot                   10.088310     -6544.260550     -6534.172240
   rho*exc                -2.654215      -127.283912      -129.938126
   rho*vxc                -3.453769      -168.542794      -171.996563
   valence chg             3.701093         7.298907        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.590564 Ekcore=      3171.756639 Ektot    =     3359.347203
 Exc=    -129.938126 Ees   =     -6534.172240 EKohnSham=    -3304.763163
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 3 RMS DQ= 1.06E-4  last it= 2.36E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 1.06D-04
   tj: 0.08940   0.16482
 mixrho: add corrections to qcell smrho = -0.24059D-07 -0.30633D-09

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.763166   ehk=   -3304.763163
 From last iter    ehf=   -3304.763429   ehk=   -3304.762425
 diffe(q)=  0.000263 (0.000106)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946681 ehk(eV)=-44963.946648 sev(eV)=-38.507388

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580972
   smooth rhoves     10.096230   charge     3.702483
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655358 -3.455259 -0.832807
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001787      -184.375434      -190.377221
   rhoval*ves            -46.600150      -116.653745      -163.253896
   psnuc*ves              66.792610    -12971.812338    -12905.019728
   utot                   10.096230     -6544.233042     -6534.136812
   rho*exc                -2.655358      -127.279187      -129.934545
   rho*vxc                -3.455259      -168.536557      -171.991816
   valence chg             3.702483         7.297517        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1776

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018829;  11.000000 electrons
         Sum occ. bands:   -2.825670, incl. Bloechl correction: -0.009157
 bndfp:Generating TDOS: efermi= -0.018829  dos window emin emax=  -0.727421  2.921093


 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128558    2.830941    7.297617
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.825670 Vin*nin=    -190.377221 Ek=Eb-Vin*nin=     187.551551
 Ek(core)=    3171.756639 Exc=    -129.934545 Ees=   -6534.136812 Eharris=   -3304.763167

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.001692   -184.378524   -190.380217
    E_B(band energy sum)=   -2.825670  E_B-nout*Vin=  187.554547

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580952
   smooth rhoves     10.095643   charge     3.702383
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655277 -3.455153 -0.832799
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001572      -184.377386      -190.378958
   rhoval*ves            -46.599339      -116.655998      -163.255337
   psnuc*ves              66.790625    -12971.814350    -12905.023724
   utot                   10.095643     -6544.235174     -6534.139530
   rho*exc                -2.655277      -127.279546      -129.934822
   rho*vxc                -3.455153      -168.537030      -171.992183
   valence chg             3.702383         7.297617        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.554547 Ekcore=      3171.756639 Ektot    =     3359.311186
 Exc=    -129.934822 Ees   =     -6534.139530 EKohnSham=    -3304.763167
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 8 iter from file mixm ; read 4 RMS DQ= 6.58E-6  last it= 1.06E-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 6.58D-06
   tj:-0.06603
 mixrho: add corrections to qcell smrho = -0.26040D-07 -0.33156D-09

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.763167   ehk=   -3304.763167
 From last iter    ehf=   -3304.763166   ehk=   -3304.763163
 diffe(q)= -0.000001 (0.000007)    tol= 0.000010 (0.000010)   more=F
c ehf(eV)=-44963.946694 ehk(eV)=-44963.946691 sev(eV)=-38.445497
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.1) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit 913e769c0a5a77a2254ce7ce7011c5bc7fb5168a
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Mon Feb 13 19:43:59 2023 +0900
 m_lmfinit:program LMF
 m_lmfinit:program LMF
 m_lmfinit:program LMF
INFO: linked at Tue Feb 14 12:44:43 JST 2023
===START LMF with   ===
 mpisize=           4
 m_lmfinit:program LMF
 cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7 --band:fn=syml<ctrl.cu >ctrlp.cu
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1

rval2: HAM_XCFUN               defa n= 1 val= 2.00000000

 ... Species  1
 ... Species  1
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000

 ... Species  1
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_NEVMX                defa n= 1 val= 0.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing parameters: A/B nmix wt: 0 -1 1.000000  1.000000  0.000000 beta elin wc killj=  1.000000 -1.000000 -1
 ===> for --jobgw, pwmode is switched to be  0

 ... Species  1
  bndfp (warning): no sigm file found ... LDA calculation only
pnuall: j isp pnu= 1 1 4.650000  4.340000  3.870000  4.110000  5.100000
pnzall: j isp  pz= 1 1 5.500000  5.500000  4.500000  0.000000  0.000000


mmm === MTO setting ===
mmm ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mmm rsmh1    1  2.50  2.50  1.00
mmm   eh1    1 -0.01 -0.01 -0.01
 imx=           3           3           3
 imx=           3           3           3
mmm rsmh2    1  1.30  0.00  1.00  1.30
mmm  eh2     1 -1.00 -1.00 -1.00 -0.01
mmm pz       1  5.50  5.50  4.50
mmm lh       2  3  2
 imx=           3           3           3
 imx=           4           4           4
 imx=           4           4           4

                Plat                                  Qlat
 imx=           4           4           4
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol=   78.538660
 imx=           3           3           3
 imx=           4           4           4

 LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331
SGROUP:  1 symmetry operations from 0
 SYMLAT: Bravais system is cubic       with 48 symmetry operations.
 SYMCRY: crystal invariant under  48 symmetry operations for tol=  0.000100
 ig  group op
   1  i*i
   2  i
   3  r3(1,1,-1)
   4  i*r3(1,1,-1)
   5  r3(-1,-1,1)
   6  i*r3(-1,-1,1)
   7  r3d
   8  i*r3d
   9  r3(-1,-1,-1)
  10  i*r3(-1,-1,-1)
  11  r2x
  12  mx
  13  r4x
  14  i*r4x
  15  r4(-1,0,0)
  16  i*r4(-1,0,0)
  17  r3(1,-1,-1)
  18  i*r3(1,-1,-1)
  19  r3(-1,1,1)
  20  i*r3(-1,1,1)
  21  r2(1,1,0)
  22  m(1,1,0)
  23  r2(1,0,-1)
  24  m(1,0,-1)
  25  r2y
  26  my
  27  r4y
  28  i*r4y
  29  r4(0,-1,0)
  30  i*r4(0,-1,0)
  31  r2(0,1,-1)
  32  m(0,1,-1)
  33  r2z
  34  mz
  35  r4(0,0,-1)
  36  i*r4(0,0,-1)
  37  r4z
  38  i*r4z
  39  r3(-1,1,-1)
  40  i*r3(-1,1,-1)
  41  r3(1,-1,1)
  42  i*r3(1,-1,1)
  43  r2(1,0,1)
  44  m(1,0,1)
  45  r2(1,-1,0)
  46  m(1,-1,0)
  47  r2(0,1,1)
  48  m(0,1,1)
 nnnnnn         729         889
 nnnnnn         729         889
GROUPG: the following are sufficient to generate the space group:
 Generators:trans(cart)  = i*r3(1,1,-1) r4x
 Generators::trans(frac) = i*r3(1,1,-1) r4x
MKSYM: found  48  space group operations
SPLCLS: ibas iclass ispec label(ispec)
 SPLCLS     1    1    1     A
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
 nnnnnn         729         889
 nnnnnn         729         889
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 m_qplistinit:start
  --- Readin syml file --- 
   41   0.5000   0.5000   0.5000    0.0000   0.0000   0.0000 L Gamma
   41   0.0000   0.0000   0.0000    1.0000   0.0000   0.0000 Gamma X
   21   1.0000   0.0000   0.0000    1.0000   0.5000   0.0000 X W
   41   1.0000   0.5000   0.0000    0.0000   0.0000   0.0000 W Gamma
nsyml nkp=    4  144
 -------- qplist --------           4
    1   0.500   0.500   0.500  <-- isyml= 001
    2   0.487   0.487   0.487 
    3   0.475   0.475   0.475 
    4   0.463   0.463   0.463 
    5   0.450   0.450   0.450 
    6   0.438   0.438   0.438 
    7   0.425   0.425   0.425 
    8   0.412   0.412   0.412 
    9   0.400   0.400   0.400 
   10   0.388   0.388   0.388 
   11   0.375   0.375   0.375 
   12   0.362   0.362   0.362 
   13   0.350   0.350   0.350 
   14   0.338   0.338   0.338 
   15   0.325   0.325   0.325 
   16   0.312   0.312   0.312 
   17   0.300   0.300   0.300 
   18   0.287   0.287   0.287 
   19   0.275   0.275   0.275 
   20   0.263   0.263   0.263 
   21   0.250   0.250   0.250 
   22   0.237   0.237   0.237 
   23   0.225   0.225   0.225 
   24   0.213   0.213   0.213 
   25   0.200   0.200   0.200 
   26   0.188   0.188   0.188 
   27   0.175   0.175   0.175 
   28   0.162   0.162   0.162 
   29   0.150   0.150   0.150 
   30   0.138   0.138   0.138 
   31   0.125   0.125   0.125 
   32   0.112   0.112   0.112 
   33   0.100   0.100   0.100 
   34   0.088   0.088   0.088 
   35   0.075   0.075   0.075 
   36   0.062   0.062   0.062 
   37   0.050   0.050   0.050 
   38   0.037   0.037   0.037 
   39   0.025   0.025   0.025 
   40   0.013   0.013   0.013 
   41   0.000   0.000   0.000 
   42   0.000   0.000   0.000  <-- isyml= 002
   43   0.025   0.000   0.000 
   44   0.050   0.000   0.000 
   45   0.075   0.000   0.000 
   46   0.100   0.000   0.000 
   47   0.125   0.000   0.000 
   48   0.150   0.000   0.000 
   49   0.175   0.000   0.000 
   50   0.200   0.000   0.000 
   51   0.225   0.000   0.000 
   52   0.250   0.000   0.000 
   53   0.275   0.000   0.000 
   54   0.300   0.000   0.000 
   55   0.325   0.000   0.000 
   56   0.350   0.000   0.000 
   57   0.375   0.000   0.000 
   58   0.400   0.000   0.000 
   59   0.425   0.000   0.000 
   60   0.450   0.000   0.000 
   61   0.475   0.000   0.000 
   62   0.500   0.000   0.000 
   63   0.525   0.000   0.000 
   64   0.550   0.000   0.000 
   65   0.575   0.000   0.000 
   66   0.600   0.000   0.000 
   67   0.625   0.000   0.000 
   68   0.650   0.000   0.000 
   69   0.675   0.000   0.000 
   70   0.700   0.000   0.000 
   71   0.725   0.000   0.000 
   72   0.750   0.000   0.000 
   73   0.775   0.000   0.000 
   74   0.800   0.000   0.000 
   75   0.825   0.000   0.000 
   76   0.850   0.000   0.000 
   77   0.875   0.000   0.000 
   78   0.900   0.000   0.000 
   79   0.925   0.000   0.000 
   80   0.950   0.000   0.000 
   81   0.975   0.000   0.000 
   82   1.000   0.000   0.000 
   83   1.000   0.000   0.000  <-- isyml= 003
   84   1.000   0.025   0.000 
   85   1.000   0.050   0.000 
   86   1.000   0.075   0.000 
   87   1.000   0.100   0.000 
   88   1.000   0.125   0.000 
   89   1.000   0.150   0.000 
   90   1.000   0.175   0.000 
   91   1.000   0.200   0.000 
   92   1.000   0.225   0.000 
   93   1.000   0.250   0.000 
   94   1.000   0.275   0.000 
   95   1.000   0.300   0.000 
   96   1.000   0.325   0.000 
   97   1.000   0.350   0.000 
   98   1.000   0.375   0.000 
   99   1.000   0.400   0.000 
  100   1.000   0.425   0.000 
  101   1.000   0.450   0.000 
  102   1.000   0.475   0.000 
  103   1.000   0.500   0.000 
  104   1.000   0.500   0.000  <-- isyml= 004
  105   0.975   0.487   0.000 
  106   0.950   0.475   0.000 
  107   0.925   0.463   0.000 
  108   0.900   0.450   0.000 
  109   0.875   0.438   0.000 
  110   0.850   0.425   0.000 
  111   0.825   0.412   0.000 
  112   0.800   0.400   0.000 
  113   0.775   0.388   0.000 
  114   0.750   0.375   0.000 
  115   0.725   0.362   0.000 
  116   0.700   0.350   0.000 
  117   0.675   0.338   0.000 
  118   0.650   0.325   0.000 
  119   0.625   0.312   0.000 
  120   0.600   0.300   0.000 
  121   0.575   0.287   0.000 
  122   0.550   0.275   0.000 
  123   0.525   0.263   0.000 
  124   0.500   0.250   0.000 
  125   0.475   0.237   0.000 
  126   0.450   0.225   0.000 
  127   0.425   0.213   0.000 
  128   0.400   0.200   0.000 
  129   0.375   0.188   0.000 
  130   0.350   0.175   0.000 
  131   0.325   0.162   0.000 
  132   0.300   0.150   0.000 
  133   0.275   0.138   0.000 
  134   0.250   0.125   0.000 
  135   0.225   0.112   0.000 
  136   0.200   0.100   0.000 
  137   0.175   0.088   0.000 
  138   0.150   0.075   0.000 
  139   0.125   0.062   0.000 
  140   0.100   0.050   0.000 
  141   0.075   0.037   0.000 
  142   0.050   0.025   0.000 
  143   0.025   0.013   0.000 
  144   0.000   0.000   0.000 
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.38  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 1
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580968
   smooth rhoves     10.096128   charge     3.702468
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655347 -3.455244 -0.832805
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001767      -184.375801      -190.377568
   rhoval*ves            -46.600003      -116.654177      -163.254180
   psnuc*ves              66.792258    -12971.812819    -12905.020561
   utot                   10.096128     -6544.233498     -6534.137371
   rho*exc                -2.655347      -127.279257      -129.934604
   rho*vxc                -3.455244      -168.536649      -171.991894
   valence chg             3.702468         7.297532        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of   144 k=  0.5000  0.5000  0.5000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of   144 k=  0.4875  0.4875  0.4875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of   144 k=  0.4750  0.4750  0.4750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of   144 k=  0.4625  0.4625  0.4625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of   144 k=  0.4500  0.4500  0.4500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of   144 k=  0.4375  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of   144 k=  0.4250  0.4250  0.4250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of   144 k=  0.4125  0.4125  0.4125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of   144 k=  0.4000  0.4000  0.4000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of   144 k=  0.3875  0.3875  0.3875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of   144 k=  0.3750  0.3750  0.3750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of   144 k=  0.3625  0.3625  0.3625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of   144 k=  0.3500  0.3500  0.3500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of   144 k=  0.3375  0.3375  0.3375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of   144 k=  0.3250  0.3250  0.3250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    16 of   144 k=  0.3125  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    17 of   144 k=  0.3000  0.3000  0.3000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    18 of   144 k=  0.2875  0.2875  0.2875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    19 of   144 k=  0.2750  0.2750  0.2750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    20 of   144 k=  0.2625  0.2625  0.2625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    21 of   144 k=  0.2500  0.2500  0.2500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    22 of   144 k=  0.2375  0.2375  0.2375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    23 of   144 k=  0.2250  0.2250  0.2250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    24 of   144 k=  0.2125  0.2125  0.2125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    25 of   144 k=  0.2000  0.2000  0.2000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    26 of   144 k=  0.1875  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    27 of   144 k=  0.1750  0.1750  0.1750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    28 of   144 k=  0.1625  0.1625  0.1625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    29 of   144 k=  0.1500  0.1500  0.1500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    30 of   144 k=  0.1375  0.1375  0.1375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    31 of   144 k=  0.1250  0.1250  0.1250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    32 of   144 k=  0.1125  0.1125  0.1125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    33 of   144 k=  0.1000  0.1000  0.1000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    34 of   144 k=  0.0875  0.0875  0.0875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    35 of   144 k=  0.0750  0.0750  0.0750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    36 of   144 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.4312
  Writing bands to bands file for gnuplot ...
 bndfp: kpt    1 of  144 k jsp=  0.50000  0.50000  0.50000 1 nev=   31
 bndfp: kpt    2 of  144 k jsp=  0.48750  0.48750  0.48750 1 nev=   31
 bndfp: kpt    3 of  144 k jsp=  0.47500  0.47500  0.47500 1 nev=   31
 bndfp: kpt    4 of  144 k jsp=  0.46250  0.46250  0.46250 1 nev=   31
 bndfp: kpt    5 of  144 k jsp=  0.45000  0.45000  0.45000 1 nev=   31
 bndfp: kpt    6 of  144 k jsp=  0.43750  0.43750  0.43750 1 nev=   31
 bndfp: kpt    7 of  144 k jsp=  0.42500  0.42500  0.42500 1 nev=   31
 bndfp: kpt    8 of  144 k jsp=  0.41250  0.41250  0.41250 1 nev=   31
 bndfp: kpt    9 of  144 k jsp=  0.40000  0.40000  0.40000 1 nev=   31
 bndfp: kpt   10 of  144 k jsp=  0.38750  0.38750  0.38750 1 nev=   31
 bndfp: kpt   11 of  144 k jsp=  0.37500  0.37500  0.37500 1 nev=   31
 bndfp: kpt   12 of  144 k jsp=  0.36250  0.36250  0.36250 1 nev=   31
 bndfp: kpt   13 of  144 k jsp=  0.35000  0.35000  0.35000 1 nev=   31
 bndfp: kpt   14 of  144 k jsp=  0.33750  0.33750  0.33750 1 nev=   31
 bndfp: kpt   15 of  144 k jsp=  0.32500  0.32500  0.32500 1 nev=   31
 bndfp: kpt   16 of  144 k jsp=  0.31250  0.31250  0.31250 1 nev=   31
 bndfp: kpt   17 of  144 k jsp=  0.30000  0.30000  0.30000 1 nev=   31
 bndfp: kpt   18 of  144 k jsp=  0.28750  0.28750  0.28750 1 nev=   31
 bndfp: kpt   19 of  144 k jsp=  0.27500  0.27500  0.27500 1 nev=   31
 bndfp: kpt   20 of  144 k jsp=  0.26250  0.26250  0.26250 1 nev=   31
 bndfp: kpt   21 of  144 k jsp=  0.25000  0.25000  0.25000 1 nev=   31
 bndfp: kpt   22 of  144 k jsp=  0.23750  0.23750  0.23750 1 nev=   31
 bndfp: kpt   23 of  144 k jsp=  0.22500  0.22500  0.22500 1 nev=   31
 bndfp: kpt   24 of  144 k jsp=  0.21250  0.21250  0.21250 1 nev=   31
 bndfp: kpt   25 of  144 k jsp=  0.20000  0.20000  0.20000 1 nev=   31
 bndfp: kpt   26 of  144 k jsp=  0.18750  0.18750  0.18750 1 nev=   31
 bndfp: kpt   27 of  144 k jsp=  0.17500  0.17500  0.17500 1 nev=   31
 bndfp: kpt   28 of  144 k jsp=  0.16250  0.16250  0.16250 1 nev=   31
 bndfp: kpt   29 of  144 k jsp=  0.15000  0.15000  0.15000 1 nev=   31
 bndfp: kpt   30 of  144 k jsp=  0.13750  0.13750  0.13750 1 nev=   31
 bndfp: kpt   31 of  144 k jsp=  0.12500  0.12500  0.12500 1 nev=   31
 bndfp: kpt   32 of  144 k jsp=  0.11250  0.11250  0.11250 1 nev=   31
 bndfp: kpt   33 of  144 k jsp=  0.10000  0.10000  0.10000 1 nev=   31
 bndfp: kpt   34 of  144 k jsp=  0.08750  0.08750  0.08750 1 nev=   31
 bndfp: kpt   35 of  144 k jsp=  0.07500  0.07500  0.07500 1 nev=   31
 bndfp: kpt   36 of  144 k jsp=  0.06250  0.06250  0.06250 1 nev=   31
 bndfp: kpt   37 of  144 k jsp=  0.05000  0.05000  0.05000 1 nev=   31
 bndfp: kpt   38 of  144 k jsp=  0.03750  0.03750  0.03750 1 nev=   31
 bndfp: kpt   39 of  144 k jsp=  0.02500  0.02500  0.02500 1 nev=   31
 bndfp: kpt   40 of  144 k jsp=  0.01250  0.01250  0.01250 1 nev=   31
 bndfp: kpt   41 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   42 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   43 of  144 k jsp=  0.02500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   44 of  144 k jsp=  0.05000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   45 of  144 k jsp=  0.07500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   46 of  144 k jsp=  0.10000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   47 of  144 k jsp=  0.12500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   48 of  144 k jsp=  0.15000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   49 of  144 k jsp=  0.17500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   50 of  144 k jsp=  0.20000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   51 of  144 k jsp=  0.22500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   52 of  144 k jsp=  0.25000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   53 of  144 k jsp=  0.27500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   54 of  144 k jsp=  0.30000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   55 of  144 k jsp=  0.32500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   56 of  144 k jsp=  0.35000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   57 of  144 k jsp=  0.37500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   58 of  144 k jsp=  0.40000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   59 of  144 k jsp=  0.42500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   60 of  144 k jsp=  0.45000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   61 of  144 k jsp=  0.47500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   62 of  144 k jsp=  0.50000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   63 of  144 k jsp=  0.52500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   64 of  144 k jsp=  0.55000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   65 of  144 k jsp=  0.57500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   66 of  144 k jsp=  0.60000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   67 of  144 k jsp=  0.62500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   68 of  144 k jsp=  0.65000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   69 of  144 k jsp=  0.67500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   70 of  144 k jsp=  0.70000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   71 of  144 k jsp=  0.72500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   72 of  144 k jsp=  0.75000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   73 of  144 k jsp=  0.77500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   74 of  144 k jsp=  0.80000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   75 of  144 k jsp=  0.82500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   76 of  144 k jsp=  0.85000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   77 of  144 k jsp=  0.87500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   78 of  144 k jsp=  0.90000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   79 of  144 k jsp=  0.92500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   80 of  144 k jsp=  0.95000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   81 of  144 k jsp=  0.97500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   82 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   83 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   84 of  144 k jsp=  1.00000  0.02500  0.00000 1 nev=   31
 bndfp: kpt   85 of  144 k jsp=  1.00000  0.05000  0.00000 1 nev=   31
 bndfp: kpt   86 of  144 k jsp=  1.00000  0.07500  0.00000 1 nev=   31
 bndfp: kpt   87 of  144 k jsp=  1.00000  0.10000  0.00000 1 nev=   31
 bndfp: kpt   88 of  144 k jsp=  1.00000  0.12500  0.00000 1 nev=   31
 bndfp: kpt   89 of  144 k jsp=  1.00000  0.15000  0.00000 1 nev=   31
 bndfp: kpt   90 of  144 k jsp=  1.00000  0.17500  0.00000 1 nev=   31
 bndfp: kpt   91 of  144 k jsp=  1.00000  0.20000  0.00000 1 nev=   31
 bndfp: kpt   92 of  144 k jsp=  1.00000  0.22500  0.00000 1 nev=   31
 bndfp: kpt   93 of  144 k jsp=  1.00000  0.25000  0.00000 1 nev=   31
 bndfp: kpt   94 of  144 k jsp=  1.00000  0.27500  0.00000 1 nev=   31
 bndfp: kpt   95 of  144 k jsp=  1.00000  0.30000  0.00000 1 nev=   31
 bndfp: kpt   96 of  144 k jsp=  1.00000  0.32500  0.00000 1 nev=   31
 bndfp: kpt   97 of  144 k jsp=  1.00000  0.35000  0.00000 1 nev=   31
 bndfp: kpt   98 of  144 k jsp=  1.00000  0.37500  0.00000 1 nev=   31
 bndfp: kpt   99 of  144 k jsp=  1.00000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  100 of  144 k jsp=  1.00000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  101 of  144 k jsp=  1.00000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  102 of  144 k jsp=  1.00000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  103 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  104 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  105 of  144 k jsp=  0.97500  0.48750  0.00000 1 nev=   31
 bndfp: kpt  106 of  144 k jsp=  0.95000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  107 of  144 k jsp=  0.92500  0.46250  0.00000 1 nev=   31
 bndfp: kpt  108 of  144 k jsp=  0.90000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  109 of  144 k jsp=  0.87500  0.43750  0.00000 1 nev=   31
 bndfp: kpt  110 of  144 k jsp=  0.85000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  111 of  144 k jsp=  0.82500  0.41250  0.00000 1 nev=   31
 bndfp: kpt  112 of  144 k jsp=  0.80000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  113 of  144 k jsp=  0.77500  0.38750  0.00000 1 nev=   31
 bndfp: kpt  114 of  144 k jsp=  0.75000  0.37500  0.00000 1 nev=   31
 bndfp: kpt  115 of  144 k jsp=  0.72500  0.36250  0.00000 1 nev=   31
 bndfp: kpt  116 of  144 k jsp=  0.70000  0.35000  0.00000 1 nev=   31
 bndfp: kpt  117 of  144 k jsp=  0.67500  0.33750  0.00000 1 nev=   31
 bndfp: kpt  118 of  144 k jsp=  0.65000  0.32500  0.00000 1 nev=   31
 bndfp: kpt  119 of  144 k jsp=  0.62500  0.31250  0.00000 1 nev=   31
 bndfp: kpt  120 of  144 k jsp=  0.60000  0.30000  0.00000 1 nev=   31
 bndfp: kpt  121 of  144 k jsp=  0.57500  0.28750  0.00000 1 nev=   31
 bndfp: kpt  122 of  144 k jsp=  0.55000  0.27500  0.00000 1 nev=   31
 bndfp: kpt  123 of  144 k jsp=  0.52500  0.26250  0.00000 1 nev=   31
 bndfp: kpt  124 of  144 k jsp=  0.50000  0.25000  0.00000 1 nev=   31
 bndfp: kpt  125 of  144 k jsp=  0.47500  0.23750  0.00000 1 nev=   31
 bndfp: kpt  126 of  144 k jsp=  0.45000  0.22500  0.00000 1 nev=   31
 bndfp: kpt  127 of  144 k jsp=  0.42500  0.21250  0.00000 1 nev=   31
 bndfp: kpt  128 of  144 k jsp=  0.40000  0.20000  0.00000 1 nev=   31
 bndfp: kpt  129 of  144 k jsp=  0.37500  0.18750  0.00000 1 nev=   31
 bndfp: kpt  130 of  144 k jsp=  0.35000  0.17500  0.00000 1 nev=   31
 bndfp: kpt  131 of  144 k jsp=  0.32500  0.16250  0.00000 1 nev=   31
 bndfp: kpt  132 of  144 k jsp=  0.30000  0.15000  0.00000 1 nev=   31
 bndfp: kpt  133 of  144 k jsp=  0.27500  0.13750  0.00000 1 nev=   31
 bndfp: kpt  134 of  144 k jsp=  0.25000  0.12500  0.00000 1 nev=   31
 bndfp: kpt  135 of  144 k jsp=  0.22500  0.11250  0.00000 1 nev=   31
 bndfp: kpt  136 of  144 k jsp=  0.20000  0.10000  0.00000 1 nev=   31
 bndfp: kpt  137 of  144 k jsp=  0.17500  0.08750  0.00000 1 nev=   31
 bndfp: kpt  138 of  144 k jsp=  0.15000  0.07500  0.00000 1 nev=   31
 bndfp: kpt  139 of  144 k jsp=  0.12500  0.06250  0.00000 1 nev=   31
 bndfp: kpt  140 of  144 k jsp=  0.10000  0.05000  0.00000 1 nev=   31
 bndfp: kpt  141 of  144 k jsp=  0.07500  0.03750  0.00000 1 nev=   31
 bndfp: kpt  142 of  144 k jsp=  0.05000  0.02500  0.00000 1 nev=   31
 bndfp: kpt  143 of  144 k jsp=  0.02500  0.01250  0.00000 1 nev=   31
 bndfp: kpt  144 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
Exit 0 procid= 0 plot band mode done
