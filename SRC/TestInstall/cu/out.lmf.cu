#include "hip/hip_runtime.h"
===START LMFA   =====================================
 mpisize=           1

mmm === MTO setting ===
mmm ispec lmxb lpzex nkapii nkaphh=    1    3    0    1    1
mmm rsmh1    1  2.50  2.50  1.00  0.00
mmm   eh1    1 -0.01 -0.01 -0.01 -0.01
mmm rsmh2    1  0.00  0.00  0.00  0.00
mmm  eh2     1  0.00  0.00  0.00  0.00
mmm pz       1  5.50  5.50  4.50  0.00
mmm lh       1  2  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 ADDBAS: basis is already complete --- no sites added
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 ig  group op
   1  i*i                                
   2  i                                  
   3  r3(1,1,-1)                         
   4  i*r3(1,1,-1)                       
   5  r3(-1,-1,1)                        
   6  i*r3(-1,-1,1)                      
   7  r3d                                
   8  i*r3d                              
   9  r3(-1,-1,-1)                       
  10  i*r3(-1,-1,-1)                     
  11  r2x                                
  12  mx                                 
  13  r4x                                
  14  i*r4x                              
  15  r4(-1,0,0)                         
  16  i*r4(-1,0,0)                       
  17  r3(1,-1,-1)                        
  18  i*r3(1,-1,-1)                      
  19  r3(-1,1,1)                         
  20  i*r3(-1,1,1)                       
  21  r2(1,1,-0)                         
  22  m(1,1,-0)                          
  23  r2(1,-0,-1)                        
  24  m(1,-0,-1)                         
  25  r2y                                
  26  my                                 
  27  r4y                                
  28  i*r4y                              
  29  r4(0,-1,0)                         
  30  i*r4(0,-1,0)                       
  31  r2(0,1,-1)                         
  32  m(0,1,-1)                          
  33  r2z                                
  34  mz                                 
  35  r4(0,0,-1)                         
  36  i*r4(0,0,-1)                       
  37  r4z                                
  38  i*r4z                              
  39  r3(-1,1,-1)                        
  40  i*r3(-1,1,-1)                      
  41  r3(1,-1,1)                         
  42  i*r3(1,-1,1)                       
  43  r2(1,0,1)                          
  44  m(1,0,1)                           
  45  r2(1,-1,0)                         
  46  m(1,-1,0)                          
  47  r2(-0,1,1)                         
  48  m(-0,1,1)                          
 GROUPG: the following are sufficient to generate the space group:
 Generator(cart): i*r3(1,1,-1) r4x
 Generator(frac): i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 goto freats

ttt: pnu qat=  1  0     4.650     1.000
ttt: pnu qat=  1  1     4.340     0.000
ttt: pnu qat=  1  2     3.870    10.000
ttt: pnu qat=  1  3     4.110     0.000

conf:------------------------------------------------------
conf:SPEC_ATOM= A : --- Table for atomic configuration ---
conf:  isp  l  int(P) int(P)z    Qval     Qcore   CoreConf
conf:    1  0       4  5         1.000    6.000 => 1,2,3,
conf:    1  1       4  5         0.000   12.000 => 2,3,
conf:    1  2       3  4        10.000    0.000 => 
conf:    1  3       4  0         0.000    0.000 => 
usedQ=     1.000     0.000    10.000     0.000

conf: Species A:  Z=29  Qc=18  R=2.311271  Q=0
conf:   rmt=2.311271  rmax=48.805862  a=0.025  nr=393  nr(rmax)=515
 goto atomc xxx
 atomsc nmcore=           0

  iter     qint         drho          vh0          rho0          vsum     beta
    1   29.000000   4.725E+03      145.0000    0.1442E+03      -58.2772   0.30
   51   29.000000   4.211E-05      274.8263    0.2631E+05     -130.7915   0.30

 end of atomsc xxxxx
 vsum=  -130.79144076069792                1

 sumev=-4.333254  etot=-3304.416258  eref=-3304.434500  diff= 0.018242

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)
   4s      -0.36411         0.890       2.256       3.582     0.643062
   5s      -0.00028         3.669      10.794      19.873     0.990448
   4p      -0.06295         0.975       3.484       7.414     0.901829
   5p       0.00796         6.760      30.414      48.806*    0.999240
   3d      -0.39691         0.000       0.600       3.429     0.056076
   4d       0.01308         1.868      33.290      48.806*    0.999995
   4f       0.01948         0.000      35.393      48.806*    1.000000

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000141
   3p      -5.29682         0.260       0.619       1.078     0.000727

 Optimise free-atom basis for species A, rmt=2.311271
 l  it    Rsm      Eh     stiffR   stiffE      Eval      Exact     Pnu    Ql
 0  10   1.156  -0.108       0.0   6566.8   -0.34267  -0.36411    4.76   1.00
 1  11   1.156  -0.100       0.0  -3498.9    0.17087  -0.06295    4.56   0.00
 2   6   1.156  -0.509       0.0     20.5   -0.39483  -0.39691    3.89  10.00
 eigenvalue sum:  exact  -4.33325    opt basis  -4.29094    error 0.04231
 tailsm: init

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.31127, rsm= 1.15564
  ---E:energies of smHankels. C:fitting coeeficient for core tail. ---
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000    -15.0000
 C:    -0.07160    10.75053    -187.492    1222.023    -4717.79    21166.81
        r          rho         fit         diff
    2.311271    0.017797    0.017766    0.000031
    2.967767    0.005662    0.005658    0.000005
    3.810725    0.001517    0.001518   -0.000001
    4.893104    0.000305    0.000305   -0.000000
    6.282906    0.000041    0.000041   -0.000001
    8.067448    0.000003    0.000003    0.000000
    q(fit):     1.203836    rms diff:   0.000016
    fit: r>rmt  1.203836   r<rmt  3.442816   qtot  4.646652
    rho: r>rmt  1.203836   r<rmt  9.796164   qtot 11.000000
 tailsm: end
conf: Core rhoc(rmt)= 0.003922 spillout= 0.004646
 Fit with Hankel e=-24.082483 coeff=764.352513
      r            rhoc          fit
    2.311271    0.02095279    0.02095279
    2.429779    0.01229068    0.01231367
    2.753317    0.00285262    0.00285190
    3.119934    0.00054243    0.00053465
    3.535366    0.00008235    0.00007888
    4.006112    0.00000969    0.00000887
    4.539536    0.00000085    0.00000073
    5.143985    0.00000005    0.00000004
 end of freats: spid nmcore=A                  0
Sum of reference energies:                  -3304.434500000000
OK! end of LMFA ======================
===START LMF   =====================================
 mpisize=           4
  bndfp (warning): no sigm file found ... LDA calculation only

mmm === MTO setting ===
mmm ispec lmxb lpzex nkapii nkaphh=    1    3    0    1    1
mmm rsmh1    1  2.50  2.50  1.00  0.00
mmm   eh1    1 -0.01 -0.01 -0.01 -0.01
mmm rsmh2    1  0.00  0.00  0.00  0.00
mmm  eh2     1  0.00  0.00  0.00  0.00
mmm pz       1  5.50  5.50  4.50  0.00
mmm lh       1  2  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 ADDBAS: basis is already complete --- no sites added
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 ig  group op
   1  i*i                                
   2  i                                  
   3  r3(1,1,-1)                         
   4  i*r3(1,1,-1)                       
   5  r3(-1,-1,1)                        
   6  i*r3(-1,-1,1)                      
   7  r3d                                
   8  i*r3d                              
   9  r3(-1,-1,-1)                       
  10  i*r3(-1,-1,-1)                     
  11  r2x                                
  12  mx                                 
  13  r4x                                
  14  i*r4x                              
  15  r4(-1,0,0)                         
  16  i*r4(-1,0,0)                       
  17  r3(1,-1,-1)                        
  18  i*r3(1,-1,-1)                      
  19  r3(-1,1,1)                         
  20  i*r3(-1,1,1)                       
  21  r2(1,1,-0)                         
  22  m(1,1,-0)                          
  23  r2(1,-0,-1)                        
  24  m(1,-0,-1)                         
  25  r2y                                
  26  my                                 
  27  r4y                                
  28  i*r4y                              
  29  r4(0,-1,0)                         
  30  i*r4(0,-1,0)                       
  31  r2(0,1,-1)                         
  32  m(0,1,-1)                          
  33  r2z                                
  34  mz                                 
  35  r4(0,0,-1)                         
  36  i*r4(0,0,-1)                       
  37  r4z                                
  38  i*r4z                              
  39  r3(-1,1,-1)                        
  40  i*r3(-1,1,-1)                      
  41  r3(1,-1,1)                         
  42  i*r3(1,-1,1)                       
  43  r2(1,0,1)                          
  44  m(1,0,1)                           
  45  r2(1,-1,0)                         
  46  m(1,-1,0)                          
  47  r2(-0,1,1)                         
  48  m(-0,1,1)                          
 GROUPG: the following are sufficient to generate the space group:
 Generator(cart): i*r3(1,1,-1) r4x
 Generator(frac): i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found
 >> level: 1  CPUsec=      0.06  enter lmfp
 gen_hamindex: not readin QGpsi.

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    3    4         3  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1

 GVLST2: gmax = 9.0 a.u. created 941 vectors of 1331 (70%)
         (input) mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 
 goto end of reading rst or atm           1

 iors  : read restart file (binary, mesh density) 
 iors  : empty file ... nothing read

 rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected A,       read A        with rmt=  2.3113  mesh   393  0.025

 ovlpfa: overlap smooth part of FA densities
 total smooth Q = 4.646654

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
 end of reading rst or atm
 end of reading rst or atm
 end of reading rst or atm
    1    9.796164    3.442818   10.275300    3.921954    6.353346

 Smooth charge on mesh:            4.646654
 Sum of local charges:             6.353346
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:       -0.000000
 end of reading rst or atm
 m_qplistinit:start
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    1, (   16 1),  (   30 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    2, (   31 1),  (   45 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    3, (   46 1),  (   60 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    0, (    1 1),  (   15 1)

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1  A         0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

 --- BNDFP:  begin iteration 1 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.554993  avg sphere pot= 0.633521  vconst=-0.554993

 smooth rhoves     11.022237   charge     4.646654
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -3.843801   rhomu =   -5.010455  avg vxc =   -0.851784 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000013

 Energy terms:             smooth           local           total
   rhoval*vef            -12.156987      -177.336819      -189.493805
   rhoval*ves            -46.689417      -115.324371      -162.013788
   psnuc*ves              68.733890    -12976.662453    -12907.928563
   utot                   11.022237     -6545.993412     -6534.971175
   rho*exc                -3.843801      -126.414296      -130.258096
   rho*vxc                -5.010455      -167.409313      -172.419769
   valence chg             4.646654         6.353346        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0404

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144577;  11.000000 electrons
         Sum occ. bands:   -0.853241, incl. Bloechl correction: -0.006586
Generating TDOS: efermi, and dos window=    0.1446  -0.5000   1.6446
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1    9.927753    3.113494    6.814259

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.508835   -0.349850    4.650000    4.662521    4.500000    4.662521
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.534756   -0.166636    4.340000    4.404039    4.250000    4.404039
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.860630   -0.041120    3.870000    3.867798    3.147584    3.867798
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.023532   -0.061245    4.110000    4.125360    4.102416    4.110000

 Harris energy:
 sumev=       -0.853241  val*vef=    -189.493805   sumtv=     188.640564
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -130.258096     utot=   -6534.971175    ehar=   -3304.832068

 srhov:     -6.360832   -168.222947   -174.583779 sumev=   -0.853241   sumtv=  173.730538
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.677273  avg sphere pot= 0.653661  vconst=-0.677273

 smooth rhoves     13.178917   charge     4.185741
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -3.054117   rhomu =   -3.974963  avg vxc =   -0.866699 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -7.030294      -175.113437      -182.143731
   rhoval*ves            -50.182882      -106.215168      -156.398050
   psnuc*ves              76.540716    -12962.871091    -12886.330375
   utot                   13.178917     -6534.543129     -6521.364212
   rho*exc                -3.054117      -125.587140      -128.641257
   rho*vxc                -3.974963      -166.302313      -170.277276
   valence chg             4.185741         6.814259        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      173.730538  sumtc=      3171.756639   ekin=     3345.487177
 rhoep=     -128.641257   utot=     -6521.364212   ehks=    -3304.518292
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrealsmooth= T
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 0.  RMS DQ=3.60e-2
 AMIX: nmix=0 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=3.60D-02
 mixrho: add corrections to qcell smrho = -0.23316D-07 -0.29688D-09
 unscreened rms difference:  smooth  0.045849   local  0.019256
   screened rms difference:  smooth  0.045849   local  0.019256   tot  0.035987

 iors  : write restart file (binary, mesh density) 

   it  1  of 12    ehf=      -0.397568   ehk=      -0.083792
h nk=8 bigbas=0 ehf(eV)=-5.4092284 ehk(eV)=-1.1400635 sev(eV)=-11.6090277

 --- BNDFP:  begin iteration 2 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.633049  avg sphere pot= 0.653661  vconst=-0.633049

 smooth rhoves     12.286148   charge     4.185741
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -3.109207   rhomu =   -4.047536  avg vxc =   -0.858397 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000645

 Energy terms:             smooth           local           total
   rhoval*vef             -7.713664      -175.301697      -183.015361
   rhoval*ves            -48.957953      -107.937662      -156.895615
   psnuc*ves              73.530250    -12964.307473    -12890.777223
   utot                   12.286148     -6536.122568     -6523.836419
   rho*exc                -3.109207      -125.879109      -128.988316
   rho*vxc                -4.047536      -166.689181      -170.736716
   valence chg             4.185741         6.814259        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0599

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.191168;  11.000000 electrons
         Sum occ. bands:   -7.093537, incl. Bloechl correction: -0.013293
Generating TDOS: efermi, and dos window=   -0.1912  -0.5000   1.3088

 mkrout:  Qtrue      sm,loc       local
   1   10.391031    1.889094    8.501937

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.422469   -0.510146    4.662521    4.665863    4.500000    4.665863
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.259043   -0.416122    4.404039    4.372230    4.250000    4.372230
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.699171   -0.662756    3.867798    3.901921    3.147584    3.901921
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.010349   -0.615446    4.110000    4.111973    4.102416    4.110000

 Harris energy:
 sumev=       -7.093537  val*vef=    -183.015361   sumtv=     175.921823
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -128.988316     utot=   -6523.836419    ehar=   -3305.146273

 srhov:     -4.200079   -222.070552   -226.270631 sumev=   -7.093537   sumtv=  219.177094
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.400483  avg sphere pot= 0.664691  vconst=-0.400483

 smooth rhoves      4.818396   charge     2.498063
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -1.661083   rhomu =   -2.159363  avg vxc =   -0.746182 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -3.180685      -205.534191      -208.714876
   rhoval*ves            -36.498704      -142.152533      -178.651237
   psnuc*ves              46.135497    -12991.549490    -12945.413994
   utot                    4.818396     -6566.851012     -6562.032615
   rho*exc                -1.661083      -131.002602      -132.663685
   rho*vxc                -2.159363      -173.451554      -175.610917
   valence chg             2.498063         8.501937        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000

 Kohn-Sham energy:
 sumtv=      219.177094  sumtc=      3171.756639   ekin=     3390.933733
 rhoep=     -132.663685   utot=     -6562.032615   ehks=    -3303.762566
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 1.  RMS DQ=9.09e-2  last it=3.60e-2
 AMIX: nmix=1 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=9.09D-02
   tj: 0.82101
 mixrho: add corrections to qcell smrho = -0.11103D-07 -0.14137D-09
 unscreened rms difference:  smooth  0.024246   local  0.064870
   screened rms difference:  smooth  0.024246   local  0.064870   tot  0.090874

 iors  : write restart file (binary, mesh density) 

   it  2  of 12    ehf=      -0.711773   ehk=       0.671934
 From last iter    ehf=      -0.397568   ehk=      -0.083792
 diffe(q)= -0.314205 (0.090874)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-9.6842426 ehk(eV)=9.1421935 sev(eV)=-96.5132507

 --- BNDFP:  begin iteration 3 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.604915  avg sphere pot= 0.655635  vconst=-0.604915

 smooth rhoves     10.955967   charge     3.883666
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.817548   rhomu =   -3.666817  avg vxc =   -0.843500 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000597

 Energy terms:             smooth           local           total
   rhoval*vef             -6.576698      -181.158821      -187.735520
   rhoval*ves            -47.654907      -113.403665      -161.058572
   psnuc*ves              69.566842    -12968.686245    -12899.119404
   utot                   10.955967     -6541.044955     -6530.088988
   rho*exc                -2.817548      -126.698751      -129.516299
   rho*vxc                -3.666817      -167.770394      -171.437211
   valence chg             3.883666         7.116334        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0592

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.121679;  11.000000 electrons
         Sum occ. bands:   -4.729081, incl. Bloechl correction: -0.011778
Generating TDOS: efermi, and dos window=   -0.1217  -0.5000   1.3783

 mkrout:  Qtrue      sm,loc       local
   1   10.274203    2.258616    8.015587

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.448182   -0.471870    4.665863    4.661731    4.500000    4.661731
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.303498   -0.374406    4.372230    4.371002    4.250000    4.371002
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.508935   -0.428368    3.901921    3.890746    3.147584    3.890746
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.013587   -0.419417    4.110000    4.116636    4.102416    4.110000

 Harris energy:
 sumev=       -4.729081  val*vef=    -187.735520   sumtv=     183.006439
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.516299     utot=   -6530.088988    ehar=   -3304.842208

 srhov:     -4.839652   -203.034353   -207.874005 sumev=   -4.729081   sumtv=  203.144925
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.476735  avg sphere pot= 0.666104  vconst=-0.476735

 smooth rhoves      6.772582   charge     2.984414
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.045659   rhomu =   -2.660386  avg vxc =   -0.784527 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -4.183503      -195.306394      -199.489896
   rhoval*ves            -41.181371      -129.701838      -170.883209
   psnuc*ves              54.726535    -12980.016447    -12925.289912
   utot                    6.772582     -6554.859143     -6548.086561
   rho*exc                -2.045659      -129.270665      -131.316325
   rho*vxc                -2.660386      -171.163634      -173.824020
   valence chg             2.984414         8.015587        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      203.144925  sumtc=      3171.756639   ekin=     3374.901564
 rhoep=     -131.316325   utot=     -6548.086561   ehks=    -3304.501321
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 2.  RMS DQ=4.40e-2  last it=9.09e-2
 AMIX: nmix=2 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=4.40D-02
   tj:-1.03927  -0.10625
 mixrho: add corrections to qcell smrho = -0.30441D-06 -0.38759D-08
 unscreened rms difference:  smooth  0.012620   local  0.032800
   screened rms difference:  smooth  0.012620   local  0.032800   tot  0.044025

 iors  : write restart file (binary, mesh density) 

   it  3  of 12    ehf=      -0.407708   ehk=      -0.066821
 From last iter    ehf=      -0.711773   ehk=       0.671934
 diffe(q)=  0.304065 (0.044025)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-5.54719 ehk(eV)=-.9091513 sev(eV)=-64.3429236

 --- BNDFP:  begin iteration 4 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.545146  avg sphere pot= 0.668895  vconst=-0.545146

 smooth rhoves      8.671497   charge     3.362221
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.347351   rhomu =   -3.053460  avg vxc =   -0.813370 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000549

 Energy terms:             smooth           local           total
   rhoval*vef             -4.920304      -185.951724      -190.872028
   rhoval*ves            -44.678446      -118.834502      -163.512948
   psnuc*ves              62.021441    -12969.566049    -12907.544608
   utot                    8.671497     -6544.200275     -6535.528778
   rho*exc                -2.347351      -127.805101      -130.152452
   rho*vxc                -3.053460      -169.227031      -172.280492
   valence chg             3.362221         7.637779        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0608

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.066062;  11.000000 electrons
         Sum occ. bands:   -1.734814, incl. Bloechl correction: -0.007558
Generating TDOS: efermi, and dos window=    0.0661  -0.5000   1.5661

 mkrout:  Qtrue      sm,loc       local
   1   10.006010    2.935456    7.070553

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.497640   -0.390258    4.661731    4.662100    4.500000    4.662100
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.475293   -0.227619    4.371002    4.395126    4.250000    4.395126
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.011970   -0.128804    3.890746    3.872855    3.147584    3.872855
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021106   -0.143522    4.110000    4.123593    4.102416    4.110000

 Harris energy:
 sumev=       -1.734814  val*vef=    -190.872028   sumtv=     189.137214
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -130.152452     utot=   -6535.528778    ehar=   -3304.787376

 srhov:     -5.820463   -174.784474   -180.604937 sumev=   -1.734814   sumtv=  178.870123
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.633468  avg sphere pot= 0.658285  vconst=-0.633468

 smooth rhoves     11.622224   charge     3.929447
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.832043   rhomu =   -3.685389  avg vxc =   -0.850506 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -6.380268      -178.810105      -185.190373
   rhoval*ves            -48.593219      -110.325860      -158.919079
   psnuc*ves              71.837667    -12965.241024    -12893.403357
   utot                   11.622224     -6537.783442     -6526.161218
   rho*exc                -2.832043      -126.303874      -129.135916
   rho*vxc                -3.685389      -167.247614      -170.933003
   valence chg             3.929447         7.070553        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      178.870123  sumtc=      3171.756639   ekin=     3350.626762
 rhoep=     -129.135916   utot=     -6526.161218   ehks=    -3304.670372
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 3.  RMS DQ=2.43e-2  last it=4.40e-2
 AMIX: nmix=3 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=2.43D-02
   tj: 0.76321  -0.24310  -0.00278
 mixrho: add corrections to qcell smrho = -0.15776D-06 -0.20086D-08
 unscreened rms difference:  smooth  0.007483   local  0.019601
   screened rms difference:  smooth  0.007483   local  0.019601   tot  0.024318

 iors  : write restart file (binary, mesh density) 

   it  4  of 12    ehf=      -0.352876   ehk=      -0.235872
 From last iter    ehf=      -0.407708   ehk=      -0.066821
 diffe(q)=  0.054831 (0.024318)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-4.8011663 ehk(eV)=-3.2092281 sev(eV)=-23.6035374

 --- BNDFP:  begin iteration 5 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.570017  avg sphere pot= 0.662708  vconst=-0.570017

 smooth rhoves      9.528920   charge     3.555451
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.515365   rhomu =   -3.272549  avg vxc =   -0.825511 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000560

 Energy terms:             smooth           local           total
   rhoval*vef             -5.478201      -184.923907      -190.402109
   rhoval*ves            -45.930153      -117.336551      -163.266704
   psnuc*ves              64.987992    -12970.137903    -12905.149910
   utot                    9.528920     -6543.737227     -6534.208307
   rho*exc                -2.515365      -127.430514      -129.945879
   rho*vxc                -3.272549      -168.734285      -172.006835
   valence chg             3.555451         7.444549        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0618

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014429;  11.000000 electrons
         Sum occ. bands:   -2.765369, incl. Bloechl correction: -0.009079
Generating TDOS: efermi, and dos window=   -0.0144  -0.5000   1.4856

 mkrout:  Qtrue      sm,loc       local
   1   10.107144    2.683239    7.423905

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.474968   -0.427641    4.662100    4.659693    4.500000    4.659693
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.409567   -0.282707    4.395126    4.386369    4.250000    4.386369
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.204393   -0.232830    3.872855    3.878975    3.147584    3.878975
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018215   -0.237482    4.110000    4.121228    4.102416    4.110000

 Harris energy:
 sumev=       -2.765369  val*vef=    -190.402109   sumtv=     187.636739
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.945879     utot=   -6534.208307    ehar=   -3304.760807

 srhov:     -5.487174   -184.461023   -189.948197 sumev=   -2.765369   sumtv=  187.182828
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.574641  avg sphere pot= 0.662487  vconst=-0.574641

 smooth rhoves      9.656570   charge     3.576095
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.531012   rhomu =   -3.292924  avg vxc =   -0.827227 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.513162      -184.684059      -190.197221
   rhoval*ves            -46.124076      -116.991394      -163.115470
   psnuc*ves              65.437216    -12969.929462    -12904.492246
   utot                    9.656570     -6543.460428     -6533.803858
   rho*exc                -2.531012      -127.365102      -129.896114
   rho*vxc                -3.292924      -168.647975      -171.940899
   valence chg             3.576095         7.423905        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.182828  sumtc=      3171.756639   ekin=     3358.939467
 rhoep=     -129.896114   utot=     -6533.803858   ehks=    -3304.760505
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=9.70e-4  last it=2.43e-2
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=9.70D-04
   tj:-0.04331  -0.00109
 mixrho: add corrections to qcell smrho = -0.12110D-07 -0.15419D-09
 unscreened rms difference:  smooth  0.000347   local  0.000768
   screened rms difference:  smooth  0.000347   local  0.000768   tot  0.000970

 iors  : write restart file (binary, mesh density) 

   it  5  of 12    ehf=      -0.326307   ehk=      -0.326005
 From last iter    ehf=      -0.352876   ehk=      -0.235872
 diffe(q)=  0.026570 (0.000970)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-4.4396633 ehk(eV)=-4.435552 sev(eV)=-37.6250635

 --- BNDFP:  begin iteration 6 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571791  avg sphere pot= 0.662665  vconst=-0.571791

 smooth rhoves      9.571667   charge     3.561435
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.519250   rhomu =   -3.277600  avg vxc =   -0.826128 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.484067      -184.917312      -190.401379
   rhoval*ves            -45.999700      -117.282483      -163.282183
   psnuc*ves              65.143033    -12970.133038    -12904.990005
   utot                    9.571667     -6543.707760     -6534.136094
   rho*exc                -2.519250      -127.411797      -129.931048
   rho*vxc                -3.277600      -168.709607      -171.987206
   valence chg             3.561435         7.438565        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0618

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020515;  11.000000 electrons
         Sum occ. bands:   -2.851611, incl. Bloechl correction: -0.009203
Generating TDOS: efermi, and dos window=   -0.0205  -0.5000   1.4795

 mkrout:  Qtrue      sm,loc       local
   1   10.116130    2.663360    7.452770

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.474100   -0.430105    4.659693    4.659887    4.500000    4.659887
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.402209   -0.289089    4.386369    4.385081    4.250000    4.385081
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.221874   -0.241356    3.878975    3.879541    3.147584    3.879541
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.017946   -0.246180    4.110000    4.121017    4.102416    4.110000

 Harris energy:
 sumev=       -2.851611  val*vef=    -190.401379   sumtv=     187.549768
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.931048     utot=   -6534.136094    ehar=   -3304.760734

 srhov:     -5.458720   -185.248220   -190.706941 sumev=   -2.851611   sumtv=  187.855330
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.569663  avg sphere pot= 0.662829  vconst=-0.569663

 smooth rhoves      9.501721   charge     3.547230
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.506989   rhomu =   -3.261612  avg vxc =   -0.825222 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.446393      -185.134697      -190.581090
   rhoval*ves            -45.901266      -117.530591      -163.431857
   psnuc*ves              64.904709    -12970.299804    -12905.395095
   utot                    9.501721     -6543.915197     -6534.413476
   rho*exc                -2.506989      -127.452153      -129.959142
   rho*vxc                -3.261612      -168.762844      -172.024457
   valence chg             3.547230         7.452770        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.855330  sumtc=      3171.756639   ekin=     3359.611969
 rhoep=     -129.959142   utot=     -6534.413476   ehks=    -3304.760649
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=6.83e-4  last it=9.70e-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=6.83D-04
   tj: 0.33583   0.00509
 mixrho: add corrections to qcell smrho = -0.34565D-07 -0.44010D-09
 unscreened rms difference:  smooth  0.000201   local  0.000518
   screened rms difference:  smooth  0.000201   local  0.000518   tot  0.000683

 iors  : write restart file (binary, mesh density) 

   it  6  of 12    ehf=      -0.326234   ehk=      -0.326149
 From last iter    ehf=      -0.326307   ehk=      -0.326005
 diffe(q)=  0.000072 (0.000683)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-4.4386771 ehk(eV)=-4.4375148 sev(eV)=-38.7984487

 --- BNDFP:  begin iteration 7 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571487  avg sphere pot= 0.662691  vconst=-0.571487

 smooth rhoves      9.560764   charge     3.558869
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.516925   rhomu =   -3.274566  avg vxc =   -0.825985 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.475824      -184.951073      -190.426897
   rhoval*ves            -45.985079      -117.318257      -163.303336
   psnuc*ves              65.106608    -12970.154285    -12905.047678
   utot                    9.560764     -6543.736271     -6534.175507
   rho*exc                -2.516925      -127.418209      -129.935134
   rho*vxc                -3.274566      -168.718059      -171.992625
   valence chg             3.558869         7.441131        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0624

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.019274;  11.000000 electrons
         Sum occ. bands:   -2.833626, incl. Bloechl correction: -0.009175
Generating TDOS: efermi, and dos window=   -0.0193  -0.5000   1.4807

 mkrout:  Qtrue      sm,loc       local
   1   10.114510    2.667674    7.446837

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.474528   -0.429518    4.659887    4.659941    4.500000    4.659941
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.403127   -0.288311    4.385081    4.385194    4.250000    4.385194
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.218864   -0.239543    3.879541    3.879438    3.147584    3.879438
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.017991   -0.244580    4.110000    4.121059    4.102416    4.110000

 Harris energy:
 sumev=       -2.833626  val*vef=    -190.426897   sumtv=     187.593271
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.935134     utot=   -6534.175507    ehar=   -3304.760730

 srhov:     -5.465520   -185.071917   -190.537437 sumev=   -2.833626   sumtv=  187.703811
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.570633  avg sphere pot= 0.662778  vconst=-0.570633

 smooth rhoves      9.532766   charge     3.553163
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.511985   rhomu =   -3.268125  avg vxc =   -0.825624 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.460586      -185.029146      -190.489732
   rhoval*ves            -45.945769      -117.409172      -163.354940
   psnuc*ves              65.011301    -12970.207442    -12905.196141
   utot                    9.532766     -6543.808307     -6534.275540
   rho*exc                -2.511985      -127.433643      -129.945628
   rho*vxc                -3.268125      -168.738413      -172.006538
   valence chg             3.553163         7.446837        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.703811  sumtc=      3171.756639   ekin=     3359.460450
 rhoep=     -129.945628   utot=     -6534.275540   ehks=    -3304.760718
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=2.57e-4  last it=6.83e-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=2.57D-04
   tj:-0.28729   0.11022
 mixrho: add corrections to qcell smrho = -0.43586D-07 -0.55496D-09
 unscreened rms difference:  smooth  0.000082   local  0.000202
   screened rms difference:  smooth  0.000082   local  0.000202   tot  0.000257

 iors  : write restart file (binary, mesh density) 

   it  7  of 12    ehf=      -0.326230   ehk=      -0.326218
 From last iter    ehf=      -0.326234   ehk=      -0.326149
 diffe(q)=  0.000004 (0.000257)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-4.4386265 ehk(eV)=-4.4384605 sev(eV)=-38.5537531

 --- BNDFP:  begin iteration 8 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571307  avg sphere pot= 0.662732  vconst=-0.571307

 smooth rhoves      9.554408   charge     3.557396
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.515588   rhomu =   -3.272822  avg vxc =   -0.825903 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.471168      -184.960527      -190.431695
   rhoval*ves            -45.976525      -117.329935      -163.306459
   psnuc*ves              65.085341    -12970.151648    -12905.066307
   utot                    9.554408     -6543.740791     -6534.186383
   rho*exc                -2.515588      -127.421102      -129.936690
   rho*vxc                -3.272822      -168.721864      -171.994687
   valence chg             3.557396         7.442604        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0626

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018741;  11.000000 electrons
         Sum occ. bands:   -2.825995, incl. Bloechl correction: -0.009164
Generating TDOS: efermi, and dos window=   -0.0187  -0.5000   1.4813

 mkrout:  Qtrue      sm,loc       local
   1   10.113771    2.669475    7.444296

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.474661   -0.429287    4.659941    4.659944    4.500000    4.659944
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.403634   -0.287883    4.385194    4.385268    4.250000    4.385268
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.217464   -0.238781    3.879438    3.879391    3.147584    3.879391
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018012   -0.243869    4.110000    4.121076    4.102416    4.110000

 Harris energy:
 sumev=       -2.825995  val*vef=    -190.431695   sumtv=     187.605700
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.936690     utot=   -6534.186383    ehar=   -3304.760733

 srhov:     -5.468001   -184.999057   -190.467058 sumev=   -2.825995   sumtv=  187.641063
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571059  avg sphere pot= 0.662754  vconst=-0.571059

 smooth rhoves      9.546210   charge     3.555704
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.514113   rhomu =   -3.270898  avg vxc =   -0.825798 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.466570      -184.986147      -190.452717
   rhoval*ves            -45.965088      -117.358947      -163.324035
   psnuc*ves              65.057508    -12970.170512    -12905.113004
   utot                    9.546210     -6543.764730     -6534.218519
   rho*exc                -2.514113      -127.425803      -129.939915
   rho*vxc                -3.270898      -168.728065      -171.998963
   valence chg             3.555704         7.444296        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.641063  sumtc=      3171.756639   ekin=     3359.397703
 rhoep=     -129.939915   utot=     -6534.218519   ehks=    -3304.760732
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=7.98e-5  last it=2.57e-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=7.98D-05
   tj:-0.44767
 mixrho: add corrections to qcell smrho = -0.45494D-07 -0.57926D-09
 unscreened rms difference:  smooth  0.000026   local  0.000061
   screened rms difference:  smooth  0.000026   local  0.000061   tot  0.000080

 iors  : write restart file (binary, mesh density) 

   it  8  of 12    ehf=      -0.326233   ehk=      -0.326232
 From last iter    ehf=      -0.326230   ehk=      -0.326218
 diffe(q)= -0.000002 (0.000080)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf(eV)=-4.4386604 ehk(eV)=-4.438643 sev(eV)=-38.4499223

 --- BNDFP:  begin iteration 9 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571245  avg sphere pot= 0.662743  vconst=-0.571245

 smooth rhoves      9.552147   charge     3.556842
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.515072   rhomu =   -3.272149  avg vxc =   -0.825875 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.469316      -184.966836      -190.436152
   rhoval*ves            -45.973562      -117.336602      -163.310164
   psnuc*ves              65.077856    -12970.154088    -12905.076232
   utot                    9.552147     -6543.745345     -6534.193198
   rho*exc                -2.515072      -127.422323      -129.937396
   rho*vxc                -3.272149      -168.723473      -171.995623
   valence chg             3.556842         7.443158        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0633

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018528;  11.000000 electrons
         Sum occ. bands:   -2.822934, incl. Bloechl correction: -0.009159
Generating TDOS: efermi, and dos window=   -0.0185  -0.5000   1.4815

 mkrout:  Qtrue      sm,loc       local
   1   10.113462    2.670237    7.443225

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.474718   -0.429201    4.659944    4.659947    4.500000    4.659947
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.403844   -0.287714    4.385268    4.385300    4.250000    4.385300
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.216880   -0.238474    3.879391    3.879372    3.147584    3.879372
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018021   -0.243583    4.110000    4.121084    4.102416    4.110000

 Harris energy:
 sumev=       -2.822934  val*vef=    -190.436152   sumtv=     187.613219
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.937396     utot=   -6534.193198    ehar=   -3304.760735

 srhov:     -5.469141   -184.967426   -190.436566 sumev=   -2.822934   sumtv=  187.613633
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571238  avg sphere pot= 0.662745  vconst=-0.571238

 smooth rhoves      9.551870   charge     3.556775
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.515010   rhomu =   -3.272068  avg vxc =   -0.825871 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.469098      -184.967195      -190.436294
   rhoval*ves            -45.973204      -117.337041      -163.310245
   psnuc*ves              65.076944    -12970.153809    -12905.076865
   utot                    9.551870     -6543.745425     -6534.193555
   rho*exc                -2.515010      -127.422442      -129.937452
   rho*vxc                -3.272068      -168.723629      -171.995698
   valence chg             3.556775         7.443225        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.613633  sumtc=      3171.756639   ekin=     3359.370272
 rhoep=     -129.937452   utot=     -6534.193555   ehks=    -3304.760735
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=2.35e-6  last it=7.98e-5
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2405  beta=1.00000  tm= 5.00000  rmsdel=2.35D-06
   tj:-0.02441
 mixrho: add corrections to qcell smrho = -0.45484D-07 -0.57913D-09
 unscreened rms difference:  smooth  0.000001   local  0.000002
   screened rms difference:  smooth  0.000001   local  0.000002   tot  0.000002

 iors  : write restart file (binary, mesh density) 

   it  9  of 12    ehf=      -0.326235   ehk=      -0.326235
 From last iter    ehf=      -0.326233   ehk=      -0.326232
 diffe(q)= -0.000002 (0.000002)    tol= 0.000010 (0.000010)   more=F
c nk=8 bigbas=0 ehf(eV)=-4.4386899 ehk(eV)=-4.4386878 sev(eV)=-38.4082693
 >>      0.96   exit  lmfp            0.90
OK! end of LMF ======================
===START LMF   =====================================
 mpisize=           4
  bndfp (warning): no sigm file found ... LDA calculation only

mmm === MTO setting ===
mmm ispec lmxb lpzex nkapii nkaphh=    1    4    0    2    2
mmm rsmh1    1  2.50  2.50  1.00  0.00  0.00
mmm   eh1    1 -0.01 -0.01 -0.01 -0.01 -0.01
mmm rsmh2    1  1.30  0.00  1.00  1.30  0.00
mmm  eh2     1 -1.00 -1.00 -1.00 -0.01 -0.01
mmm pz       1  5.50  5.50  4.50  0.00  0.00
mmm lh       1  2  3  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 ADDBAS: basis is already complete --- no sites added
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 ig  group op
   1  i*i                                
   2  i                                  
   3  r3(1,1,-1)                         
   4  i*r3(1,1,-1)                       
   5  r3(-1,-1,1)                        
   6  i*r3(-1,-1,1)                      
   7  r3d                                
   8  i*r3d                              
   9  r3(-1,-1,-1)                       
  10  i*r3(-1,-1,-1)                     
  11  r2x                                
  12  mx                                 
  13  r4x                                
  14  i*r4x                              
  15  r4(-1,0,0)                         
  16  i*r4(-1,0,0)                       
  17  r3(1,-1,-1)                        
  18  i*r3(1,-1,-1)                      
  19  r3(-1,1,1)                         
  20  i*r3(-1,1,1)                       
  21  r2(1,1,-0)                         
  22  m(1,1,-0)                          
  23  r2(1,-0,-1)                        
  24  m(1,-0,-1)                         
  25  r2y                                
  26  my                                 
  27  r4y                                
  28  i*r4y                              
  29  r4(0,-1,0)                         
  30  i*r4(0,-1,0)                       
  31  r2(0,1,-1)                         
  32  m(0,1,-1)                          
  33  r2z                                
  34  mz                                 
  35  r4(0,0,-1)                         
  36  i*r4(0,0,-1)                       
  37  r4z                                
  38  i*r4z                              
  39  r3(-1,1,-1)                        
  40  i*r3(-1,1,-1)                      
  41  r3(1,-1,1)                         
  42  i*r3(1,-1,1)                       
  43  r2(1,0,1)                          
  44  m(1,0,1)                           
  45  r2(1,-1,0)                         
  46  m(1,-1,0)                          
  47  r2(-0,1,1)                         
  48  m(-0,1,1)                          
 GROUPG: the following are sufficient to generate the space group:
 Generator(cart): i*r3(1,1,-1) r4x
 Generator(frac): i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found
 >> level: 1  CPUsec=      0.11  enter lmfp
 gen_hamindex: not readin QGpsi.

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    4    4         4  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1

 GVLST2: gmax = 9.0 a.u. created 941 vectors of 1331 (70%)
         (input) mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     869 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 goto end of reading rst or atm           1

 iors  : read restart file (binary, mesh density) 
         use from  restart file: ef window, positions, pnu 
         ignore in restart file: *
         site   1, species A       : augmentation lmax changed from 3 to 4
         site   1, species A       : inflate local density from nlm= 16 to 25
 end of reading rst or atm
 m_qplistinit:start
 end of reading rst or atm
 end of reading rst or atm
 end of reading rst or atm
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    2, (   31 1),  (   45 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    3, (   46 1),  (   60 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    0, (    1 1),  (   15 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    1, (   16 1),  (   30 1)

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1  A         0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

 --- BNDFP:  begin iteration 1 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.571242  avg sphere pot= 0.662745  vconst=-0.571242

 smooth rhoves      9.552001   charge     3.556801
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.515033   rhomu =   -3.272098  avg vxc =   -0.825873 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.469166      -184.966732      -190.435899
   rhoval*ves            -45.973388      -117.336522      -163.309910
   psnuc*ves              65.077390    -12970.153415    -12905.076025
   utot                    9.552001     -6543.744968     -6534.192967
   rho*exc                -2.515033      -127.422363      -129.937395
   rho*vxc                -3.272098      -168.723525      -171.995622
   valence chg             3.556801         7.443199        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1069

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018806;  11.000000 electrons
         Sum occ. bands:   -2.825415, incl. Bloechl correction: -0.009153
Generating TDOS: efermi, and dos window=   -0.0188  -0.5000   1.4812
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1   10.128183    2.833411    7.294772

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.477550   -0.430054    4.659947    4.659574    4.500000    4.659574
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.409683   -0.289034    4.385300    4.384836    4.250000    4.384836
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.215375   -0.238670    3.879372    3.879149    3.147584    3.879149
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021180   -0.245040    4.110000    4.121040    4.102416    4.110000
 4     1    0.004395   -0.241226    5.100000    5.085239    5.077979    5.100000

 Harris energy:
 sumev=       -2.825415  val*vef=    -190.435899   sumtv=     187.610483
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.937395     utot=   -6534.192967    ehar=   -3304.763240

 srhov:     -5.929810   -184.355896   -190.285706 sumev=   -2.825415   sumtv=  187.460291
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.581228  avg sphere pot= 0.653085  vconst=-0.581228

 smooth rhoves     10.109125   charge     3.705228
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.657844   rhomu =   -3.458503  avg vxc =   -0.832964 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -6.009813      -184.303400      -190.313213
   rhoval*ves            -46.615397      -116.581954      -163.197351
   psnuc*ves              66.833647    -12971.741471    -12904.907824
   utot                   10.109125     -6544.161712     -6534.052587
   rho*exc                -2.657844      -127.269667      -129.927512
   rho*vxc                -3.458503      -168.523981      -171.982484
   valence chg             3.705228         7.294772        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.460291  sumtc=      3171.756639   ekin=     3359.216930
 rhoep=     -129.927512   utot=     -6534.052587   ehks=    -3304.763169
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrealsmooth= T
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 0.  RMS DQ=4.03e-3
 AMIX: nmix=0 mmix=8  nelts=  2567  beta=1.00000  tm= 5.00000  rmsdel=4.03D-03
 mixrho: add corrections to qcell smrho = -0.28596D-07 -0.36410D-09
 unscreened rms difference:  smooth  0.003489   local  0.004828
   screened rms difference:  smooth  0.003489   local  0.004828   tot  0.004029

 iors  : write restart file (binary, mesh density) 

   it  1  of 12    ehf=      -0.328740   ehk=      -0.328669
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf(eV)=-4.4727703 ehk(eV)=-4.4717991 sev(eV)=-38.4420351

 --- BNDFP:  begin iteration 2 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.581163  avg sphere pot= 0.653085  vconst=-0.581163

 smooth rhoves     10.109074   charge     3.705228
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.657570   rhomu =   -3.458141  avg vxc =   -0.833027 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000127

 Energy terms:             smooth           local           total
   rhoval*vef             -6.009705      -184.305076      -190.314780
   rhoval*ves            -46.616015      -116.582921      -163.198936
   psnuc*ves              66.834162    -12971.742020    -12904.907858
   utot                   10.109074     -6544.162471     -6534.053397
   rho*exc                -2.657570      -127.269970      -129.927540
   rho*vxc                -3.458141      -168.524383      -171.982525
   valence chg             3.705228         7.294772        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1141

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020713;  11.000000 electrons
         Sum occ. bands:   -2.853660, incl. Bloechl correction: -0.009204
Generating TDOS: efermi, and dos window=   -0.0207  -0.5000   1.4793
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1   10.131732    2.824840    7.306891

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.476763   -0.430575    4.659574    4.659654    4.500000    4.659654
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.407335   -0.290439    4.384836    4.384698    4.250000    4.384698
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.222235   -0.241551    3.879149    3.878964    3.147584    3.878964
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021025   -0.247691    4.110000    4.120990    4.102416    4.110000
 4     1    0.004375   -0.244081    5.100000    5.085219    5.077979    5.100000

 Harris energy:
 sumev=       -2.853660  val*vef=    -190.314780   sumtv=     187.461120
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.927540     utot=   -6534.053397    ehar=   -3304.763177

 srhov:     -5.992884   -184.684129   -190.677013 sumev=   -2.853660   sumtv=  187.823353
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.579170  avg sphere pot= 0.653068  vconst=-0.579170

 smooth rhoves     10.043246   charge     3.693109
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.647612   rhomu =   -3.445163  avg vxc =   -0.832142 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -5.979839      -184.558522      -190.538361
   rhoval*ves            -46.526457      -116.862469      -163.388925
   psnuc*ves              66.612948    -12971.992208    -12905.379260
   utot                   10.043246     -6544.427338     -6534.384093
   rho*exc                -2.647612      -127.311359      -129.958971
   rho*vxc                -3.445163      -168.579039      -172.024202
   valence chg             3.693109         7.306891        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.823353  sumtc=      3171.756639   ekin=     3359.579993
 rhoep=     -129.958971   utot=     -6534.384093   ehks=    -3304.763071
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 1.  RMS DQ=7.06e-4  last it=4.03e-3
 AMIX: nmix=1 mmix=8  nelts=  2567  beta=1.00000  tm= 5.00000  rmsdel=7.06D-04
   tj: 0.08237
 mixrho: add corrections to qcell smrho = -0.27092D-07 -0.34495D-09
 unscreened rms difference:  smooth  0.000177   local  0.000504
   screened rms difference:  smooth  0.000177   local  0.000504   tot  0.000706

 iors  : write restart file (binary, mesh density) 

   it  2  of 12    ehf=      -0.328677   ehk=      -0.328571
 From last iter    ehf=      -0.328740   ehk=      -0.328669
 diffe(q)=  0.000063 (0.000706)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf(eV)=-4.4719128 ehk(eV)=-4.4704729 sev(eV)=-38.8263262

 --- BNDFP:  begin iteration 3 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.579969  avg sphere pot= 0.653070  vconst=-0.579969

 smooth rhoves     10.060686   charge     3.694107
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.647618   rhomu =   -3.445160  avg vxc =   -0.832355 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000134

 Energy terms:             smooth           local           total
   rhoval*vef             -5.972883      -184.535218      -190.508101
   rhoval*ves            -46.555155      -116.812162      -163.367317
   psnuc*ves              66.676528    -12971.948724    -12905.272195
   utot                   10.060686     -6544.380443     -6534.319756
   rho*exc                -2.647618      -127.303347      -129.950964
   rho*vxc                -3.445160      -168.568437      -172.013597
   valence chg             3.694107         7.305893        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1143

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014055;  11.000000 electrons
         Sum occ. bands:   -2.757338, incl. Bloechl correction: -0.009050
Generating TDOS: efermi, and dos window=   -0.0141  -0.5000   1.4859
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1   10.121519    2.844848    7.276671

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.478617   -0.427974    4.659654    4.659807    4.500000    4.659807
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.413467   -0.285576    4.384698    4.385633    4.250000    4.385633
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.203556   -0.231840    3.878964    3.878370    3.147584    3.878370
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021441   -0.238755    4.110000    4.121233    4.102416    4.110000
 4     1    0.004439   -0.234481    5.100000    5.085324    5.077979    5.100000

 Harris energy:
 sumev=       -2.757338  val*vef=    -190.508101   sumtv=     187.750763
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.950964     utot=   -6534.319756    ehar=   -3304.763318

 srhov:     -6.020272   -183.674396   -189.694667 sumev=   -2.757338   sumtv=  186.937329
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.584657  avg sphere pot= 0.653080  vconst=-0.584657

 smooth rhoves     10.213952   charge     3.723329
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.672642   rhomu =   -3.477787  avg vxc =   -0.834273 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -6.050087      -183.956396      -190.006483
   rhoval*ves            -46.758791      -116.183008      -162.941799
   psnuc*ves              67.186695    -12971.399062    -12904.212367
   utot                   10.213952     -6543.791035     -6533.577083
   rho*exc                -2.672642      -127.207021      -129.879663
   rho*vxc                -3.477787      -168.441259      -171.919046
   valence chg             3.723329         7.276671        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      186.937329  sumtc=      3171.756639   ekin=     3358.693969
 rhoep=     -129.879663   utot=     -6533.577083   ehks=    -3304.762776
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 2.  RMS DQ=1.64e-3  last it=7.06e-4
 AMIX: nmix=2 mmix=8  nelts=  2567  beta=1.00000  tm= 5.00000  rmsdel=1.64D-03
   tj: 0.69965  -0.00288
 mixrho: add corrections to qcell smrho = -0.27555D-07 -0.35084D-09
 unscreened rms difference:  smooth  0.000386   local  0.001200
   screened rms difference:  smooth  0.000386   local  0.001200   tot  0.001644

 iors  : write restart file (binary, mesh density) 

   it  3  of 12    ehf=      -0.328818   ehk=      -0.328276
 From last iter    ehf=      -0.328677   ehk=      -0.328571
 diffe(q)= -0.000141 (0.001644)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf(eV)=-4.4738332 ehk(eV)=-4.4664635 sev(eV)=-37.5157855

 --- BNDFP:  begin iteration 4 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.580840  avg sphere pot= 0.653072  vconst=-0.580840

 smooth rhoves     10.095156   charge     3.702237
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.655089   rhomu =   -3.454908  avg vxc =   -0.832803 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.000601      -184.377573      -190.378174
   rhoval*ves            -46.597890      -116.656710      -163.254600
   psnuc*ves              66.788201    -12971.812644    -12905.024443
   utot                   10.095156     -6544.234677     -6534.139521
   rho*exc                -2.655089      -127.279699      -129.934788
   rho*vxc                -3.454908      -168.537230      -171.992138
   valence chg             3.702237         7.297763        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1174

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018665;  11.000000 electrons
         Sum occ. bands:   -2.823670, incl. Bloechl correction: -0.009156
Generating TDOS: efermi, and dos window=   -0.0187  -0.5000   1.4813
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1   10.128469    2.831122    7.297347

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.477343   -0.429770    4.659807    4.659704    4.500000    4.659704
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.409334   -0.288903    4.385633    4.385006    4.250000    4.385006
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.216244   -0.238530    3.878370    3.878782    3.147584    3.878782
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021154   -0.244903    4.110000    4.121067    4.102416    4.110000
 4     1    0.004395   -0.241090    5.100000    5.085253    5.077979    5.100000

 Harris energy:
 sumev=       -2.823670  val*vef=    -190.378174   sumtv=     187.554505
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.934788     utot=   -6534.139521    ehar=   -3304.763165

 srhov:     -6.001565   -184.368100   -190.369665 sumev=   -2.823670   sumtv=  187.545996
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.580908  avg sphere pot= 0.653065  vconst=-0.580908

 smooth rhoves     10.097097   charge     3.702653
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.655499   rhomu =   -3.455443  avg vxc =   -0.832819 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -6.001920      -184.370822      -190.372742
   rhoval*ves            -46.600334      -116.649596      -163.249930
   psnuc*ves              66.794528    -12971.808066    -12905.013538
   utot                   10.097097     -6544.228831     -6534.131734
   rho*exc                -2.655499      -127.278566      -129.934066
   rho*vxc                -3.455443      -168.535736      -171.991180
   valence chg             3.702653         7.297347        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.545996  sumtc=      3171.756639   ekin=     3359.302635
 rhoep=     -129.934066   utot=     -6534.131734   ehks=    -3304.763165
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 3.  RMS DQ=1.87e-5  last it=1.64e-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2567  beta=1.00000  tm= 5.00000  rmsdel=1.87D-05
   tj:-0.01137
 mixrho: add corrections to qcell smrho = -0.24169D-07 -0.30774D-09
 unscreened rms difference:  smooth  0.000020   local  0.000015
   screened rms difference:  smooth  0.000020   local  0.000015   tot  0.000019

 iors  : write restart file (binary, mesh density) 

   it  4  of 12    ehf=      -0.328665   ehk=      -0.328665
 From last iter    ehf=      -0.328818   ehk=      -0.328276
 diffe(q)=  0.000153 (0.000019)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf(eV)=-4.4717536 ehk(eV)=-4.4717501 sev(eV)=-38.4182852

 --- BNDFP:  begin iteration 5 of 12
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.580867  avg sphere pot= 0.653065  vconst=-0.580867

 smooth rhoves     10.095876   charge     3.702418
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.655288   rhomu =   -3.455167  avg vxc =   -0.832805 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.001281      -184.375641      -190.376922
   rhoval*ves            -46.598717      -116.654784      -163.253501
   psnuc*ves              66.790469    -12971.812622    -12905.022153
   utot                   10.095876     -6544.233703     -6534.137827
   rho*exc                -2.655288      -127.279351      -129.934639
   rho*vxc                -3.455167      -168.536773      -171.991940
   valence chg             3.702418         7.297582        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.1161

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018709;  11.000000 electrons
         Sum occ. bands:   -2.824260, incl. Bloechl correction: -0.009157
Generating TDOS: efermi, and dos window=   -0.0187  -0.5000   1.4813
  mmmmm m_bandcal_2nd

 mkrout:  Qtrue      sm,loc       local
   1   10.128519    2.830998    7.297521

 Symmetrize density..

 Make new boundary conditions for phi,phidot..
  pnunew: ebar: 
    without lo    : ebar = center of gravity of occupied states
    with lo & PZ>P: ebar for lo is meaningless(zero is shown). Use empty-sphere PZ.
                    ebar for valence is at the center of gravity of occ. states.
    with lo & PZ<P: ebar for lo is by atomic calculation phidx for given PZ
                    ebar for valence is at the Fermi energy.
  idmod=0-> If P=prty(fractional part is log.-derivative)<pfeee, we use pfree.

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.477331   -0.429793    4.659704    4.659701    4.500000    4.659701
 0     0      ---       0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.409302   -0.288938    4.385006    4.385001    4.250000    4.385001
 1     0      ---       0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    9.216341   -0.238588    3.878782    3.878784    3.147584    3.878784
 2     0      ---       0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021151   -0.244957    4.110000    4.121066    4.102416    4.110000
 4     1    0.004394   -0.241147    5.100000    5.085252    5.077979    5.100000

 Harris energy:
 sumev=       -2.824260  val*vef=    -190.376922   sumtv=     187.552662
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.934639     utot=   -6534.137827    ehar=   -3304.763165

 srhov:     -6.001430   -184.374036   -190.375466 sumev=   -2.824260   sumtv=  187.551205
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.580879  avg sphere pot= 0.653065  vconst=-0.580879

 smooth rhoves     10.096160   charge     3.702479
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.655352   rhomu =   -3.455252  avg vxc =   -0.832807 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647

 Energy terms:             smooth           local           total
   rhoval*vef             -6.001482      -184.374477      -190.375960
   rhoval*ves            -46.599071      -116.653592      -163.252663
   psnuc*ves              66.791390    -12971.811706    -12905.020316
   utot                   10.096160     -6544.232649     -6534.136489
   rho*exc                -2.655352      -127.279168      -129.934520
   rho*vxc                -3.455252      -168.536530      -171.991782
   valence chg             3.702479         7.297521        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Kohn-Sham energy:
 sumtv=      187.551205  sumtc=      3171.756639   ekin=     3359.307845
 rhoep=     -129.934520   utot=     -6534.136489   ehks=    -3304.763165
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 wgtsmooth=   2.7410122234342145E-002
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=3.14e-6  last it=1.87e-5
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2567  beta=1.00000  tm= 5.00000  rmsdel=3.14D-06
   tj:-0.19971
 mixrho: add corrections to qcell smrho = -0.26509D-07 -0.33752D-09
 unscreened rms difference:  smooth  0.000005   local  0.000002
   screened rms difference:  smooth  0.000005   local  0.000002   tot  0.000003

 iors  : write restart file (binary, mesh density) 

   it  5  of 12    ehf=      -0.328665   ehk=      -0.328665
 From last iter    ehf=      -0.328665   ehk=      -0.328665
 diffe(q)=  0.000000 (0.000003)    tol= 0.000010 (0.000010)   more=F
c nk=8 bigbas=1 pwmode=0 oveps=0 ehf(eV)=-4.4717483 ehk(eV)=-4.4717457 sev(eV)=-38.4263222
 >>      1.24   exit  lmfp            1.13
OK! end of LMF ======================
===START LMF   =====================================
 mpisize=           4
  bndfp (warning): no sigm file found ... LDA calculation only

mmm === MTO setting ===
mmm ispec lmxb lpzex nkapii nkaphh=    1    4    0    2    2
mmm rsmh1    1  2.50  2.50  1.00  0.00  0.00
mmm   eh1    1 -0.01 -0.01 -0.01 -0.01 -0.01
mmm rsmh2    1  1.30  0.00  1.00  1.30  0.00
mmm  eh2     1 -1.00 -1.00 -1.00 -0.01 -0.01
mmm pz       1  5.50  5.50  4.50  0.00  0.00
mmm lh       1  2  3  2

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 ADDBAS: basis is already complete --- no sites added
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 ig  group op
   1  i*i                                
   2  i                                  
   3  r3(1,1,-1)                         
   4  i*r3(1,1,-1)                       
   5  r3(-1,-1,1)                        
   6  i*r3(-1,-1,1)                      
   7  r3d                                
   8  i*r3d                              
   9  r3(-1,-1,-1)                       
  10  i*r3(-1,-1,-1)                     
  11  r2x                                
  12  mx                                 
  13  r4x                                
  14  i*r4x                              
  15  r4(-1,0,0)                         
  16  i*r4(-1,0,0)                       
  17  r3(1,-1,-1)                        
  18  i*r3(1,-1,-1)                      
  19  r3(-1,1,1)                         
  20  i*r3(-1,1,1)                       
  21  r2(1,1,-0)                         
  22  m(1,1,-0)                          
  23  r2(1,-0,-1)                        
  24  m(1,-0,-1)                         
  25  r2y                                
  26  my                                 
  27  r4y                                
  28  i*r4y                              
  29  r4(0,-1,0)                         
  30  i*r4(0,-1,0)                       
  31  r2(0,1,-1)                         
  32  m(0,1,-1)                          
  33  r2z                                
  34  mz                                 
  35  r4(0,0,-1)                         
  36  i*r4(0,0,-1)                       
  37  r4z                                
  38  i*r4z                              
  39  r3(-1,1,-1)                        
  40  i*r3(-1,1,-1)                      
  41  r3(1,-1,1)                         
  42  i*r3(1,-1,1)                       
  43  r2(1,0,1)                          
  44  m(1,0,1)                           
  45  r2(1,-1,0)                         
  46  m(1,-1,0)                          
  47  r2(-0,1,1)                         
  48  m(-0,1,1)                          
 GROUPG: the following are sufficient to generate the space group:
 Generator(cart): i*r3(1,1,-1) r4x
 Generator(frac): i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found
 >> level: 1  CPUsec=      0.05  enter lmfp
 gen_hamindex: not readin QGpsi.

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    4    4         4  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1
 goto end of reading rst or atm           1

 GVLST2: gmax = 9.0 a.u. created 941 vectors of 1331 (70%)
         (input) mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     869 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 goto end of reading rst or atm           1

 iors  : read restart file (binary, mesh density) 
         use from  restart file: ef window, positions, pnu 
         ignore in restart file: *
 end of reading rst or atm
 end of reading rst or atm
 m_qplistinit:start
  --- Readin syml file --- 
 end of reading rst or atm
 end of reading rst or atm
   41   0.5000   0.5000   0.5000    0.0000   0.0000   0.0000 L Gamma
   41   0.0000   0.0000   0.0000    1.0000   0.0000   0.0000 Gamma X
   21   1.0000   0.0000   0.0000    1.0000   0.5000   0.0000 X W
   41   1.0000   0.5000   0.0000    0.0000   0.0000   0.0000 W Gamma
nsyml nkp=    4  144
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    3, (  109 1),  (  144 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    2, (   73 1),  (  108 1)
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    1, (   37 1),  (   72 1)
 -------- qplist --------           4
    1   0.500   0.500   0.500  <-- isyml= 001
    2   0.487   0.487   0.487 
    3   0.475   0.475   0.475 
    4   0.463   0.463   0.463 
    5   0.450   0.450   0.450 
    6   0.438   0.438   0.438 
    7   0.425   0.425   0.425 
    8   0.412   0.412   0.412 
    9   0.400   0.400   0.400 
   10   0.388   0.388   0.388 
   11   0.375   0.375   0.375 
   12   0.362   0.362   0.362 
   13   0.350   0.350   0.350 
   14   0.338   0.338   0.338 
   15   0.325   0.325   0.325 
   16   0.312   0.312   0.312 
   17   0.300   0.300   0.300 
   18   0.287   0.287   0.287 
   19   0.275   0.275   0.275 
   20   0.263   0.263   0.263 
   21   0.250   0.250   0.250 
   22   0.237   0.237   0.237 
   23   0.225   0.225   0.225 
   24   0.213   0.213   0.213 
   25   0.200   0.200   0.200 
   26   0.188   0.188   0.188 
   27   0.175   0.175   0.175 
   28   0.162   0.162   0.162 
   29   0.150   0.150   0.150 
   30   0.138   0.138   0.138 
   31   0.125   0.125   0.125 
   32   0.112   0.112   0.112 
   33   0.100   0.100   0.100 
   34   0.088   0.088   0.088 
   35   0.075   0.075   0.075 
   36   0.062   0.062   0.062 
   37   0.050   0.050   0.050 
   38   0.037   0.037   0.037 
   39   0.025   0.025   0.025 
   40   0.013   0.013   0.013 
   41   0.000   0.000   0.000 
   42   0.000   0.000   0.000  <-- isyml= 002
   43   0.025   0.000   0.000 
   44   0.050   0.000   0.000 
   45   0.075   0.000   0.000 
   46   0.100   0.000   0.000 
   47   0.125   0.000   0.000 
   48   0.150   0.000   0.000 
   49   0.175   0.000   0.000 
   50   0.200   0.000   0.000 
   51   0.225   0.000   0.000 
   52   0.250   0.000   0.000 
   53   0.275   0.000   0.000 
   54   0.300   0.000   0.000 
   55   0.325   0.000   0.000 
   56   0.350   0.000   0.000 
   57   0.375   0.000   0.000 
   58   0.400   0.000   0.000 
   59   0.425   0.000   0.000 
   60   0.450   0.000   0.000 
   61   0.475   0.000   0.000 
   62   0.500   0.000   0.000 
   63   0.525   0.000   0.000 
   64   0.550   0.000   0.000 
   65   0.575   0.000   0.000 
   66   0.600   0.000   0.000 
   67   0.625   0.000   0.000 
   68   0.650   0.000   0.000 
   69   0.675   0.000   0.000 
   70   0.700   0.000   0.000 
   71   0.725   0.000   0.000 
   72   0.750   0.000   0.000 
   73   0.775   0.000   0.000 
   74   0.800   0.000   0.000 
   75   0.825   0.000   0.000 
   76   0.850   0.000   0.000 
   77   0.875   0.000   0.000 
   78   0.900   0.000   0.000 
   79   0.925   0.000   0.000 
   80   0.950   0.000   0.000 
   81   0.975   0.000   0.000 
   82   1.000   0.000   0.000 
   83   1.000   0.000   0.000  <-- isyml= 003
   84   1.000   0.025   0.000 
   85   1.000   0.050   0.000 
   86   1.000   0.075   0.000 
   87   1.000   0.100   0.000 
   88   1.000   0.125   0.000 
   89   1.000   0.150   0.000 
   90   1.000   0.175   0.000 
   91   1.000   0.200   0.000 
   92   1.000   0.225   0.000 
   93   1.000   0.250   0.000 
   94   1.000   0.275   0.000 
   95   1.000   0.300   0.000 
   96   1.000   0.325   0.000 
   97   1.000   0.350   0.000 
   98   1.000   0.375   0.000 
   99   1.000   0.400   0.000 
  100   1.000   0.425   0.000 
  101   1.000   0.450   0.000 
  102   1.000   0.475   0.000 
  103   1.000   0.500   0.000 
  104   1.000   0.500   0.000  <-- isyml= 004
  105   0.975   0.487   0.000 
  106   0.950   0.475   0.000 
  107   0.925   0.463   0.000 
  108   0.900   0.450   0.000 
  109   0.875   0.438   0.000 
  110   0.850   0.425   0.000 
  111   0.825   0.412   0.000 
  112   0.800   0.400   0.000 
  113   0.775   0.388   0.000 
  114   0.750   0.375   0.000 
  115   0.725   0.362   0.000 
  116   0.700   0.350   0.000 
  117   0.675   0.338   0.000 
  118   0.650   0.325   0.000 
  119   0.625   0.312   0.000 
  120   0.600   0.300   0.000 
  121   0.575   0.287   0.000 
  122   0.550   0.275   0.000 
  123   0.525   0.263   0.000 
  124   0.500   0.250   0.000 
  125   0.475   0.237   0.000 
  126   0.450   0.225   0.000 
  127   0.425   0.213   0.000 
  128   0.400   0.200   0.000 
  129   0.375   0.188   0.000 
  130   0.350   0.175   0.000 
  131   0.325   0.162   0.000 
  132   0.300   0.150   0.000 
  133   0.275   0.138   0.000 
  134   0.250   0.125   0.000 
  135   0.225   0.112   0.000 
  136   0.200   0.100   0.000 
  137   0.175   0.088   0.000 
  138   0.150   0.075   0.000 
  139   0.125   0.062   0.000 
  140   0.100   0.050   0.000 
  141   0.075   0.037   0.000 
  142   0.050   0.025   0.000 
  143   0.025   0.013   0.000 
  144   0.000   0.000   0.000 
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)=    0, (    1 1),  (   36 1)

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1  A         0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

 --- BNDFP:  begin iteration 1 of 1
 esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves:: avg es pot at rmt= 0.580873  avg sphere pot= 0.653065  vconst=-0.580873

 smooth rhoves     10.095981   charge     3.702445
 smvxcm: all smrho_w is positive
 smooth rhoeps =   -2.655321   rhomu =   -3.455211  avg vxc =   -0.832805 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.001387      -184.375221      -190.376608
   rhoval*ves            -46.598836      -116.654383      -163.253219
   psnuc*ves              66.790798    -12971.812428    -12905.021630
   utot                   10.095981     -6544.233406     -6534.137424
   rho*exc                -2.655321      -127.279285      -129.934607
   rho*vxc                -3.455211      -168.536686      -171.991897
   valence chg             3.702445         7.297555        11.000000
   core charge            18.000000        -0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:     -0.00000
  m_bandcal_init: start
 bndfp: kpt     1 of   144 k=  0.5000  0.5000  0.5000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of   144 k=  0.4875  0.4875  0.4875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of   144 k=  0.4750  0.4750  0.4750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of   144 k=  0.4625  0.4625  0.4625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of   144 k=  0.4500  0.4500  0.4500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of   144 k=  0.4375  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of   144 k=  0.4250  0.4250  0.4250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of   144 k=  0.4125  0.4125  0.4125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of   144 k=  0.4000  0.4000  0.4000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of   144 k=  0.3875  0.3875  0.3875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of   144 k=  0.3750  0.3750  0.3750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of   144 k=  0.3625  0.3625  0.3625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of   144 k=  0.3500  0.3500  0.3500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of   144 k=  0.3375  0.3375  0.3375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of   144 k=  0.3250  0.3250  0.3250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    16 of   144 k=  0.3125  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    17 of   144 k=  0.3000  0.3000  0.3000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    18 of   144 k=  0.2875  0.2875  0.2875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    19 of   144 k=  0.2750  0.2750  0.2750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    20 of   144 k=  0.2625  0.2625  0.2625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    21 of   144 k=  0.2500  0.2500  0.2500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    22 of   144 k=  0.2375  0.2375  0.2375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    23 of   144 k=  0.2250  0.2250  0.2250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    24 of   144 k=  0.2125  0.2125  0.2125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    25 of   144 k=  0.2000  0.2000  0.2000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    26 of   144 k=  0.1875  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    27 of   144 k=  0.1750  0.1750  0.1750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    28 of   144 k=  0.1625  0.1625  0.1625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    29 of   144 k=  0.1500  0.1500  0.1500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    30 of   144 k=  0.1375  0.1375  0.1375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    31 of   144 k=  0.1250  0.1250  0.1250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    32 of   144 k=  0.1125  0.1125  0.1125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    33 of   144 k=  0.1000  0.1000  0.1000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    34 of   144 k=  0.0875  0.0875  0.0875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    35 of   144 k=  0.0750  0.0750  0.0750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    36 of   144 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.2732
  Writing bands to bands file ...
ikpoff=    2   41
ikpoff=    3   82
ikpoff=    4  103
ikpoff=    5  144
 bndfp: kpt    1 of  144 k jsp=  0.50000  0.50000  0.50000 1 nev=   31
 bndfp: kpt    2 of  144 k jsp=  0.48750  0.48750  0.48750 1 nev=   31
 bndfp: kpt    3 of  144 k jsp=  0.47500  0.47500  0.47500 1 nev=   31
 bndfp: kpt    4 of  144 k jsp=  0.46250  0.46250  0.46250 1 nev=   31
 bndfp: kpt    5 of  144 k jsp=  0.45000  0.45000  0.45000 1 nev=   31
 bndfp: kpt    6 of  144 k jsp=  0.43750  0.43750  0.43750 1 nev=   31
 bndfp: kpt    7 of  144 k jsp=  0.42500  0.42500  0.42500 1 nev=   31
 bndfp: kpt    8 of  144 k jsp=  0.41250  0.41250  0.41250 1 nev=   31
 bndfp: kpt    9 of  144 k jsp=  0.40000  0.40000  0.40000 1 nev=   31
 bndfp: kpt   10 of  144 k jsp=  0.38750  0.38750  0.38750 1 nev=   31
 bndfp: kpt   11 of  144 k jsp=  0.37500  0.37500  0.37500 1 nev=   31
 bndfp: kpt   12 of  144 k jsp=  0.36250  0.36250  0.36250 1 nev=   31
 bndfp: kpt   13 of  144 k jsp=  0.35000  0.35000  0.35000 1 nev=   31
 bndfp: kpt   14 of  144 k jsp=  0.33750  0.33750  0.33750 1 nev=   31
 bndfp: kpt   15 of  144 k jsp=  0.32500  0.32500  0.32500 1 nev=   31
 bndfp: kpt   16 of  144 k jsp=  0.31250  0.31250  0.31250 1 nev=   31
 bndfp: kpt   17 of  144 k jsp=  0.30000  0.30000  0.30000 1 nev=   31
 bndfp: kpt   18 of  144 k jsp=  0.28750  0.28750  0.28750 1 nev=   31
 bndfp: kpt   19 of  144 k jsp=  0.27500  0.27500  0.27500 1 nev=   31
 bndfp: kpt   20 of  144 k jsp=  0.26250  0.26250  0.26250 1 nev=   31
 bndfp: kpt   21 of  144 k jsp=  0.25000  0.25000  0.25000 1 nev=   31
 bndfp: kpt   22 of  144 k jsp=  0.23750  0.23750  0.23750 1 nev=   31
 bndfp: kpt   23 of  144 k jsp=  0.22500  0.22500  0.22500 1 nev=   31
 bndfp: kpt   24 of  144 k jsp=  0.21250  0.21250  0.21250 1 nev=   31
 bndfp: kpt   25 of  144 k jsp=  0.20000  0.20000  0.20000 1 nev=   31
 bndfp: kpt   26 of  144 k jsp=  0.18750  0.18750  0.18750 1 nev=   31
 bndfp: kpt   27 of  144 k jsp=  0.17500  0.17500  0.17500 1 nev=   31
 bndfp: kpt   28 of  144 k jsp=  0.16250  0.16250  0.16250 1 nev=   31
 bndfp: kpt   29 of  144 k jsp=  0.15000  0.15000  0.15000 1 nev=   31
 bndfp: kpt   30 of  144 k jsp=  0.13750  0.13750  0.13750 1 nev=   31
 bndfp: kpt   31 of  144 k jsp=  0.12500  0.12500  0.12500 1 nev=   31
 bndfp: kpt   32 of  144 k jsp=  0.11250  0.11250  0.11250 1 nev=   31
 bndfp: kpt   33 of  144 k jsp=  0.10000  0.10000  0.10000 1 nev=   31
 bndfp: kpt   34 of  144 k jsp=  0.08750  0.08750  0.08750 1 nev=   31
 bndfp: kpt   35 of  144 k jsp=  0.07500  0.07500  0.07500 1 nev=   31
 bndfp: kpt   36 of  144 k jsp=  0.06250  0.06250  0.06250 1 nev=   31
 bndfp: kpt   37 of  144 k jsp=  0.05000  0.05000  0.05000 1 nev=   31
 bndfp: kpt   38 of  144 k jsp=  0.03750  0.03750  0.03750 1 nev=   31
 bndfp: kpt   39 of  144 k jsp=  0.02500  0.02500  0.02500 1 nev=   31
 bndfp: kpt   40 of  144 k jsp=  0.01250  0.01250  0.01250 1 nev=   31
 bndfp: kpt   41 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   42 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   43 of  144 k jsp=  0.02500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   44 of  144 k jsp=  0.05000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   45 of  144 k jsp=  0.07500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   46 of  144 k jsp=  0.10000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   47 of  144 k jsp=  0.12500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   48 of  144 k jsp=  0.15000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   49 of  144 k jsp=  0.17500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   50 of  144 k jsp=  0.20000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   51 of  144 k jsp=  0.22500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   52 of  144 k jsp=  0.25000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   53 of  144 k jsp=  0.27500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   54 of  144 k jsp=  0.30000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   55 of  144 k jsp=  0.32500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   56 of  144 k jsp=  0.35000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   57 of  144 k jsp=  0.37500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   58 of  144 k jsp=  0.40000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   59 of  144 k jsp=  0.42500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   60 of  144 k jsp=  0.45000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   61 of  144 k jsp=  0.47500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   62 of  144 k jsp=  0.50000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   63 of  144 k jsp=  0.52500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   64 of  144 k jsp=  0.55000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   65 of  144 k jsp=  0.57500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   66 of  144 k jsp=  0.60000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   67 of  144 k jsp=  0.62500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   68 of  144 k jsp=  0.65000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   69 of  144 k jsp=  0.67500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   70 of  144 k jsp=  0.70000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   71 of  144 k jsp=  0.72500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   72 of  144 k jsp=  0.75000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   73 of  144 k jsp=  0.77500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   74 of  144 k jsp=  0.80000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   75 of  144 k jsp=  0.82500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   76 of  144 k jsp=  0.85000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   77 of  144 k jsp=  0.87500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   78 of  144 k jsp=  0.90000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   79 of  144 k jsp=  0.92500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   80 of  144 k jsp=  0.95000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   81 of  144 k jsp=  0.97500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   82 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   83 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   84 of  144 k jsp=  1.00000  0.02500  0.00000 1 nev=   31
 bndfp: kpt   85 of  144 k jsp=  1.00000  0.05000  0.00000 1 nev=   31
 bndfp: kpt   86 of  144 k jsp=  1.00000  0.07500  0.00000 1 nev=   31
 bndfp: kpt   87 of  144 k jsp=  1.00000  0.10000  0.00000 1 nev=   31
 bndfp: kpt   88 of  144 k jsp=  1.00000  0.12500  0.00000 1 nev=   31
 bndfp: kpt   89 of  144 k jsp=  1.00000  0.15000  0.00000 1 nev=   31
 bndfp: kpt   90 of  144 k jsp=  1.00000  0.17500  0.00000 1 nev=   31
 bndfp: kpt   91 of  144 k jsp=  1.00000  0.20000  0.00000 1 nev=   31
 bndfp: kpt   92 of  144 k jsp=  1.00000  0.22500  0.00000 1 nev=   31
 bndfp: kpt   93 of  144 k jsp=  1.00000  0.25000  0.00000 1 nev=   31
 bndfp: kpt   94 of  144 k jsp=  1.00000  0.27500  0.00000 1 nev=   31
 bndfp: kpt   95 of  144 k jsp=  1.00000  0.30000  0.00000 1 nev=   31
 bndfp: kpt   96 of  144 k jsp=  1.00000  0.32500  0.00000 1 nev=   31
 bndfp: kpt   97 of  144 k jsp=  1.00000  0.35000  0.00000 1 nev=   31
 bndfp: kpt   98 of  144 k jsp=  1.00000  0.37500  0.00000 1 nev=   31
 bndfp: kpt   99 of  144 k jsp=  1.00000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  100 of  144 k jsp=  1.00000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  101 of  144 k jsp=  1.00000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  102 of  144 k jsp=  1.00000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  103 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  104 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  105 of  144 k jsp=  0.97500  0.48750  0.00000 1 nev=   31
 bndfp: kpt  106 of  144 k jsp=  0.95000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  107 of  144 k jsp=  0.92500  0.46250  0.00000 1 nev=   31
 bndfp: kpt  108 of  144 k jsp=  0.90000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  109 of  144 k jsp=  0.87500  0.43750  0.00000 1 nev=   31
 bndfp: kpt  110 of  144 k jsp=  0.85000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  111 of  144 k jsp=  0.82500  0.41250  0.00000 1 nev=   31
 bndfp: kpt  112 of  144 k jsp=  0.80000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  113 of  144 k jsp=  0.77500  0.38750  0.00000 1 nev=   31
 bndfp: kpt  114 of  144 k jsp=  0.75000  0.37500  0.00000 1 nev=   31
 bndfp: kpt  115 of  144 k jsp=  0.72500  0.36250  0.00000 1 nev=   31
 bndfp: kpt  116 of  144 k jsp=  0.70000  0.35000  0.00000 1 nev=   31
 bndfp: kpt  117 of  144 k jsp=  0.67500  0.33750  0.00000 1 nev=   31
 bndfp: kpt  118 of  144 k jsp=  0.65000  0.32500  0.00000 1 nev=   31
 bndfp: kpt  119 of  144 k jsp=  0.62500  0.31250  0.00000 1 nev=   31
 bndfp: kpt  120 of  144 k jsp=  0.60000  0.30000  0.00000 1 nev=   31
 bndfp: kpt  121 of  144 k jsp=  0.57500  0.28750  0.00000 1 nev=   31
 bndfp: kpt  122 of  144 k jsp=  0.55000  0.27500  0.00000 1 nev=   31
 bndfp: kpt  123 of  144 k jsp=  0.52500  0.26250  0.00000 1 nev=   31
 bndfp: kpt  124 of  144 k jsp=  0.50000  0.25000  0.00000 1 nev=   31
 bndfp: kpt  125 of  144 k jsp=  0.47500  0.23750  0.00000 1 nev=   31
 bndfp: kpt  126 of  144 k jsp=  0.45000  0.22500  0.00000 1 nev=   31
 bndfp: kpt  127 of  144 k jsp=  0.42500  0.21250  0.00000 1 nev=   31
 bndfp: kpt  128 of  144 k jsp=  0.40000  0.20000  0.00000 1 nev=   31
 bndfp: kpt  129 of  144 k jsp=  0.37500  0.18750  0.00000 1 nev=   31
 bndfp: kpt  130 of  144 k jsp=  0.35000  0.17500  0.00000 1 nev=   31
 bndfp: kpt  131 of  144 k jsp=  0.32500  0.16250  0.00000 1 nev=   31
 bndfp: kpt  132 of  144 k jsp=  0.30000  0.15000  0.00000 1 nev=   31
 bndfp: kpt  133 of  144 k jsp=  0.27500  0.13750  0.00000 1 nev=   31
 bndfp: kpt  134 of  144 k jsp=  0.25000  0.12500  0.00000 1 nev=   31
 bndfp: kpt  135 of  144 k jsp=  0.22500  0.11250  0.00000 1 nev=   31
 bndfp: kpt  136 of  144 k jsp=  0.20000  0.10000  0.00000 1 nev=   31
 bndfp: kpt  137 of  144 k jsp=  0.17500  0.08750  0.00000 1 nev=   31
 bndfp: kpt  138 of  144 k jsp=  0.15000  0.07500  0.00000 1 nev=   31
 bndfp: kpt  139 of  144 k jsp=  0.12500  0.06250  0.00000 1 nev=   31
 bndfp: kpt  140 of  144 k jsp=  0.10000  0.05000  0.00000 1 nev=   31
 bndfp: kpt  141 of  144 k jsp=  0.07500  0.03750  0.00000 1 nev=   31
 bndfp: kpt  142 of  144 k jsp=  0.05000  0.02500  0.00000 1 nev=   31
 bndfp: kpt  143 of  144 k jsp=  0.02500  0.01250  0.00000 1 nev=   31
 bndfp: kpt  144 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
Exit 0 plot band mode done
 CPU time:    0.423s     Fri Oct  8 19:20:38 2021   on 

  ==== xxxxxxxxx ====     calls      == cpu time ===   depth 1
  entry   xxxx  xxxx                per call  total  (depth is by TIM= in ctrl.*.)
      0      0      0        1       0.42       0.42   main
      0      0    -10        0       0.00       0.00   `--lmfp
