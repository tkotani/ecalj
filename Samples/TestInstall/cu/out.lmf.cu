#include "hip/hip_runtime.h"
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.2) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit c35df491f292381d706d43a93760b974f67f0241
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Sep 19 14:51:50 2023 +0900
INFO: linked at Tue Sep 19 17:03:40 JST 2023
=== START LFMA ===
 mpisize=           1
m_lmfinit: LMFA
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8<ctrl.cu >ctrlp.cu
 end of readctrlp
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_READPSKIPF          defa n= 1 val= 1.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_ATOM@1             val=  A
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HOLE@1           val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_ATOM@1             val=  A
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: SYMGRP                  val= 
rval2: SYMGRPAF                val= 
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_MIX                val=  A3
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing param: A/B nmix wt= 0 3 1.000000  1.000000 beta wc killj=  1.000000 -1.000000 -1
 >> level: 1  CPUsec=      0.00  enter m_lmfinit
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  2
 >>      0.01   exit  m_lmfinit       0.01
freats:

conf:------------------------------------------------------
conf:SPEC_ATOM= A : --- Table for atomic configuration ---
conf:  isp  l  int(P) int(P)z    Qval     Qcore   CoreConf
conf:    1  0       4  5         1.000    6.000 => 1,2,3,
conf:    1  1       4  5         0.000   12.000 => 2,3,
conf:    1  2       3  4        10.000    0.000 => 
conf:    1  3       4  0         0.000    0.000 => 
usedQ=     1.000     0.000    10.000     0.000
conf: Species  A        Z=  29.00 Qc=  18.000 R=  2.311271 Q=  0.000000 nsp= 1 mom=  0.000000
conf: rmt rmax a=  2.311271  48.805862  0.025000 nrmt nr= 393 515
 goto atomc xxx
 atomsc nmcore=           0

 end of atomsc xxxxx
 vsum=  -130.79144076069792                1
sumev= -4.333254 etot= -3304.416258 eref=  0.000000 etot-eref= -3304.416258

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)       pnu
   4s      -0.36411         0.890       2.256       3.582     0.643062       4.761  0
   5s      -0.00028         3.669      10.794      19.873     0.990448       5.848  1
   4p      -0.06295         0.975       3.484       7.414     0.901829       4.561  0
   5p       0.00796         6.760      30.414      48.806*    0.999240       5.593  1
   3d      -0.39691         0.000       0.600       3.429     0.056076       3.888  0
   4d       0.01308         1.868      33.290      48.806*    0.999995       4.148  1
   4f       0.01948         0.000      35.393      48.806*    1.000000       4.137  0

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000141
   3p      -5.29682         0.260       0.619       1.078     0.000727
 tailsm: init

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.31127, rsm= 1.15564
  ---E:energies of smHankels. C:fitting coeeficient for core tail. ---
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000   -15.00000
 C:    -0.07160    10.75053  -187.49213  1222.02349 -4717.78530 21166.80769
        r          rho         fit         diff
    2.311271    0.017797    0.017766    0.000031
    2.967767    0.005662    0.005658    0.000005
    3.810725    0.001517    0.001518   -0.000001
    4.893104    0.000305    0.000305   -0.000000
    6.282906    0.000041    0.000041   -0.000001
    8.067448    0.000003    0.000003    0.000000
    q(fit):     1.203836    rms diff:   0.000016
    fit: r>rmt  1.203836   r<rmt  3.442816   qtot  4.646652
    rho: r>rmt  1.203836   r<rmt  9.796164   qtot 11.000000
 tailsm:  fit tails to        6 functions with

 rsm=  0.11556D+01 rms error=  0.16285D-04
conf: Core rhoc(rmt)= 0.003922 spillout= 0.004646
 Fit with Hankel e=-24.082483420971577 coeff=764.35251301008668
      r            rhoc          fit
    2.311271    0.02095279    0.02095279
    2.429779    0.01229068    0.01231367
    2.753317    0.00285262    0.00285190
    3.119934    0.00054243    0.00053465
    3.535366    0.00008235    0.00007888
    4.006112    0.00000969    0.00000887
    4.539536    0.00000085    0.00000073
    5.143985    0.00000005    0.00000004
 end of freats: spid nmcore=A                  0
Sum of reference energies:                      0.000000000000
CPU time:    0.039s     Tue Sep 19 17:16:35 2023 on process=0

  ==== procid=0 ====     calls      == cpu time ===   depth 1
  entry   xxxx  xxxx                per call  total  (depth is by TIM= in ctrl.*.)
      0      0      0        1       0.04       0.04   main
      0      0      0        1       0.01       0.01   `--m_lmfinit
Exit 0 procid= 0 OK! end of LMFA ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.2) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit c35df491f292381d706d43a93760b974f67f0241
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Sep 19 14:51:50 2023 +0900
INFO: linked at Tue Sep 19 17:03:40 JST 2023
===START LMF with   --no-iactiv cu -vnk=8 -vbigbas=f ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=f<ctrl.cu >ctrlp.cu
 end of readctrlp
 end of readctrlp
 end of readctrlp
 end of readctrlp
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_READPSKIPF          defa n= 1 val= 1.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_ATOM@1             val=  A
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  0.00000000  0.00000000
rval2: SPEC_EH2@1              requ n= 0 val= 
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 3.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 3.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HOLE@1           val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_ATOM@1             val=  A
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: SYMGRP                  val= 
rval2: SYMGRPAF                val= 
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_MIX                val=  A3
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing param: A/B nmix wt= 0 3 1.000000  1.000000 beta wc killj=  1.000000 -1.000000 -1
 >> level: 1  CPUsec=      0.00  enter m_lmfinit
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    2    1    1    1
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  2
 >>      0.01   exit  m_lmfinit       0.01
 >> level: 1  CPUsec=      0.01  enter m_lattic_init

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
 SYMGRP = find
 SYMGRP = find
 SYMGRP = find
  Cell vol=   78.538660

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkq= 331
 >>      0.01   exit  m_lattic_ini    0.00
 >> level: 1  CPUsec=      0.01  enter m_mksym_init
SpaceGroupSym of Lattice: ========start========================== 
 SYMGRP = find
  Generators except find: 
 sgroup:  1 symmetry operations from 0 generators
 symlat: Bravais system is cubic        with 48 symmetry operations.
 symcry: crystal invariant under 48 following symmetry operations for tol=  0.000100
   Enlarging ngen= 1  ng nggen= 48 6
   Enlarging ngen= 2  ng nggen= 48 48
 groupg: the following are sufficient to generate the space group:
  Generators:  trans(cart)= i*r3(1,1,-1) r4x
  Generators:: trans(frac)= i*r3(1,1,-1) r4x
 gensym: ig group ops (:vector means translation in cartesian)
    1  e
    2  i*r3(1,1,-1)
    3  r3(-1,-1,1)
    4  i
    5  r3(1,1,-1)
    6  i*r3(-1,-1,1)
    7  r4x
    8  m(1,0,-1)
    9  r4z
   10  i*r4x
   11  r2(1,0,-1)
   12  i*r4z
   13  r2x
   14  i*r3(-1,1,1)
   15  r3d
   16  mx
   17  r3(-1,1,1)
   18  i*r3d
   19  r4(-1,0,0)
   20  i*r4y
   21  r2(1,1,0)
   22  i*r4(-1,0,0)
   23  r4y
   24  m(1,1,0)
   25  i*r3(-1,-1,-1)
   26  r3(-1,1,-1)
   27  my
   28  r3(-1,-1,-1)
   29  i*r3(-1,1,-1)
   30  r2y
   31  i*r4(0,0,-1)
   32  r2(0,1,-1)
   33  i*r4(0,-1,0)
   34  r4(0,0,-1)
   35  m(0,1,-1)
   36  r4(0,-1,0)
   37  r3(1,-1,-1)
   38  mz
   39  r3(1,-1,1)
   40  i*r3(1,-1,-1)
   41  r2z
   42  i*r3(1,-1,1)
   43  r2(1,0,1)
   44  m(1,0,1)
   45  m(0,1,1)
   46  r2(0,1,1)
   47  r2(1,-1,0)
   48  m(1,-1,0)
 gensym: site permutation table for group operations ...
  ib/ig:  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 23 24 25 26 27 28 29 30 31 32 33 34 35 36 37 38 39 40 41 42 43 44 45 46 47 48
      1:  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1
 splcls:  ibas iclass ispec label(ispec)
            1     1     1     A
SpaceGroupSym of Lattice: ========end =========================== 

 >>      0.46   exit  m_mksym_init    0.45
 >> level: 1  CPUsec=      0.46  enter m_mkqp_init
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
 >>      0.46   exit  m_mkqp_init     0.00
 >> level: 1  CPUsec=      0.46  enter m_supot_init
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors
 >>      0.46   exit  m_supot_init    0.01

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
 >> level: 1  CPUsec=      0.46  enter m_suham_init
 >>      0.46   exit  m_suham_init    0.00
 >> level: 1  CPUsec=      0.46  enter m_qplist_init
 m_qplistinit:start
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 2 31 1 45 1
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 1 16 1 30 1
 >>      0.47   exit  m_qplist_ini    0.00
 >> level: 1  CPUsec=      0.47  enter m_qplist_qpsdivider
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 0 1 1 15 1
 >>      0.47   exit  m_qplist_qps    0.00
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 3 46 1 60 1
 >> level: 1  CPUsec=      0.47  enter m_igv2xall_init
 >>      0.47   exit  m_igv2xall_i    0.00
 >> level: 1  CPUsec=      0.47  enter lmfp

 iors  : read rst restart file (binary mesh density)
 iors  : empty file ... nothing read

rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected A,       read A        with rmt=  2.3113  mesh   393  0.025
  ovlpfa: overlap smooth part of FA densities
 site 1 spec 1 pos 0.0000  0.0000  0.0000 Qsmooth 4.6466523386126539 mom 4.6466523386126539
 total smooth Q =  4.6466523386126539

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.796164    3.442816   10.275300    3.921952    6.353348

 Smooth charge on mesh:            4.646652
 Sum of local charges:             6.353348
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:       -0.000000

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.555104
   smooth rhoves     11.022231   charge     4.646652
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.843799 -5.010453 -0.851784
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:   -0.000099
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff            -12.157495      -177.337532      -189.495027
   rhoval*ves            -46.690633      -115.324376      -162.015010
   psnuc*ves              68.735095    -12976.662436    -12907.927341
   Eestatic               11.022231     -6545.993406     -6534.971175
   rho*exc                -3.843799      -126.414298      -130.258096
   rho*vxc                -5.010453      -167.409316      -172.419769
   valence chg             4.646652         6.353348        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0590

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144466;  11.000000 electrons
         Sum occ. bands:   -0.854464, incl. Bloechl correction: -0.006586
 bndfp:Generating TDOS: efermi=  0.144466  dos window emin emax=  -0.672855  3.084388

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1    9.927753    3.113495    6.814258
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -0.854464 Vin*nin=    -189.495027 Ek=Eb-Vin*nin=     188.640563
 Ek(core)=    3171.756639 Exc=    -130.258096 Ees=   -6534.971175 Eharris=   -3304.832069

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.361299   -168.223685   -174.584984
    E_B(band energy sum)=   -0.854464  E_B-nout*Vin=  173.730520

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178921   charge     4.185742
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054119 -3.974965 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030713      -175.114090      -182.144802
   rhoval*ves            -50.183967      -106.215152      -156.399120
   psnuc*ves              76.541810    -12962.871083    -12886.329273
   Eestatic               13.178921     -6534.543118     -6521.364196
   rho*exc                -3.054119      -125.587138      -128.641257
   rho*vxc                -3.974965      -166.302310      -170.277275
   valence chg             4.185742         6.814258        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.730520 Ekcore=      3171.756639 Ektot    =     3345.487160
 Exc=    -128.641257 Ees   =     -6521.364196 EKohnSham=    -3304.518294
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 0 RMS DQ= 4.34E-2
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.34D-02
 mixrho: add corrections to qcell smrho = -0.23414D-07 -0.29812D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.832069   ehk=   -3304.518294
h ehf(eV)=-44964.884161 ehk(eV)=-44960.615005 sev(eV)=-11.625666

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.677372
   smooth rhoves     13.178921   charge     4.185742
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.054119 -3.974965 -0.866699
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000553
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -7.030713      -175.114090      -182.144802
   rhoval*ves            -50.183967      -106.215152      -156.399120
   psnuc*ves              76.541810    -12962.871083    -12886.329273
   Eestatic               13.178921     -6534.543118     -6521.364196
   rho*exc                -3.054119      -125.587138      -128.641257
   rho*vxc                -3.974965      -166.302310      -170.277275
   valence chg             4.185742         6.814258        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0628

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.257389;  11.000000 electrons
         Sum occ. bands:   -9.368175, incl. Bloechl correction: -0.013546
 bndfp:Generating TDOS: efermi= -0.257389  dos window emin emax=  -0.960038  2.682534

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.456129    1.689131    8.766998
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -9.368175 Vin*nin=    -182.144802 Ek=Eb-Vin*nin=     172.776627
 Ek(core)=    3171.756639 Exc=    -128.641257 Ees=   -6521.364196 Eharris=   -3305.472187

 mkekin:
   nout*Vin = smpart,onsite,total=:     -3.922792   -233.404202   -237.326995
    E_B(band energy sum)=   -9.368175  E_B-nout*Vin=  227.958820

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.358666
   smooth rhoves      3.884961   charge     2.233002
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.460402 -1.898029 -0.723071
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -2.689751      -210.699261      -213.389012
   rhoval*ves            -33.640133      -148.867620      -182.507754
   psnuc*ves              41.410055    -12997.787102    -12956.377047
   Eestatic                3.884961     -6573.327361     -6569.442400
   rho*exc                -1.460402      -131.953517      -133.413919
   rho*vxc                -1.898029      -174.707795      -176.605824
   valence chg             2.233002         8.766998        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      227.958820 Ekcore=      3171.756639 Ektot    =     3399.715459
 Exc=    -133.413919 Ees   =     -6569.442400 EKohnSham=    -3303.140860
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 1 RMS DQ= 1.23E-1  last it= 4.34E-2
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.23D-01
   tj: 0.82067
 mixrho: add corrections to qcell smrho = -0.95446D-08 -0.12153D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3305.472187   ehk=   -3303.140860
 From last iter    ehf=   -3304.832069   ehk=   -3304.518294
 diffe(q)= -0.640118 (0.123259)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44973.593478 ehk(eV)=-44941.873915 sev(eV)=-127.461513

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.620217
   smooth rhoves     11.106636   charge     3.835548
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.748440 -3.576345 -0.844917
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000503
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.129116      -182.318056      -188.447172
   rhoval*ves            -48.028198      -113.797128      -161.825326
   psnuc*ves              70.241470    -12969.132735    -12898.891265
   Eestatic               11.106636     -6541.464931     -6530.358295
   rho*exc                -2.748440      -126.719600      -129.468040
   rho*vxc                -3.576345      -167.797339      -171.373685
   valence chg             3.835548         7.164452        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0664

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.147954;  11.000000 electrons
         Sum occ. bands:   -5.246328, incl. Bloechl correction: -0.012263
 bndfp:Generating TDOS: efermi= -0.147954  dos window emin emax=  -0.773958  2.791969

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.295260    2.201485    8.093775
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -5.246328 Vin*nin=    -188.447172 Ek=Eb-Vin*nin=     183.200844
 Ek(core)=    3171.756639 Exc=    -129.468040 Ees=   -6530.358295 Eharris=   -3304.868852

 mkekin:
   nout*Vin = smpart,onsite,total=:     -4.778835   -205.206269   -209.985104
    E_B(band energy sum)=   -5.246328  E_B-nout*Vin=  204.738776

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.464060
   smooth rhoves      6.431640   charge     2.906225
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -1.983202 -2.579009 -0.778515
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.017967      -196.242856      -200.260823
   rhoval*ves            -40.458944      -131.023884      -171.482828
   psnuc*ves              53.322223    -12980.758093    -12927.435870
   Eestatic                6.431640     -6555.890989     -6549.459349
   rho*exc                -1.983202      -129.491101      -131.474303
   rho*vxc                -2.579009      -171.454410      -174.033419
   valence chg             2.906225         8.093775        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      204.738776 Ekcore=      3171.756639 Ektot    =     3376.495416
 Exc=    -131.474303 Ees   =     -6549.459349 EKohnSham=    -3304.438236
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 2 RMS DQ= 5.10E-2  last it= 1.23E-1
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 5.10D-02
   tj:-0.76356  -0.08459
 mixrho: add corrections to qcell smrho = -0.31774D-06 -0.40457D-08

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.868852   ehk=   -3304.438236
 From last iter    ehf=   -3305.472187   ehk=   -3303.140860
 diffe(q)=  0.603335 (0.050979)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44965.384622 ehk(eV)=-44959.525753 sev(eV)=-71.380490

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.526491
   smooth rhoves      8.265620   charge     3.312034
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.317478 -3.014677 -0.807663
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000460
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -4.927924      -185.839813      -190.767736
   rhoval*ves            -43.922458      -119.352380      -163.274838
   psnuc*ves              60.453697    -12969.268553    -12908.814856
   Eestatic                8.265620     -6544.310467     -6536.044847
   rho*exc                -2.317478      -127.956831      -130.274309
   rho*vxc                -3.014677      -169.427033      -172.441710
   valence chg             3.312034         7.687966        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0674

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.128604;  11.000000 electrons
         Sum occ. bands:   -1.033470, incl. Bloechl correction: -0.006675
 bndfp:Generating TDOS: efermi=  0.128604  dos window emin emax=  -0.685112  3.068526

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1    9.933588    3.103760    6.829828
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -1.033470 Vin*nin=    -190.767736 Ek=Eb-Vin*nin=     189.734267
 Ek(core)=    3171.756639 Exc=    -130.274309 Ees=   -6536.044847 Eharris=   -3304.828250

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.025684   -168.674550   -174.700234
    E_B(band energy sum)=   -1.033470  E_B-nout*Vin=  173.666764

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.674508
   smooth rhoves     13.078737   charge     4.170172
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -3.040794 -3.957592 -0.865692
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.992418      -175.025870      -182.018287
   rhoval*ves            -50.088098      -106.176242      -156.264340
   psnuc*ves              76.245572    -12962.576619    -12886.331047
   Eestatic               13.078737     -6534.376430     -6521.297693
   rho*exc                -3.040794      -125.607826      -128.648620
   rho*vxc                -3.957592      -166.329387      -170.286979
   valence chg             4.170172         6.829828        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      173.666764 Ekcore=      3171.756639 Ektot    =     3345.423404
 Exc=    -128.648620 Ees   =     -6521.297693 EKohnSham=    -3304.522909
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 3 RMS DQ= 4.16E-2  last it= 5.10E-2
 AMIX: nmix=3 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 4.16D-02
   tj: 0.73785  -0.16385   0.00001
 mixrho: add corrections to qcell smrho = -0.17814D-06 -0.22681D-08

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.828250   ehk=   -3304.522909
 From last iter    ehf=   -3304.868852   ehk=   -3304.438236
 diffe(q)=  0.040602 (0.041568)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44964.832204 ehk(eV)=-44960.677801 sev(eV)=-14.061181

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570982
   smooth rhoves      9.540928   charge     3.554986
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.513698 -3.270360 -0.825710
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000476
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.466470      -185.003848      -190.470318
   rhoval*ves            -45.957742      -117.377125      -163.334867
   psnuc*ves              65.039599    -12970.222382    -12905.182783
   Eestatic                9.540928     -6543.799753     -6534.258825
   rho*exc                -2.513698      -127.432462      -129.946160
   rho*vxc                -3.270360      -168.736878      -172.007238
   valence chg             3.554986         7.445014        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0702

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.015777;  11.000000 electrons
         Sum occ. bands:   -2.782774, incl. Bloechl correction: -0.009094
 bndfp:Generating TDOS: efermi= -0.015777  dos window emin emax=  -0.726205  2.924145

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.109614    2.680045    7.429568
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.782774 Vin*nin=    -190.470318 Ek=Eb-Vin*nin=     187.687544
 Ek(core)=    3171.756639 Exc=    -129.946160 Ees=   -6534.258825 Eharris=   -3304.760801

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.486624   -184.573434   -190.060058
    E_B(band energy sum)=   -2.782774  E_B-nout*Vin=  187.277284

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.573586
   smooth rhoves      9.624006   charge     3.570432
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.526488 -3.287030 -0.826800
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.501840      -184.735689      -190.237530
   rhoval*ves            -46.076904      -117.067168      -163.144072
   psnuc*ves              65.324916    -12969.956021    -12904.631105
   Eestatic                9.624006     -6543.511594     -6533.887588
   rho*exc                -2.526488      -127.380479      -129.906967
   rho*vxc                -3.287030      -168.668248      -171.955279
   valence chg             3.570432         7.429568        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.277284 Ekcore=      3171.756639 Ektot    =     3359.033924
 Exc=    -129.906967 Ees   =     -6533.887588 EKohnSham=    -3304.760632
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.00E-3  last it= 4.16E-2
 AMIX: nmix=3 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.00D-03
   tj: 0.44916   0.77702  -0.17280
 mixrho: add corrections to qcell smrho = -0.18788D-06 -0.23922D-08

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.760801   ehk=   -3304.760632
 From last iter    ehf=   -3304.828250   ehk=   -3304.522909
 diffe(q)=  0.067449 (0.001005)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914508 ehk(eV)=-44963.912208 sev(eV)=-37.861861

--- BNDFP:  begin iteration 6 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.570952
   smooth rhoves      9.539980   charge     3.554823
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.513572 -3.270196 -0.825696
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000476
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.466154      -185.005910      -190.472064
   rhoval*ves            -45.956334      -117.379690      -163.336023
   psnuc*ves              65.036294    -12970.225610    -12905.189316
   Eestatic                9.539980     -6543.802650     -6534.262670
   rho*exc                -2.513572      -127.433134      -129.946706
   rho*vxc                -3.270196      -168.737765      -172.007961
   valence chg             3.554823         7.445177        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0703

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.015593;  11.000000 electrons
         Sum occ. bands:   -2.780136, incl. Bloechl correction: -0.009090
 bndfp:Generating TDOS: efermi= -0.015593  dos window emin emax=  -0.726161  2.924330

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.109347    2.680715    7.428632
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.780136 Vin*nin=    -190.472064 Ek=Eb-Vin*nin=     187.691928
 Ek(core)=    3171.756639 Exc=    -129.946706 Ees=   -6534.262670 Eharris=   -3304.760808

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.487752   -184.548038   -190.035790
    E_B(band energy sum)=   -2.780136  E_B-nout*Vin=  187.255653

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.573742
   smooth rhoves      9.628974   charge     3.571368
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.527274 -3.288055 -0.826864
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.504057      -184.720846      -190.224903
   rhoval*ves            -46.083957      -117.049618      -163.133574
   psnuc*ves              65.341906    -12969.944196    -12904.602290
   Eestatic                9.628974     -6543.496907     -6533.867932
   rho*exc                -2.527274      -127.377703      -129.904977
   rho*vxc                -3.288055      -168.664584      -171.952640
   valence chg             3.571368         7.428632        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.255653 Ekcore=      3171.756639 Ektot    =     3359.012293
 Exc=    -129.904977 Ees   =     -6533.867932 EKohnSham=    -3304.760616
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.07E-3  last it= 1.00E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: Reducing nmix to  0: t_j exceeds tm: tj= 16.51272
 AMIX: nmix=0 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.07D-03
 mixrho: add corrections to qcell smrho = -0.44398D-07 -0.56531D-09

 iors  : write rst restart file (binary mesh density)

   it  6  of 12    ehf=   -3304.760808   ehk=   -3304.760616
 From last iter    ehf=   -3304.760801   ehk=   -3304.760632
 diffe(q)= -0.000007 (0.001069)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.914602 ehk(eV)=-44963.911994 sev(eV)=-37.825980

--- BNDFP:  begin iteration 7 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.573742
   smooth rhoves      9.628974   charge     3.571368
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.527274 -3.288055 -0.826864
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000476
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.504056      -184.720846      -190.224903
   rhoval*ves            -46.083957      -117.049618      -163.133574
   psnuc*ves              65.341906    -12969.944196    -12904.602290
   Eestatic                9.628974     -6543.496907     -6533.867932
   rho*exc                -2.527274      -127.377703      -129.904977
   rho*vxc                -3.288055      -168.664584      -171.952640
   valence chg             3.571368         7.428632        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0675

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.028573;  11.000000 electrons
         Sum occ. bands:   -2.969770, incl. Bloechl correction: -0.009381
 bndfp:Generating TDOS: efermi= -0.028573  dos window emin emax=  -0.729427  2.911350

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.127553    2.635162    7.492391
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.969770 Vin*nin=    -190.224903 Ek=Eb-Vin*nin=     187.255132
 Ek(core)=    3171.756639 Exc=    -129.904977 Ees=   -6533.867932 Eharris=   -3304.761137

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.414125   -186.414302   -191.828426
    E_B(band energy sum)=   -2.969770  E_B-nout*Vin=  188.858656

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.563137
   smooth rhoves      9.293950   charge     3.507609
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.473856 -3.218424 -0.822494
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.353622      -185.824732      -191.178353
   rhoval*ves            -45.598523      -118.333951      -163.932475
   psnuc*ves              64.186424    -12970.903331    -12906.716907
   Eestatic                9.293950     -6544.618641     -6535.324691
   rho*exc                -2.473856      -127.575645      -130.049502
   rho*vxc                -3.218424      -168.925840      -172.144263
   valence chg             3.507609         7.492391        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      188.858656 Ekcore=      3171.756639 Ektot    =     3360.615295
 Exc=    -130.049502 Ees   =     -6535.324691 EKohnSham=    -3304.758897
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 3.74E-3  last it= 1.07E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 3.74D-03
   tj: 0.77877
 mixrho: add corrections to qcell smrho = -0.43842D-07 -0.55822D-09

 iors  : write rst restart file (binary mesh density)

   it  7  of 12    ehf=   -3304.761137   ehk=   -3304.758897
 From last iter    ehf=   -3304.760808   ehk=   -3304.760616
 diffe(q)= -0.000329 (0.003741)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.919079 ehk(eV)=-44963.888598 sev(eV)=-40.406102

--- BNDFP:  begin iteration 8 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571396
   smooth rhoves      9.554344   charge     3.557262
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515432 -3.272619 -0.825902
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.470626      -184.965999      -190.436625
   rhoval*ves            -45.977587      -117.333569      -163.311157
   psnuc*ves              65.086276    -12970.156387    -12905.070111
   Eestatic                9.554344     -6543.744978     -6534.190634
   rho*exc                -2.515432      -127.421484      -129.936916
   rho*vxc                -3.272619      -168.722369      -171.994988
   valence chg             3.557262         7.442738        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0698

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018794;  11.000000 electrons
         Sum occ. bands:   -2.826449, incl. Bloechl correction: -0.009163
 bndfp:Generating TDOS: efermi= -0.018794  dos window emin emax=  -0.726915  2.921129

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.113695    2.669668    7.444027
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.826449 Vin*nin=    -190.436625 Ek=Eb-Vin*nin=     187.610175
 Ek(core)=    3171.756639 Exc=    -129.936916 Ees=   -6534.190634 Eharris=   -3304.760736

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.468708   -184.990629   -190.459338
    E_B(band energy sum)=   -2.826449  E_B-nout*Vin=  187.632888

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571188
   smooth rhoves      9.547628   charge     3.555973
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.514339 -3.271193 -0.825816
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.467507      -184.980961      -190.448468
   rhoval*ves            -45.968045      -117.352476      -163.320521
   psnuc*ves              65.063302    -12970.164857    -12905.101555
   Eestatic                9.547628     -6543.758666     -6534.211038
   rho*exc                -2.514339      -127.424885      -129.939224
   rho*vxc                -3.271193      -168.726853      -171.998047
   valence chg             3.555973         7.444027        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.632888 Ekcore=      3171.756639 Ektot    =     3359.389528
 Exc=    -129.939224 Ees   =     -6534.211038 EKohnSham=    -3304.760735
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 6.09E-5  last it= 3.74E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 6.09D-05
   tj:-0.00067   0.05075
 mixrho: add corrections to qcell smrho = -0.48863D-07 -0.62215D-09

 iors  : write rst restart file (binary mesh density)

   it  8  of 12    ehf=   -3304.760736   ehk=   -3304.760735
 From last iter    ehf=   -3304.761137   ehk=   -3304.758897
 diffe(q)=  0.000402 (0.000061)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.913615 ehk(eV)=-44963.913605 sev(eV)=-38.456106

--- BNDFP:  begin iteration 9 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571323
   smooth rhoves      9.551920   charge     3.556787
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515022 -3.272084 -0.825872
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469436      -184.967205      -190.436641
   rhoval*ves            -45.974191      -117.336442      -163.310633
   psnuc*ves              65.078031    -12970.153161    -12905.075130
   Eestatic                9.551920     -6543.744801     -6534.192881
   rho*exc                -2.515022      -127.422389      -129.937411
   rho*vxc                -3.272084      -168.723559      -171.995643
   valence chg             3.556787         7.443213        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    18   18    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    18   18    0
 ... Done MPI k-loop: elapsed time=   0.0831

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018603;  11.000000 electrons
         Sum occ. bands:   -2.823725, incl. Bloechl correction: -0.009159
 bndfp:Generating TDOS: efermi= -0.018603  dos window emin emax=  -0.726856  2.921320

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.113446    2.670272    7.443175
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.823725 Vin*nin=    -190.436641 Ek=Eb-Vin*nin=     187.612916
 Ek(core)=    3171.756639 Exc=    -129.937411 Ees=   -6534.192881 Eharris=   -3304.760737

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.469472   -184.966815   -190.436287
    E_B(band energy sum)=   -2.823725  E_B-nout*Vin=  187.612563

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571331
   smooth rhoves      9.552142   charge     3.556825
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515053 -3.272124 -0.825875
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469516      -184.967129      -190.436645
   rhoval*ves            -45.974524      -117.336182      -163.310706
   psnuc*ves              65.078808    -12970.153283    -12905.074475
   Eestatic                9.552142     -6543.744733     -6534.192591
   rho*exc                -2.515053      -127.422295      -129.937348
   rho*vxc                -3.272124      -168.723436      -171.995560
   valence chg             3.556825         7.443175        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.612563 Ekcore=      3171.756639 Ektot    =     3359.369202
 Exc=    -129.937348 Ees   =     -6534.192591 EKohnSham=    -3304.760737
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 1.80E-6  last it= 6.09E-5
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2074  beta=1.00000  tm= 5.00000  rmsdel= 1.80D-06
   tj: 0.02506
 mixrho: add corrections to qcell smrho = -0.45181D-07 -0.57527D-09

 iors  : write rst restart file (binary mesh density)

   it  9  of 12    ehf=   -3304.760737   ehk=   -3304.760737
 From last iter    ehf=   -3304.760736   ehk=   -3304.760735
 diffe(q)= -0.000001 (0.000002)    tol= 0.000010 (0.000010)   more=F
c ehf(eV)=-44963.913632 ehk(eV)=-44963.913630 sev(eV)=-38.419032
 >>      1.84   exit  lmfp            1.38
CPU time:    1.845s     Tue Sep 19 17:16:37 2023 on process=0

  ==== procid=0 ====     calls      == cpu time ===   depth 1
  entry   xxxx  xxxx                per call  total  (depth is by TIM= in ctrl.*.)
      0      0      0        1       1.85       1.85   main
      0      0      0        1       0.01       0.01   |--m_lmfinit
      0      0      0        1       0.00       0.00   |--m_lattic_init
      0      0      0        1       0.45       0.45   |--m_mksym_init
      0      0      0        1       0.00       0.00   |--m_mkqp_init
      0      0      0        1       0.01       0.01   |--m_supot_init
      0      0      0        1       0.00       0.00   |--m_suham_init
      0      0      0        1       0.00       0.00   |--m_qplist_init
      0      0      0        1       0.00       0.00   |--m_qplist_qpsdivider
      0      0      0        1       0.00       0.00   |--m_igv2xall_init
      0      0      0        1       1.38       1.38   `--lmfp
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.2) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit c35df491f292381d706d43a93760b974f67f0241
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Sep 19 14:51:50 2023 +0900
INFO: linked at Tue Sep 19 17:03:40 JST 2023
===START LMF with   --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7 ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7<ctrl.cu >ctrlp.cu
 end of readctrlp
 end of readctrlp
 end of readctrlp
 end of readctrlp
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_READPSKIPF          defa n= 1 val= 1.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_ATOM@1             val=  A
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HOLE@1           val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_ATOM@1             val=  A
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: SYMGRP                  val= 
rval2: SYMGRPAF                val= 
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_MIX                val=  A3
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing param: A/B nmix wt= 0 3 1.000000  1.000000 beta wc killj=  1.000000 -1.000000 -1
 >> level: 1  CPUsec=      0.00  enter m_lmfinit
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110  5.100
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto rsmh2    1  1.30  0.00  1.00  1.30
mto  eh2     1 -1.00 -1.00 -1.00 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  3  2
 >>      0.01   exit  m_lmfinit       0.01
 >> level: 1  CPUsec=      0.01  enter m_lattic_init

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
 SYMGRP = find
 SYMGRP = find
  Cell vol=   78.538660
 SYMGRP = find

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkq= 331
 >>      0.01   exit  m_lattic_ini    0.00
 >> level: 1  CPUsec=      0.01  enter m_mksym_init
SpaceGroupSym of Lattice: ========start========================== 
 SYMGRP = find
  Generators except find: 
 sgroup:  1 symmetry operations from 0 generators
 symlat: Bravais system is cubic        with 48 symmetry operations.
 symcry: crystal invariant under 48 following symmetry operations for tol=  0.000100
   Enlarging ngen= 1  ng nggen= 48 6
   Enlarging ngen= 2  ng nggen= 48 48
 groupg: the following are sufficient to generate the space group:
  Generators:  trans(cart)= i*r3(1,1,-1) r4x
  Generators:: trans(frac)= i*r3(1,1,-1) r4x
 gensym: ig group ops (:vector means translation in cartesian)
    1  e
    2  i*r3(1,1,-1)
    3  r3(-1,-1,1)
    4  i
    5  r3(1,1,-1)
    6  i*r3(-1,-1,1)
    7  r4x
    8  m(1,0,-1)
    9  r4z
   10  i*r4x
   11  r2(1,0,-1)
   12  i*r4z
   13  r2x
   14  i*r3(-1,1,1)
   15  r3d
   16  mx
   17  r3(-1,1,1)
   18  i*r3d
   19  r4(-1,0,0)
   20  i*r4y
   21  r2(1,1,0)
   22  i*r4(-1,0,0)
   23  r4y
   24  m(1,1,0)
   25  i*r3(-1,-1,-1)
   26  r3(-1,1,-1)
   27  my
   28  r3(-1,-1,-1)
   29  i*r3(-1,1,-1)
   30  r2y
   31  i*r4(0,0,-1)
   32  r2(0,1,-1)
   33  i*r4(0,-1,0)
   34  r4(0,0,-1)
   35  m(0,1,-1)
   36  r4(0,-1,0)
   37  r3(1,-1,-1)
   38  mz
   39  r3(1,-1,1)
   40  i*r3(1,-1,-1)
   41  r2z
   42  i*r3(1,-1,1)
   43  r2(1,0,1)
   44  m(1,0,1)
   45  m(0,1,1)
   46  r2(0,1,1)
   47  r2(1,-1,0)
   48  m(1,-1,0)
 gensym: site permutation table for group operations ...
  ib/ig:  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 23 24 25 26 27 28 29 30 31 32 33 34 35 36 37 38 39 40 41 42 43 44 45 46 47 48
      1:  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1
 splcls:  ibas iclass ispec label(ispec)
            1     1     1     A
SpaceGroupSym of Lattice: ========end =========================== 

 >>      0.52   exit  m_mksym_init    0.51
 >> level: 1  CPUsec=      0.52  enter m_mkqp_init
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
 >>      0.52   exit  m_mkqp_init     0.00
 >> level: 1  CPUsec=      0.52  enter m_supot_init
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors
 >>      0.53   exit  m_supot_init    0.01

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 >> level: 1  CPUsec=      0.53  enter m_suham_init
 >>      0.53   exit  m_suham_init    0.00
 >> level: 1  CPUsec=      0.53  enter m_qplist_init
 m_qplistinit:start
 >>      0.53   exit  m_qplist_ini    0.00
 >> level: 1  CPUsec=      0.53  enter m_qplist_qpsdivider
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 1 16 1 30 1
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 2 31 1 45 1
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 0 1 1 15 1
 >>      0.53   exit  m_qplist_qps    0.00
 >> level: 1  CPUsec=      0.53  enter m_igv2xall_init
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 3 46 1 60 1
 >>      0.53   exit  m_igv2xall_i    0.00
 >> level: 1  CPUsec=      0.53  enter lmfp
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.39  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00
         site   1, species A       : augmentation lmax changed from 3 to 4
         site   1, species A       : inflate local density from nlm= 16 to 25

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.571327
   smooth rhoves      9.552029   charge     3.556804
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.515035 -3.272101 -0.825873
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000474
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.469466      -184.967476      -190.436941
   rhoval*ves            -45.974362      -117.336590      -163.310952
   psnuc*ves              65.078419    -12970.153573    -12905.075154
   Eestatic                9.552029     -6543.745082     -6534.193053
   rho*exc                -2.515035      -127.422360      -129.937395
   rho*vxc                -3.272101      -168.723521      -171.995622
   valence chg             3.556804         7.443196        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.3012

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018893;  11.000000 electrons
         Sum occ. bands:   -2.826375, incl. Bloechl correction: -0.009153
 bndfp:Generating TDOS: efermi= -0.018893  dos window emin emax=  -0.727731  2.921030

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128187    2.833439    7.294748
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.826375 Vin*nin=    -190.436941 Ek=Eb-Vin*nin=     187.610567
 Ek(core)=    3171.756639 Exc=    -129.937395 Ees=   -6534.193053 Eharris=   -3304.763242

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.930171   -184.356448   -190.286618
    E_B(band energy sum)=   -2.826375  E_B-nout*Vin=  187.460244

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581322
   smooth rhoves     10.109259   charge     3.705252
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657871 -3.458537 -0.832965
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010212      -184.303948      -190.314160
   rhoval*ves            -46.616553      -116.581739      -163.198292
   psnuc*ves              66.835071    -12971.741857    -12904.906786
   Eestatic               10.109259     -6544.161798     -6534.052539
   rho*exc                -2.657871      -127.269644      -129.927515
   rho*vxc                -3.458537      -168.523951      -171.982488
   valence chg             3.705252         7.294748        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.460244 Ekcore=      3171.756639 Ektot    =     3359.216883
 Exc=    -129.927515 Ees   =     -6534.052539 EKohnSham=    -3304.763171
 mixrealsmooth= T
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 0 RMS DQ= 4.32E-3
 AMIX: nmix=0 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 4.32D-03
 mixrho: add corrections to qcell smrho = -0.28210D-07 -0.35918D-09

 iors  : write rst restart file (binary mesh density)

   it  1  of 12    ehf=   -3304.763242   ehk=   -3304.763171
i ehf(eV)=-44963.947718 ehk(eV)=-44963.946747 sev(eV)=-38.455092

--- BNDFP:  begin iteration 2 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581322
   smooth rhoves     10.109259   charge     3.705252
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.657871 -3.458537 -0.832965
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000042
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.010212      -184.303948      -190.314160
   rhoval*ves            -46.616553      -116.581739      -163.198292
   psnuc*ves              66.835071    -12971.741857    -12904.906786
   Eestatic               10.109259     -6544.161798     -6534.052539
   rho*exc                -2.657871      -127.269644      -129.927515
   rho*vxc                -3.458537      -168.523951      -171.982488
   valence chg             3.705252         7.294748        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.3187

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020771;  11.000000 electrons
         Sum occ. bands:   -2.853922, incl. Bloechl correction: -0.009202
 bndfp:Generating TDOS: efermi= -0.020771  dos window emin emax=  -0.727840  2.919152

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.131603    2.824923    7.306680
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.853922 Vin*nin=    -190.314160 Ek=Eb-Vin*nin=     187.460237
 Ek(core)=    3171.756639 Exc=    -129.927515 Ees=   -6534.052539 Eharris=   -3304.763177

 mkekin:
   nout*Vin = smpart,onsite,total=:     -5.993067   -184.681211   -190.674278
    E_B(band energy sum)=   -2.853922  E_B-nout*Vin=  187.820356

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579313
   smooth rhoves     10.044731   charge     3.693320
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.647769 -3.445368 -0.832160
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.980433      -184.558086      -190.538519
   rhoval*ves            -46.529594      -116.859913      -163.389507
   psnuc*ves              66.619056    -12971.992530    -12905.373475
   Eestatic               10.044731     -6544.426221     -6534.381491
   rho*exc                -2.647769      -127.310810      -129.958579
   rho*vxc                -3.445368      -168.578315      -172.023683
   valence chg             3.693320         7.306680        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.820356 Ekcore=      3171.756639 Ektot    =     3359.576995
 Exc=    -129.958579 Ees   =     -6534.381491 EKohnSham=    -3304.763075
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 1 RMS DQ= 7.91E-4  last it= 4.32E-3
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 7.91D-04
   tj: 0.08152
 mixrho: add corrections to qcell smrho = -0.26744D-07 -0.34052D-09

 iors  : write rst restart file (binary mesh density)

   it  2  of 12    ehf=   -3304.763177   ehk=   -3304.763075
 From last iter    ehf=   -3304.763242   ehk=   -3304.763171
 diffe(q)=  0.000065 (0.000791)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946832 ehk(eV)=-44963.945439 sev(eV)=-38.829896

--- BNDFP:  begin iteration 3 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.579477
   smooth rhoves     10.049983   charge     3.694293
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.648592 -3.446441 -0.832226
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.982858      -184.537387      -190.520245
   rhoval*ves            -46.536698      -116.837238      -163.373936
   psnuc*ves              66.636665    -12971.972096    -12905.335431
   Eestatic               10.049983     -6544.404667     -6534.354684
   rho*exc                -2.648592      -127.307454      -129.956046
   rho*vxc                -3.446441      -168.573883      -172.020323
   valence chg             3.694293         7.305707        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.3197

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.012075;  11.000000 electrons
         Sum occ. bands:   -2.729577, incl. Bloechl correction: -0.009009
 bndfp:Generating TDOS: efermi= -0.012075  dos window emin emax=  -0.725702  2.927848

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.118797    2.849662    7.269135
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.729577 Vin*nin=    -190.520245 Ek=Eb-Vin*nin=     187.790668
 Ek(core)=    3171.756639 Exc=    -129.956046 Ees=   -6534.354684 Eharris=   -3304.763423

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.025692   -183.437952   -189.463644
    E_B(band energy sum)=   -2.729577  E_B-nout*Vin=  186.734067

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.586159
   smooth rhoves     10.257402   charge     3.730865
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.678840 -3.485865 -0.834811
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.067514      -183.822792      -189.890306
   rhoval*ves            -46.818539      -116.027336      -162.845875
   psnuc*ves              67.333342    -12971.272306    -12903.938964
   Eestatic               10.257402     -6543.649821     -6533.392419
   rho*exc                -2.678840      -127.181891      -129.860731
   rho*vxc                -3.485865      -168.408083      -171.893948
   valence chg             3.730865         7.269135        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      186.734067 Ekcore=      3171.756639 Ektot    =     3358.490706
 Exc=    -129.860731 Ees   =     -6533.392419 EKohnSham=    -3304.762444
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 2 RMS DQ= 2.33E-3  last it= 7.91E-4
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 2.33D-03
   tj: 0.74740  -0.00311
 mixrho: add corrections to qcell smrho = -0.27292D-07 -0.34750D-09

 iors  : write rst restart file (binary mesh density)

   it  3  of 12    ehf=   -3304.763423   ehk=   -3304.762444
 From last iter    ehf=   -3304.763177   ehk=   -3304.763075
 diffe(q)= -0.000246 (0.002327)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.950174 ehk(eV)=-44963.936866 sev(eV)=-37.138081

--- BNDFP:  begin iteration 4 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.581057
   smooth rhoves     10.098699   charge     3.702884
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655673 -3.455669 -0.832838
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.002563      -184.371298      -190.373861
   rhoval*ves            -46.603638      -116.647876      -163.251514
   psnuc*ves              66.801035    -12971.809140    -12905.008105
   Eestatic               10.098699     -6544.228508     -6534.129810
   rho*exc                -2.655673      -127.277965      -129.933638
   rho*vxc                -3.455669      -168.534944      -171.990614
   valence chg             3.702884         7.297116        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.3129

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.019147;  11.000000 electrons
         Sum occ. bands:   -2.830219, incl. Bloechl correction: -0.009164
 bndfp:Generating TDOS: efermi= -0.019147  dos window emin emax=  -0.727515  2.920776

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.129011    2.830110    7.298901
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.830219 Vin*nin=    -190.373861 Ek=Eb-Vin*nin=     187.543643
 Ek(core)=    3171.756639 Exc=    -129.933638 Ees=   -6534.129810 Eharris=   -3304.763166

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.000768   -184.420024   -190.420792
    E_B(band energy sum)=   -2.830219  E_B-nout*Vin=  187.590574

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580714
   smooth rhoves     10.088326   charge     3.701099
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.654220 -3.453776 -0.832708
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -5.998645      -184.401363      -190.400008
   rhoval*ves            -46.589292      -116.683560      -163.272851
   psnuc*ves              66.765944    -12971.837591    -12905.071647
   Eestatic               10.088326     -6544.260575     -6534.172249
   rho*exc                -2.654220      -127.283907      -129.938127
   rho*vxc                -3.453776      -168.542789      -171.996564
   valence chg             3.701099         7.298901        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.590574 Ekcore=      3171.756639 Ektot    =     3359.347213
 Exc=    -129.938127 Ees   =     -6534.172249 EKohnSham=    -3304.763163
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 3 RMS DQ= 1.06E-4  last it= 2.33E-3
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 1.06D-04
   tj: 0.08971   0.16465
 mixrho: add corrections to qcell smrho = -0.24084D-07 -0.30665D-09

 iors  : write rst restart file (binary mesh density)

   it  4  of 12    ehf=   -3304.763166   ehk=   -3304.763163
 From last iter    ehf=   -3304.763423   ehk=   -3304.762444
 diffe(q)=  0.000257 (0.000106)    tol= 0.000010 (0.000010)   more=T
i ehf(eV)=-44963.946681 ehk(eV)=-44963.946648 sev(eV)=-38.507388

--- BNDFP:  begin iteration 5 of 12
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580972
   smooth rhoves     10.096246   charge     3.702488
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655363 -3.455266 -0.832807
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001810      -184.375409      -190.377220
   rhoval*ves            -46.600168      -116.653727      -163.253894
   psnuc*ves              66.792659    -12971.812391    -12905.019731
   Eestatic               10.096246     -6544.233059     -6534.136813
   rho*exc                -2.655363      -127.279183      -129.934546
   rho*vxc                -3.455266      -168.536551      -171.991816
   valence chg             3.702488         7.297512        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of    60 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of    60 k= -0.0625  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of    60 k= -0.1875  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of    60 k= -0.3125  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of    60 k= -0.4375  0.5625  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of    60 k= -0.5625  0.6875  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of    60 k= -0.6875  0.8125  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of    60 k= -0.8125  0.9375  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of    60 k=  0.0625  0.0625  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of    60 k= -0.0625  0.1875  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of    60 k= -0.1875  0.3125  0.5625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of    60 k= -0.3125  0.4375  0.6875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of    60 k= -0.4375  0.5625  0.8125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of    60 k= -0.5625  0.6875  0.9375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of    60 k= -0.6875  0.8125  1.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.3240

 bzwts: --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018829;  11.000000 electrons
         Sum occ. bands:   -2.825667, incl. Bloechl correction: -0.009157
 bndfp:Generating TDOS: efermi= -0.018829  dos window emin emax=  -0.727422  2.921094

    ikp isp            q          nev ndimh       1       2       3       4       5       6       7       8       9      10      11      12      13      14

 m_bandcal_2nd: to fill eigenfunctions**2 up to Efermi

 mkrout:  Qtrue      sm,loc       local
   1   10.128558    2.830948    7.297611
  Symmetrize density..
 Make new boundary conditions for phi,phidot..

 m_mkehkf_etot1: Harris energy: (B.1) in JPSJ84,034702
 Eb(band sum)=       -2.825667 Vin*nin=    -190.377220 Ek=Eb-Vin*nin=     187.551552
 Ek(core)=    3171.756639 Exc=    -129.934546 Ees=   -6534.136813 Eharris=   -3304.763167

 mkekin:
   nout*Vin = smpart,onsite,total=:     -6.001716   -184.378482   -190.380198
    E_B(band energy sum)=   -2.825667  E_B-nout*Vin=  187.554531

 m_mkpot_energyterms
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580952
   smooth rhoves     10.095664   charge     3.702389
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655283 -3.455161 -0.832800
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=F
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001597      -184.377351      -190.378949
   rhoval*ves            -46.599362      -116.655966      -163.255329
   psnuc*ves              66.790690    -12971.814394    -12905.023704
   Eestatic               10.095664     -6544.235180     -6534.139516
   rho*exc                -2.655283      -127.279538      -129.934821
   rho*vxc                -3.455161      -168.537021      -171.992181
   valence chg             3.702389         7.297611        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:      0.00000

 m_mkehkf_etot2: Kohn-Sham energy:  Ek = Eband-Vin*nout
 Ek=      187.554531 Ekcore=      3171.756639 Ektot    =     3359.311171
 Exc=    -129.934821 Ees   =     -6534.139516 EKohnSham=    -3304.763166
 wgtsmooth=   3.1622776601683791E-002
 mixrho: sought 3 iter from file mixm ; read 4 RMS DQ= 6.54E-6  last it= 1.06E-4
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=  2236  beta=1.00000  tm= 5.00000  rmsdel= 6.54D-06
   tj:-0.06557
 mixrho: add corrections to qcell smrho = -0.26037D-07 -0.33152D-09

 iors  : write rst restart file (binary mesh density)

   it  5  of 12    ehf=   -3304.763167   ehk=   -3304.763166
 From last iter    ehf=   -3304.763166   ehk=   -3304.763163
 diffe(q)= -0.000001 (0.000007)    tol= 0.000010 (0.000010)   more=F
c ehf(eV)=-44963.946693 ehk(eV)=-44963.946690 sev(eV)=-38.445462
 >>      2.89   exit  lmfp            2.36
CPU time:    2.892s     Tue Sep 19 17:16:41 2023 on process=0

  ==== procid=0 ====     calls      == cpu time ===   depth 1
  entry   xxxx  xxxx                per call  total  (depth is by TIM= in ctrl.*.)
      0      0      0        1       2.89       2.89   main
      0      0      0        1       0.01       0.01   |--m_lmfinit
      0      0      0        1       0.00       0.00   |--m_lattic_init
      0      0      0        1       0.51       0.51   |--m_mksym_init
      0      0      0        1       0.00       0.00   |--m_mkqp_init
      0      0      0        1       0.01       0.01   |--m_supot_init
      0      0      0        1       0.00       0.00   |--m_suham_init
      0      0      0        1       0.00       0.00   |--m_qplist_init
      0      0      0        1       0.00       0.00   |--m_qplist_qpsdivider
      0      0      0        1       0.00       0.00   |--m_igv2xall_init
      0      0      0        1       2.36       2.36   `--lmfp
Exit 0 procid= 0 OK! end of LMF ======================
INFO: Ubuntu 20.04.4 LTS \n \l
INFO: GNU Fortran (Ubuntu 9.4.0-1ubuntu1~20.04.2) 9.4.0
INFO: -O2 -g -fimplicit-none -finit-integer=NaN -finit-real=NaN -JOBJ.gfortran -IOBJ.gfortran
INFO: MATH: -lmkl_rt
INFO: git: commit c35df491f292381d706d43a93760b974f67f0241
INFO:    : Author: Takao Kotani <takaokotani@gmail.com>
INFO:    : Date:   Tue Sep 19 14:51:50 2023 +0900
INFO: linked at Tue Sep 19 17:03:40 JST 2023
===START LMF with   --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7 --band:fn=syml ===
mpisize=4
m_lmfinit: LMF
cmdl for python=/home/takao/ecalj/SRC/TestInstall/bin/ctrl2ctrlp.py  --no-iactiv cu -vnk=8 -vbigbas=t -vmetal=3 -vrsm2=1.3 -vrsmd1x=1 -vlmx=4 -vpwmode=0 -voveps=0d-7 --band:fn=syml<ctrl.cu >ctrlp.cu
 end of readctrlp
 end of readctrlp
 end of readctrlp
 end of readctrlp
rval2: STRUC_NSPEC             requ n= 1 val= 1.00000000
rval2: STRUC_NBAS              requ n= 1 val= 1.00000000
rval2: HAM_NSPIN               defa n= 1 val= 1.00000000
rval2: IO_VERBOS               defa n= 1 val= 31.00000000
rval2: IO_TIM                  defa n= 1 val= 1.00000000
rval2: STRUC_ALAT              ---- n= 1 val= 6.79800000
rval2: STRUC_DALAT             ---- n= 1 val= 0.00000000
rval2: STRUC_PLAT              requ n= 9 val= 0.00000000  0.50000000  0.50000000  0.50000000  0.00000000  0.50000000  0.50000000  0.50000000  0.00000000
rval2: OPTIONS_HF              defa n= 1 val= 0.00000000
rval2: HAM_REL                 defa n= 1 val= 1.00000000
rval2: HAM_SO                  defa n= 1 val= 0.00000000
rval2: HAM_SOCAXIS             defa n= 3 val= 0.00000000  0.00000000  1.00000000
rval2: HAM_GMAX                defa n= 1 val= 9.00000000
rval2: HAM_FTMESH              defa n= 3 val= 10.00000000  10.00000000  10.00000000
rval2: HAM_TOL                 defa n= 1 val= 0.00000100
rval2: HAM_FRZWF               defa n= 1 val= 0.00000000
rval2: HAM_XCFUN               defa n= 1 val= 2.00000000
rval2: HAM_FORCES              defa n= 1 val= 0.00000000
rval2: HAM_RDSIG               defa n= 1 val= 1.00000000
rval2: HAM_ScaledSigma         defa n= 1 val= 1.00000000
rval2: HAM_EWALD               defa n= 1 val= 0.00000000
rval2: HAM_OVEPS               defa n= 1 val= 0.00000000
rval2: HAM_PWMODE              defa n= 1 val= 0.00000000
rval2: HAM_PWEMAX              defa n= 1 val= 3.00000000
rval2: HAM_READP               defa n= 1 val= 0.00000000
rval2: HAM_READPSKIPF          defa n= 1 val= 1.00000000
rval2: HAM_V0FIX               defa n= 1 val= 0.00000000
rval2: HAM_PNUFIX              defa n= 1 val= 0.00000000
=== SPEC =1
rval2: SPEC_ATOM@1             val=  A
rval2: SPEC_Z@1                ---- n= 1 val= 29.00000000
rval2: SPEC_R@1                ---- n= 1 val= 2.31127105
rval2: SPEC_R/W@1              ---- n= 0 val= 
rval2: SPEC_R/A@1              ---- n= 0 val= 
rval2: SPEC_A@1                defa n= 1 val= 0.02500000
rval2: SPEC_NR@1               defa n= 1 val= 0.00000000
rval2: SPEC_RSMH@1             ---- n= 3 val= 2.50000000  2.50000000  1.00000000
rval2: SPEC_EH@1               requ n= 3 val= -0.01000000 -0.01000000 -0.01000000
rval2: SPEC_RSMH2@1            ---- n= 5 val= 1.30000000  0.00000000  1.00000000  1.30000000  0.00000000
rval2: SPEC_EH2@1              requ n= 4 val= -1.00000000 -1.00000000 -1.00000000 -0.01000000
rval2: SPEC_LMX@1              defa n= 1 val= 3.00000000
rval2: SPEC_LMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_LMXL@1             defa n= 1 val= 4.00000000
rval2: SPEC_P@1                ---- n= 4 val= 4.65000000  4.34000000  3.87000000  4.11000000
rval2: SPEC_Q@1                ---- n= 0 val= 
rval2: SPEC_NMCORE@1           defa n= 1 val= 0.00000000
rval2: SPEC_PZ@1               ---- n= 3 val= 5.50000000  5.50000000  4.50000000
rval2: SPEC_LFOCA@1            defa n= 1 val= 1.00000000
rval2: SPEC_KMXA@1             defa n= 1 val= 4.00000000
rval2: SPEC_RSMA@1             defa n= 1 val= 0.92450842
rval2: SPEC_IDMOD@1            ---- n= 5 val= 0.00000000  0.00000000  0.00000000  1.00000000  1.00000000
rval2: SPEC_FRZWF@1            defa n= 1 val= 0.00000000
rval2: SPEC_IDU@1              ---- n= 0 val= 
rval2: SPEC_UH@1               ---- n= 0 val= 
rval2: SPEC_JH@1               ---- n= 0 val= 
rval2: SPEC_C-HOLE@1           val= 
rval2: SPEC_C-HQ@1             defa n= 2 val= -1.00000000  0.00000000
rval2: SPEC_EREF1              defa n= 1 val= 0.00000000
=== SITE =1
rval2: SITE_ATOM@1             val=  A
rval2: SITE_POS@1              ---- n= 3 val= 0.00000000  0.00000000  0.00000000
rval2: SITE_RELAX@1            defa n= 3 val= 1.00000000  1.00000000  1.00000000
rval2: SITE_AF@1               defa n= 1 val= 0.00000000
rval2: STR_RMAXS               ---- n= 0 val= 
rval2: STR_RMAX                ---- n= 0 val= 
rval2: STR_MXNBR               defa n= 1 val= 0.00000000
rval2: BZ_NKABC                ---- n= 1 val= 8.00000000
rval2: BZ_BZJOB                ---- n= 1 val= 1.00000000
rval2: BZ_METAL                defa n= 1 val= 3.00000000
rval2: BZ_TETRA                defa n= 1 val= 1.00000000
rval2: BZ_N                    defa n= 1 val= 0.00000000
rval2: BZ_W                    defa n= 1 val= 0.00200000
rval2: BZ_ZBAK                 defa n= 1 val= 0.00000000
rval2: BZ_SAVDOS               defa n= 1 val= 1.00000000
rval2: BZ_NPTS                 defa n= 1 val= 1001.00000000
rval2: BZ_DOSMAX               defa n= 1 val= 2.93992268
rval2: BZ_EFMAX                defa n= 1 val= 5.00000000
rval2: BZ_FSMOM                defa n= 1 val= -99999.00000000
rval2: BZ_FSMOMMETHOD          defa n= 1 val= 0.00000000
rval2: SYMGRP                  val= 
rval2: SYMGRPAF                val= 
rval2: EWALD_AS                defa n= 1 val= 2.00000000
rval2: EWALD_TOL               defa n= 1 val= 0.00000000
rval2: EWALD_NKDMX             defa n= 1 val= 600.00000000
rval2: ITER_NIT                defa n= 1 val= 12.00000000
rval2: ITER_NRMIX              defa n= 1 val= 80.00000000
rval2: ITER_MIX                val=  A3
rval2: ITER_CONV               defa n= 1 val= 0.00001000
rval2: ITER_CONVC              defa n= 1 val= 0.00001000
rval2: ITER_UMIX               defa n= 1 val= 0.50000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_TOLU               defa n= 1 val= 0.00000000
rval2: ITER_b                  defa n= 1 val= 1.00000000
rval2: ITER_wc                 defa n= 1 val= -1.00000000
rval2: ITER_w                  defa n= 2 val= 1.00000000  1.00000000
rval2: ITER_k                  defa n= 1 val= -1.00000000
rval2: DYN_MODE                defa n= 1 val= 0.00000000
rval2: DYN_NIT                 defa n= 1 val= 1.00000000
rval2: DYN_HESS                defa n= 1 val= 1.00000000
rval2: DYN_XTOL                defa n= 1 val= 0.00100000
rval2: DYN_GTOL                defa n= 1 val= 0.00000000
rval2: DYN_STEP                defa n= 1 val= 0.01500000
rval2: DYN_NKILL               defa n= 1 val= 0.00000000
mixing param: A/B nmix wt= 0 3 1.000000  1.000000 beta wc killj=  1.000000 -1.000000 -1
 >> level: 1  CPUsec=      0.00  enter m_lmfinit
 ===> for --jobgw, pwmode is switched to be  0
  bndfp (warning): no sigm file found ... LDA calculation only
pnu list       ibas isp  pnu(0:lmxa) 
pnu: j isp pnu= 1 1 4.650  4.340  3.870  4.110  5.100
pnz: j isp  pz= 1 1 5.500  5.500  4.500  0.000  0.000

mto === MTO setting ===
mto ispec lmxb lpz nkapii nkaphh=    1    3    1    2    2
mto rsmh1    1  2.50  2.50  1.00
mto   eh1    1 -0.01 -0.01 -0.01
mto rsmh2    1  1.30  0.00  1.00  1.30
mto  eh2     1 -1.00 -1.00 -1.00 -0.01
mto pz       1  5.50  5.50  4.50
mto lh       2  3  2
 >>      0.01   exit  m_lmfinit       0.01
 >> level: 1  CPUsec=      0.01  enter m_lattic_init

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
 SYMGRP = find
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
 SYMGRP = find
 SYMGRP = find
  Cell vol=   78.538660

LATTC:  as= 2.000   tol= 1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkq= 331
 >>      0.01   exit  m_lattic_ini    0.00
 >> level: 1  CPUsec=      0.01  enter m_mksym_init
SpaceGroupSym of Lattice: ========start========================== 
 SYMGRP = find
  Generators except find: 
 sgroup:  1 symmetry operations from 0 generators
 symlat: Bravais system is cubic        with 48 symmetry operations.
 symcry: crystal invariant under 48 following symmetry operations for tol=  0.000100
   Enlarging ngen= 1  ng nggen= 48 6
   Enlarging ngen= 2  ng nggen= 48 48
 groupg: the following are sufficient to generate the space group:
  Generators:  trans(cart)= i*r3(1,1,-1) r4x
  Generators:: trans(frac)= i*r3(1,1,-1) r4x
 gensym: ig group ops (:vector means translation in cartesian)
    1  e
    2  i*r3(1,1,-1)
    3  r3(-1,-1,1)
    4  i
    5  r3(1,1,-1)
    6  i*r3(-1,-1,1)
    7  r4x
    8  m(1,0,-1)
    9  r4z
   10  i*r4x
   11  r2(1,0,-1)
   12  i*r4z
   13  r2x
   14  i*r3(-1,1,1)
   15  r3d
   16  mx
   17  r3(-1,1,1)
   18  i*r3d
   19  r4(-1,0,0)
   20  i*r4y
   21  r2(1,1,0)
   22  i*r4(-1,0,0)
   23  r4y
   24  m(1,1,0)
   25  i*r3(-1,-1,-1)
   26  r3(-1,1,-1)
   27  my
   28  r3(-1,-1,-1)
   29  i*r3(-1,1,-1)
   30  r2y
   31  i*r4(0,0,-1)
   32  r2(0,1,-1)
   33  i*r4(0,-1,0)
   34  r4(0,0,-1)
   35  m(0,1,-1)
   36  r4(0,-1,0)
   37  r3(1,-1,-1)
   38  mz
   39  r3(1,-1,1)
   40  i*r3(1,-1,-1)
   41  r2z
   42  i*r3(1,-1,1)
   43  r2(1,0,1)
   44  m(1,0,1)
   45  m(0,1,1)
   46  r2(0,1,1)
   47  r2(1,-1,0)
   48  m(1,-1,0)
 gensym: site permutation table for group operations ...
  ib/ig:  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 23 24 25 26 27 28 29 30 31 32 33 34 35 36 37 38 39 40 41 42 43 44 45 46 47 48
      1:  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1  1
 splcls:  ibas iclass ispec label(ispec)
            1     1     1     A
SpaceGroupSym of Lattice: ========end =========================== 

 >>      0.51   exit  m_mksym_init    0.50
 >> level: 1  CPUsec=      0.51  enter m_mkqp_init
 BZMESH:     60 irreducible QP from    8   8   8 shift=TTT
 TETIRR: sorting     3072 tetrahedra ...
     264 inequivalent tetrahedron=
 >>      0.51   exit  m_mkqp_init     0.00
 >> level: 1  CPUsec=      0.51  enter m_supot_init
MSHSIZ: mesh has 10 x 10 x 10 divisions; length =     0.481     0.481     0.481
      generated from gmax (a.u.)=      9.0000: 889 vectors of 1000 (88%)
 SGVSYM: 38 symmetry stars found for 861 reciprocal lattice vectors
 >>      0.52   exit  m_supot_init    0.01

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     813 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     813 
  A        3    1.30  -0.01   6.806    2.09E-06     411 
 >> level: 1  CPUsec=      0.52  enter m_suham_init
 >>      0.52   exit  m_suham_init    0.00
 >> level: 1  CPUsec=      0.52  enter m_qplist_init
 m_qplistinit:start
  --- Readin syml file --- 
   41   0.5000   0.5000   0.5000    0.0000   0.0000   0.0000 L Gamma
   41   0.0000   0.0000   0.0000    1.0000   0.0000   0.0000 Gamma X
   21   1.0000   0.0000   0.0000    1.0000   0.5000   0.0000 X W
   41   1.0000   0.5000   0.0000    0.0000   0.0000   0.0000 W Gamma
nsyml nkp=    4  144
 -------- qplist --------           4
    1   0.500   0.500   0.500  <-- isyml= 001
    2   0.487   0.487   0.487 
    3   0.475   0.475   0.475 
    4   0.463   0.463   0.463 
    5   0.450   0.450   0.450 
    6   0.438   0.438   0.438 
    7   0.425   0.425   0.425 
    8   0.412   0.412   0.412 
    9   0.400   0.400   0.400 
   10   0.388   0.388   0.388 
   11   0.375   0.375   0.375 
   12   0.362   0.362   0.362 
   13   0.350   0.350   0.350 
   14   0.338   0.338   0.338 
   15   0.325   0.325   0.325 
   16   0.312   0.312   0.312 
   17   0.300   0.300   0.300 
   18   0.287   0.287   0.287 
   19   0.275   0.275   0.275 
   20   0.263   0.263   0.263 
   21   0.250   0.250   0.250 
   22   0.237   0.237   0.237 
   23   0.225   0.225   0.225 
   24   0.213   0.213   0.213 
   25   0.200   0.200   0.200 
   26   0.188   0.188   0.188 
   27   0.175   0.175   0.175 
   28   0.162   0.162   0.162 
   29   0.150   0.150   0.150 
   30   0.138   0.138   0.138 
   31   0.125   0.125   0.125 
   32   0.112   0.112   0.112 
   33   0.100   0.100   0.100 
   34   0.088   0.088   0.088 
   35   0.075   0.075   0.075 
   36   0.062   0.062   0.062 
   37   0.050   0.050   0.050 
   38   0.037   0.037   0.037 
   39   0.025   0.025   0.025 
   40   0.013   0.013   0.013 
   41   0.000   0.000   0.000 
   42   0.000   0.000   0.000  <-- isyml= 002
   43   0.025   0.000   0.000 
   44   0.050   0.000   0.000 
   45   0.075   0.000   0.000 
   46   0.100   0.000   0.000 
   47   0.125   0.000   0.000 
   48   0.150   0.000   0.000 
   49   0.175   0.000   0.000 
   50   0.200   0.000   0.000 
   51   0.225   0.000   0.000 
   52   0.250   0.000   0.000 
   53   0.275   0.000   0.000 
   54   0.300   0.000   0.000 
   55   0.325   0.000   0.000 
   56   0.350   0.000   0.000 
   57   0.375   0.000   0.000 
   58   0.400   0.000   0.000 
   59   0.425   0.000   0.000 
   60   0.450   0.000   0.000 
   61   0.475   0.000   0.000 
   62   0.500   0.000   0.000 
   63   0.525   0.000   0.000 
   64   0.550   0.000   0.000 
   65   0.575   0.000   0.000 
   66   0.600   0.000   0.000 
   67   0.625   0.000   0.000 
   68   0.650   0.000   0.000 
   69   0.675   0.000   0.000 
   70   0.700   0.000   0.000 
   71   0.725   0.000   0.000 
   72   0.750   0.000   0.000 
   73   0.775   0.000   0.000 
   74   0.800   0.000   0.000 
   75   0.825   0.000   0.000 
   76   0.850   0.000   0.000 
   77   0.875   0.000   0.000 
   78   0.900   0.000   0.000 
   79   0.925   0.000   0.000 
   80   0.950   0.000   0.000 
   81   0.975   0.000   0.000 
   82   1.000   0.000   0.000 
   83   1.000   0.000   0.000  <-- isyml= 003
   84   1.000   0.025   0.000 
   85   1.000   0.050   0.000 
   86   1.000   0.075   0.000 
   87   1.000   0.100   0.000 
   88   1.000   0.125   0.000 
   89   1.000   0.150   0.000 
   90   1.000   0.175   0.000 
   91   1.000   0.200   0.000 
   92   1.000   0.225   0.000 
   93   1.000   0.250   0.000 
   94   1.000   0.275   0.000 
   95   1.000   0.300   0.000 
   96   1.000   0.325   0.000 
   97   1.000   0.350   0.000 
   98   1.000   0.375   0.000 
   99   1.000   0.400   0.000 
  100   1.000   0.425   0.000 
  101   1.000   0.450   0.000 
  102   1.000   0.475   0.000 
  103   1.000   0.500   0.000 
  104   1.000   0.500   0.000  <-- isyml= 004
  105   0.975   0.487   0.000 
  106   0.950   0.475   0.000 
  107   0.925   0.463   0.000 
  108   0.900   0.450   0.000 
  109   0.875   0.438   0.000 
  110   0.850   0.425   0.000 
  111   0.825   0.412   0.000 
  112   0.800   0.400   0.000 
  113   0.775   0.388   0.000 
  114   0.750   0.375   0.000 
  115   0.725   0.362   0.000 
  116   0.700   0.350   0.000 
  117   0.675   0.338   0.000 
  118   0.650   0.325   0.000 
  119   0.625   0.312   0.000 
  120   0.600   0.300   0.000 
  121   0.575   0.287   0.000 
  122   0.550   0.275   0.000 
  123   0.525   0.263   0.000 
  124   0.500   0.250   0.000 
  125   0.475   0.237   0.000 
  126   0.450   0.225   0.000 
  127   0.425   0.213   0.000 
  128   0.400   0.200   0.000 
  129   0.375   0.188   0.000 
  130   0.350   0.175   0.000 
  131   0.325   0.162   0.000 
  132   0.300   0.150   0.000 
  133   0.275   0.138   0.000 
  134   0.250   0.125   0.000 
  135   0.225   0.112   0.000 
  136   0.200   0.100   0.000 
  137   0.175   0.088   0.000 
  138   0.150   0.075   0.000 
  139   0.125   0.062   0.000 
  140   0.100   0.050   0.000 
  141   0.075   0.037   0.000 
  142   0.050   0.025   0.000 
  143   0.025   0.013   0.000 
  144   0.000   0.000   0.000 
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 3 109 1 144 1
 >>      0.52   exit  m_qplist_ini    0.00
 >> level: 1  CPUsec=      0.52  enter m_qplist_qpsdivider
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 1 37 1 72 1
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 2 73 1 108 1
m_qplist_qspdivider: rank,(iqini,ispini),(iqend,ispend)= 0 1 1 36 1
 >>      0.52   exit  m_qplist_qps    0.00
 >> level: 1  CPUsec=      0.52  enter m_igv2xall_init
 >>      0.52   exit  m_igv2xall_i    0.00
 >> level: 1  CPUsec=      0.52  enter lmfp
lmv7: Read rst version ID=  2.00

 iors  : read rst restart file (binary mesh density)
          use from  restart file:use window, pnu,
          ignore in restart file:
         site   1:A       :file pnu is  4.66  4.38  3.88  4.11  5.10
         site   1:A       :file pz  is  5.50  5.50  4.50  0.00  0.00

 Basis, after reading restart file
 site spec        pos (Cartesian coordinates)         pos (multiples of plat)
   1         A  0.000000   0.000000   0.000000    0.000000   0.000000   0.000000

--- BNDFP:  begin iteration 1 of 1
 m_mkpot_init: Making one-particle potential ...
  esmsmves: ESM is not turned on, you need esm_input.dat for ESM mode
 smves: Add vconst to Ele.Static Pot. so that avaraged Ves at Rmt is zero: vconst=-0.580968
   smooth rhoves     10.096145   charge     3.702474
  smvxcm: all smrho_w is positive
  smvxc2: smooth isp rhoeps rhomu vxcavg= 1 -2.655352 -3.455251 -0.832806
  locpot:
   site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
    sm core charge in MT=  0.263001 =total-spillout=  0.267647 -  0.004646
     potential shift to crystal energy zero:    0.000043
  mkpot:
   Energy terms:           smooth           local           total
   rhoval*veff             -6.001791      -184.375777      -190.377567
   rhoval*ves            -46.600022      -116.654157      -163.254180
   psnuc*ves              66.792313    -12971.812873    -12905.020560
   Eestatic               10.096145     -6544.233515     -6534.137370
   rho*exc                -2.655352      -127.279252      -129.934604
   rho*vxc                -3.455251      -168.536642      -171.991894
   valence chg             3.702474         7.297526        11.000000
   core charge            18.000000        -0.000000        18.000000
   Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
   hom background     0.00000   deviation from neutrality:     -0.00000
 m_bandcal_init: start
 bndfp: kpt     1 of   144 k=  0.5000  0.5000  0.5000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     2 of   144 k=  0.4875  0.4875  0.4875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     3 of   144 k=  0.4750  0.4750  0.4750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     4 of   144 k=  0.4625  0.4625  0.4625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     5 of   144 k=  0.4500  0.4500  0.4500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     6 of   144 k=  0.4375  0.4375  0.4375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     7 of   144 k=  0.4250  0.4250  0.4250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     8 of   144 k=  0.4125  0.4125  0.4125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt     9 of   144 k=  0.4000  0.4000  0.4000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    10 of   144 k=  0.3875  0.3875  0.3875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    11 of   144 k=  0.3750  0.3750  0.3750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    12 of   144 k=  0.3625  0.3625  0.3625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    13 of   144 k=  0.3500  0.3500  0.3500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    14 of   144 k=  0.3375  0.3375  0.3375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    15 of   144 k=  0.3250  0.3250  0.3250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    16 of   144 k=  0.3125  0.3125  0.3125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    17 of   144 k=  0.3000  0.3000  0.3000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    18 of   144 k=  0.2875  0.2875  0.2875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    19 of   144 k=  0.2750  0.2750  0.2750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    20 of   144 k=  0.2625  0.2625  0.2625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    21 of   144 k=  0.2500  0.2500  0.2500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    22 of   144 k=  0.2375  0.2375  0.2375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    23 of   144 k=  0.2250  0.2250  0.2250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    24 of   144 k=  0.2125  0.2125  0.2125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    25 of   144 k=  0.2000  0.2000  0.2000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    26 of   144 k=  0.1875  0.1875  0.1875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    27 of   144 k=  0.1750  0.1750  0.1750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    28 of   144 k=  0.1625  0.1625  0.1625 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    29 of   144 k=  0.1500  0.1500  0.1500 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    30 of   144 k=  0.1375  0.1375  0.1375 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    31 of   144 k=  0.1250  0.1250  0.1250 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    32 of   144 k=  0.1125  0.1125  0.1125 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    33 of   144 k=  0.1000  0.1000  0.1000 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    34 of   144 k=  0.0875  0.0875  0.0875 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    35 of   144 k=  0.0750  0.0750  0.0750 ndimh = nmto+napw =    31   31    0
 bndfp: kpt    36 of   144 k=  0.0625  0.0625  0.0625 ndimh = nmto+napw =    31   31    0
 ... Done MPI k-loop: elapsed time=   0.8267
  Writing bands to bands file for gnuplot ...
 bndfp: kpt    1 of  144 k jsp=  0.50000  0.50000  0.50000 1 nev=   31
 bndfp: kpt    2 of  144 k jsp=  0.48750  0.48750  0.48750 1 nev=   31
 bndfp: kpt    3 of  144 k jsp=  0.47500  0.47500  0.47500 1 nev=   31
 bndfp: kpt    4 of  144 k jsp=  0.46250  0.46250  0.46250 1 nev=   31
 bndfp: kpt    5 of  144 k jsp=  0.45000  0.45000  0.45000 1 nev=   31
 bndfp: kpt    6 of  144 k jsp=  0.43750  0.43750  0.43750 1 nev=   31
 bndfp: kpt    7 of  144 k jsp=  0.42500  0.42500  0.42500 1 nev=   31
 bndfp: kpt    8 of  144 k jsp=  0.41250  0.41250  0.41250 1 nev=   31
 bndfp: kpt    9 of  144 k jsp=  0.40000  0.40000  0.40000 1 nev=   31
 bndfp: kpt   10 of  144 k jsp=  0.38750  0.38750  0.38750 1 nev=   31
 bndfp: kpt   11 of  144 k jsp=  0.37500  0.37500  0.37500 1 nev=   31
 bndfp: kpt   12 of  144 k jsp=  0.36250  0.36250  0.36250 1 nev=   31
 bndfp: kpt   13 of  144 k jsp=  0.35000  0.35000  0.35000 1 nev=   31
 bndfp: kpt   14 of  144 k jsp=  0.33750  0.33750  0.33750 1 nev=   31
 bndfp: kpt   15 of  144 k jsp=  0.32500  0.32500  0.32500 1 nev=   31
 bndfp: kpt   16 of  144 k jsp=  0.31250  0.31250  0.31250 1 nev=   31
 bndfp: kpt   17 of  144 k jsp=  0.30000  0.30000  0.30000 1 nev=   31
 bndfp: kpt   18 of  144 k jsp=  0.28750  0.28750  0.28750 1 nev=   31
 bndfp: kpt   19 of  144 k jsp=  0.27500  0.27500  0.27500 1 nev=   31
 bndfp: kpt   20 of  144 k jsp=  0.26250  0.26250  0.26250 1 nev=   31
 bndfp: kpt   21 of  144 k jsp=  0.25000  0.25000  0.25000 1 nev=   31
 bndfp: kpt   22 of  144 k jsp=  0.23750  0.23750  0.23750 1 nev=   31
 bndfp: kpt   23 of  144 k jsp=  0.22500  0.22500  0.22500 1 nev=   31
 bndfp: kpt   24 of  144 k jsp=  0.21250  0.21250  0.21250 1 nev=   31
 bndfp: kpt   25 of  144 k jsp=  0.20000  0.20000  0.20000 1 nev=   31
 bndfp: kpt   26 of  144 k jsp=  0.18750  0.18750  0.18750 1 nev=   31
 bndfp: kpt   27 of  144 k jsp=  0.17500  0.17500  0.17500 1 nev=   31
 bndfp: kpt   28 of  144 k jsp=  0.16250  0.16250  0.16250 1 nev=   31
 bndfp: kpt   29 of  144 k jsp=  0.15000  0.15000  0.15000 1 nev=   31
 bndfp: kpt   30 of  144 k jsp=  0.13750  0.13750  0.13750 1 nev=   31
 bndfp: kpt   31 of  144 k jsp=  0.12500  0.12500  0.12500 1 nev=   31
 bndfp: kpt   32 of  144 k jsp=  0.11250  0.11250  0.11250 1 nev=   31
 bndfp: kpt   33 of  144 k jsp=  0.10000  0.10000  0.10000 1 nev=   31
 bndfp: kpt   34 of  144 k jsp=  0.08750  0.08750  0.08750 1 nev=   31
 bndfp: kpt   35 of  144 k jsp=  0.07500  0.07500  0.07500 1 nev=   31
 bndfp: kpt   36 of  144 k jsp=  0.06250  0.06250  0.06250 1 nev=   31
 bndfp: kpt   37 of  144 k jsp=  0.05000  0.05000  0.05000 1 nev=   31
 bndfp: kpt   38 of  144 k jsp=  0.03750  0.03750  0.03750 1 nev=   31
 bndfp: kpt   39 of  144 k jsp=  0.02500  0.02500  0.02500 1 nev=   31
 bndfp: kpt   40 of  144 k jsp=  0.01250  0.01250  0.01250 1 nev=   31
 bndfp: kpt   41 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   42 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   43 of  144 k jsp=  0.02500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   44 of  144 k jsp=  0.05000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   45 of  144 k jsp=  0.07500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   46 of  144 k jsp=  0.10000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   47 of  144 k jsp=  0.12500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   48 of  144 k jsp=  0.15000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   49 of  144 k jsp=  0.17500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   50 of  144 k jsp=  0.20000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   51 of  144 k jsp=  0.22500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   52 of  144 k jsp=  0.25000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   53 of  144 k jsp=  0.27500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   54 of  144 k jsp=  0.30000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   55 of  144 k jsp=  0.32500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   56 of  144 k jsp=  0.35000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   57 of  144 k jsp=  0.37500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   58 of  144 k jsp=  0.40000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   59 of  144 k jsp=  0.42500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   60 of  144 k jsp=  0.45000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   61 of  144 k jsp=  0.47500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   62 of  144 k jsp=  0.50000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   63 of  144 k jsp=  0.52500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   64 of  144 k jsp=  0.55000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   65 of  144 k jsp=  0.57500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   66 of  144 k jsp=  0.60000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   67 of  144 k jsp=  0.62500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   68 of  144 k jsp=  0.65000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   69 of  144 k jsp=  0.67500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   70 of  144 k jsp=  0.70000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   71 of  144 k jsp=  0.72500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   72 of  144 k jsp=  0.75000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   73 of  144 k jsp=  0.77500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   74 of  144 k jsp=  0.80000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   75 of  144 k jsp=  0.82500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   76 of  144 k jsp=  0.85000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   77 of  144 k jsp=  0.87500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   78 of  144 k jsp=  0.90000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   79 of  144 k jsp=  0.92500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   80 of  144 k jsp=  0.95000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   81 of  144 k jsp=  0.97500  0.00000  0.00000 1 nev=   31
 bndfp: kpt   82 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   83 of  144 k jsp=  1.00000  0.00000  0.00000 1 nev=   31
 bndfp: kpt   84 of  144 k jsp=  1.00000  0.02500  0.00000 1 nev=   31
 bndfp: kpt   85 of  144 k jsp=  1.00000  0.05000  0.00000 1 nev=   31
 bndfp: kpt   86 of  144 k jsp=  1.00000  0.07500  0.00000 1 nev=   31
 bndfp: kpt   87 of  144 k jsp=  1.00000  0.10000  0.00000 1 nev=   31
 bndfp: kpt   88 of  144 k jsp=  1.00000  0.12500  0.00000 1 nev=   31
 bndfp: kpt   89 of  144 k jsp=  1.00000  0.15000  0.00000 1 nev=   31
 bndfp: kpt   90 of  144 k jsp=  1.00000  0.17500  0.00000 1 nev=   31
 bndfp: kpt   91 of  144 k jsp=  1.00000  0.20000  0.00000 1 nev=   31
 bndfp: kpt   92 of  144 k jsp=  1.00000  0.22500  0.00000 1 nev=   31
 bndfp: kpt   93 of  144 k jsp=  1.00000  0.25000  0.00000 1 nev=   31
 bndfp: kpt   94 of  144 k jsp=  1.00000  0.27500  0.00000 1 nev=   31
 bndfp: kpt   95 of  144 k jsp=  1.00000  0.30000  0.00000 1 nev=   31
 bndfp: kpt   96 of  144 k jsp=  1.00000  0.32500  0.00000 1 nev=   31
 bndfp: kpt   97 of  144 k jsp=  1.00000  0.35000  0.00000 1 nev=   31
 bndfp: kpt   98 of  144 k jsp=  1.00000  0.37500  0.00000 1 nev=   31
 bndfp: kpt   99 of  144 k jsp=  1.00000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  100 of  144 k jsp=  1.00000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  101 of  144 k jsp=  1.00000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  102 of  144 k jsp=  1.00000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  103 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  104 of  144 k jsp=  1.00000  0.50000  0.00000 1 nev=   31
 bndfp: kpt  105 of  144 k jsp=  0.97500  0.48750  0.00000 1 nev=   31
 bndfp: kpt  106 of  144 k jsp=  0.95000  0.47500  0.00000 1 nev=   31
 bndfp: kpt  107 of  144 k jsp=  0.92500  0.46250  0.00000 1 nev=   31
 bndfp: kpt  108 of  144 k jsp=  0.90000  0.45000  0.00000 1 nev=   31
 bndfp: kpt  109 of  144 k jsp=  0.87500  0.43750  0.00000 1 nev=   31
 bndfp: kpt  110 of  144 k jsp=  0.85000  0.42500  0.00000 1 nev=   31
 bndfp: kpt  111 of  144 k jsp=  0.82500  0.41250  0.00000 1 nev=   31
 bndfp: kpt  112 of  144 k jsp=  0.80000  0.40000  0.00000 1 nev=   31
 bndfp: kpt  113 of  144 k jsp=  0.77500  0.38750  0.00000 1 nev=   31
 bndfp: kpt  114 of  144 k jsp=  0.75000  0.37500  0.00000 1 nev=   31
 bndfp: kpt  115 of  144 k jsp=  0.72500  0.36250  0.00000 1 nev=   31
 bndfp: kpt  116 of  144 k jsp=  0.70000  0.35000  0.00000 1 nev=   31
 bndfp: kpt  117 of  144 k jsp=  0.67500  0.33750  0.00000 1 nev=   31
 bndfp: kpt  118 of  144 k jsp=  0.65000  0.32500  0.00000 1 nev=   31
 bndfp: kpt  119 of  144 k jsp=  0.62500  0.31250  0.00000 1 nev=   31
 bndfp: kpt  120 of  144 k jsp=  0.60000  0.30000  0.00000 1 nev=   31
 bndfp: kpt  121 of  144 k jsp=  0.57500  0.28750  0.00000 1 nev=   31
 bndfp: kpt  122 of  144 k jsp=  0.55000  0.27500  0.00000 1 nev=   31
 bndfp: kpt  123 of  144 k jsp=  0.52500  0.26250  0.00000 1 nev=   31
 bndfp: kpt  124 of  144 k jsp=  0.50000  0.25000  0.00000 1 nev=   31
 bndfp: kpt  125 of  144 k jsp=  0.47500  0.23750  0.00000 1 nev=   31
 bndfp: kpt  126 of  144 k jsp=  0.45000  0.22500  0.00000 1 nev=   31
 bndfp: kpt  127 of  144 k jsp=  0.42500  0.21250  0.00000 1 nev=   31
 bndfp: kpt  128 of  144 k jsp=  0.40000  0.20000  0.00000 1 nev=   31
 bndfp: kpt  129 of  144 k jsp=  0.37500  0.18750  0.00000 1 nev=   31
 bndfp: kpt  130 of  144 k jsp=  0.35000  0.17500  0.00000 1 nev=   31
 bndfp: kpt  131 of  144 k jsp=  0.32500  0.16250  0.00000 1 nev=   31
 bndfp: kpt  132 of  144 k jsp=  0.30000  0.15000  0.00000 1 nev=   31
 bndfp: kpt  133 of  144 k jsp=  0.27500  0.13750  0.00000 1 nev=   31
 bndfp: kpt  134 of  144 k jsp=  0.25000  0.12500  0.00000 1 nev=   31
 bndfp: kpt  135 of  144 k jsp=  0.22500  0.11250  0.00000 1 nev=   31
 bndfp: kpt  136 of  144 k jsp=  0.20000  0.10000  0.00000 1 nev=   31
 bndfp: kpt  137 of  144 k jsp=  0.17500  0.08750  0.00000 1 nev=   31
 bndfp: kpt  138 of  144 k jsp=  0.15000  0.07500  0.00000 1 nev=   31
 bndfp: kpt  139 of  144 k jsp=  0.12500  0.06250  0.00000 1 nev=   31
 bndfp: kpt  140 of  144 k jsp=  0.10000  0.05000  0.00000 1 nev=   31
 bndfp: kpt  141 of  144 k jsp=  0.07500  0.03750  0.00000 1 nev=   31
 bndfp: kpt  142 of  144 k jsp=  0.05000  0.02500  0.00000 1 nev=   31
 bndfp: kpt  143 of  144 k jsp=  0.02500  0.01250  0.00000 1 nev=   31
 bndfp: kpt  144 of  144 k jsp=  0.00000  0.00000  0.00000 1 nev=   31
CPU time:    1.640s     Tue Sep 19 17:16:43 2023 on process=0

  ==== procid=0 ====     calls      == cpu time ===   depth 1
  entry   xxxx  xxxx                per call  total  (depth is by TIM= in ctrl.*.)
      0      0      0        1       1.64       1.64   main
      0      0      0        1       0.01       0.01   |--m_lmfinit
      0      0      0        1       0.00       0.00   |--m_lattic_init
      0      0      0        1       0.50       0.50   |--m_mksym_init
      0      0      0        1       0.00       0.00   |--m_mkqp_init
      0      0      0        1       0.01       0.01   |--m_supot_init
      0      0      0        1       0.00       0.00   |--m_suham_init
      0      0      0        1       0.00       0.00   |--m_qplist_init
      0      0      0        1       0.00       0.00   |--m_qplist_qpsdivider
      0      0      0        1       0.00       0.00   |--m_igv2xall_init
      0      0    -10        0       0.00       0.00   `--lmfp
Exit 0 procid= 0 plot band mode done
