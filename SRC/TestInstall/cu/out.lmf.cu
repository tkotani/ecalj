#include "hip/hip_runtime.h"
rdcmd:  lmfa --no-iactiv cu -vnk=8 -vbigbas=f
 -----------------------  START LMFA (80000K)  -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.

 rdctrl: reset global max nl from 5 to 4

 LMFA:     alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMFA 7.0)  verb 31,30
 pot:      XC:BH

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion

conf:SPEC_ATOM= A : --- Table for atomic configuration ---
conf int(P)z = int(P) where P is replaced by PZ if it is semicore
conf:  isp  l  int(P) int(P)z    Qval    Qcore   CoreConf
conf:    1  0       4  4        1.000    6.000 => 1,2,3,
conf:    1  1       4  4        0.000   12.000 => 2,3,
conf:    1  2       3  3       10.000    0.000 => 
conf:    1  3       4  4        0.000    0.000 => 
conf:-----------------------------------------------------

 Species A:  Z=29  Qc=18  R=2.311271  Q=0
 mesh:   rmt=2.311271  rmax=48.805862  a=0.025  nr=393  nr(rmax)=515
  Pl=  4.5     4.5     3.5     4.5    
  Ql=  1.0     0.0     10.0    0.0    

  iter     qint         drho          vh0          rho0          vsum     beta
    1   29.000000   4.725E+03      145.0000    0.1442E+03      -58.2772   0.30
   51   29.000000   4.211E-05      274.8263    0.2631E+05     -130.7915   0.30


 sumev=-4.333254  etot=-3304.416258  eref=-3304.434500  diff= 0.018242

 Free-atom wavefunctions:
 valence:      eval       node at      max at       c.t.p.   rho(r>rmt)
   4s      -0.36411         0.890       2.256       3.582     0.643062
   4p      -0.06295         0.975       3.484       7.414     0.901829
   3d      -0.39691         0.000       0.600       3.429     0.056076
   4f       0.01948         0.000      35.393      48.806*    1.000000

 core:        ecore       node at      max at       c.t.p.   rho(r>rmt)
   1s    -649.07634         0.000       0.034       0.069     0.000000
   2s     -77.91382         0.070       0.197       0.308     0.000000
   2p     -67.32532         0.000       0.158       0.335     0.000000
   3s      -8.39248         0.288       0.614       0.895     0.000141
   3p      -5.29682         0.260       0.619       1.078     0.000727

 Optimise free-atom basis for species A, rmt=2.311271
 l  it    Rsm      Eh     stiffR   stiffE      Eval      Exact     Pnu    Ql
 0   9   2.311  -0.297     106.7    449.5   -0.36395  -0.36411    4.76   1.00
 ... rsm exceeded rmt*2/3 .. repeat with rsm=rmt
 0   5   1.541  -0.133     106.7   3896.8   -0.35292  -0.36411    4.76   1.00
 1  11   2.311  -0.100     147.4     45.6   -0.04993  -0.06295    4.56   0.00
 ... rsm exceeded rmt*2/3 .. repeat with rsm=rmt
 1   1   1.541  -0.100     147.4  -1923.8    0.05244  -0.06295    4.56   0.00
 2  27   0.962  -0.116     158.7    107.6   -0.39670  -0.39691    3.89  10.00
 eigenvalue sum:  exact  -4.33325    opt basis  -4.31995    error 0.01330

 tailsm: fit tails to 6 smoothed hankels, rmt= 2.31127, rsm= 1.15564
 E:    -1.00000    -2.00000    -4.00000    -6.00000    -9.00000    -15.0000
 C:    -0.07160    10.75053    -187.492    1222.023    -4717.79    21166.81
        r          rho         fit         diff
    2.311271    0.017797    0.017766    0.000031
    2.967767    0.005662    0.005658    0.000005
    3.810725    0.001517    0.001518   -0.000001
    4.893104    0.000305    0.000305    0.000000
    6.282906    0.000041    0.000041   -0.000001
    8.067448    0.000003    0.000003    0.000000
    q(fit):     1.203836    rms diff:   0.000016
    fit: r>rmt  1.203836   r<rmt  3.442816   qtot  4.646652
    rho: r>rmt  1.203836   r<rmt  9.796164   qtot 11.000000

 coretail: q=0.00392, rho(rmt)=0.00465.  Fit with Hankel e=-24.082  coeff=764.|
      r            rhoc          fit
    2.311271    0.02095279    0.02095279
    2.429779    0.01229068    0.01231367
    2.753317    0.00285262    0.00285190
    3.119934    0.00054243    0.00053465
    3.535366    0.00008235    0.00007888
    4.006112    0.00000969    0.00000887
    4.539536    0.00000085    0.00000073
    5.143985    0.00000005    0.00000004

  Not write mtopara.* when there is PZ.

 Sum of reference energies: -3304.4345
 Exit 0 LMFA 
 wkinfo:  used    94 K  workspace of 80000 K   in   0 K calls
rdcmd:  lmf  --no-iactiv cu -vnk=8 -vbigbas=f
 -----------------------  START LMF (80000K)  -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.

 rdctrl: reset global max nl from 5 to 4

 LMF:      alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMF 7.0)  verb 31,30
 pot:      XC:BH
 bz:       metal(2), tetra, invit 

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:  60 irreducible QP from 512 ( 8 8 8 )  shift= T T T
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    3    4         3  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)

 gvlist: cutoff radius   9.000 gives    941   recips of max   1331
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors
 

 Makidx:  hamiltonian dimensions Low, Int, High, Negl: 18 0 7 7
 kappa   Low   Int   High  L+I  L+I+H  Neglected
   1       9     0     7     9    16       0
   2       9     0     0     9     9       7
  all     18     0     7    18    25       7
 suham :  16 augmentation channels, 16 local potential channels  Maximum lmxa=3

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 

 iors  : read restart file (binary, mesh density) 
 iors  : empty file ... nothing read

 rdovfa: read and overlap free-atom densities (mesh density) ...
 rdovfa: expected A,       read A        with rmt=  2.3113  mesh   393  0.025

 ovlpfa: overlap smooth part of FA densities
 site   1  spec  1  pos  0.0000  0.0000  0.0000  Qsmooth 4.646654
 total smooth Q = 4.646654

 Free atom and overlapped crystal site charges:
   ib    true(FA)    smooth(FA)  true(OV)    smooth(OV)    local
    1    9.796164    3.442818   10.275300    3.921954    6.353346

 Smooth charge on mesh:            4.646654
 Sum of local charges:             6.353346
 Total valence charge:            11.000000
 Sum of core charges:             18.000000
 Sum of nuclear charges:         -29.000000
 Homogeneous background:           0.000000
 Deviation from neutrality:        0.000000

 --- BNDFP:  begin iteration 1 of 12 ---

 avg es pot at rmt= 0.554993  avg sphere pot= 0.633521  vconst=-0.554993

 smooth rhoves     11.022237   charge     4.646654
 smooth rhoeps =   -3.843801   rhomu =   -5.010456  avg vxc =   -0.851784 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000012

 Energy terms:             smooth           local           total
   rhoval*vef            -12.156988      -177.336818      -189.493806
   rhoval*ves            -46.689418      -115.324370      -162.013788
   psnuc*ves              68.733892    -12976.662455    -12907.928563
   utot                   11.022237     -6545.993412     -6534.971175
   rho*exc                -3.843801      -126.414296      -130.258097
   rho*vxc                -5.010456      -167.409313      -172.419769
   valence chg             4.646654         6.353346        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Incompatible or missing qp weights file ...

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.6627 -0.0591 -0.0536 -0.0536  0.0167  0.0167  1.7979  1.9471  1.9471
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.3283 -0.1257 -0.0727 -0.0255  0.0508  0.0920  0.6928  1.2801  1.4813
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4087 -0.1094 -0.0570 -0.0368  0.0286  0.0633  0.7989  1.4016  1.7830
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.3857 -0.1324 -0.0620  0.0030  0.0284  0.0660  0.9994  1.1878  1.4473
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.2227 -0.1325 -0.1100  0.0005  0.0467  0.2005  0.6490  0.8702  1.1936
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4328 -0.0995 -0.0481 -0.0481  0.0391  0.0391  0.7797  1.7311  1.7311

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144577;  11.000000 electrons
         Sum occ. bands:   -0.853241, incl. Bloechl correction: -0.006586

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.6627 -0.0591 -0.0536 -0.0536  0.0167  0.0167  1.7979  1.9471  1.9471
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.3283 -0.1257 -0.0727 -0.0255  0.0508  0.0920  0.6928  1.2801  1.4813
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4087 -0.1094 -0.0570 -0.0368  0.0286  0.0633  0.7989  1.4016  1.7830
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.3857 -0.1324 -0.0620  0.0030  0.0284  0.0660  0.9994  1.1878  1.4473
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.2227 -0.1325 -0.1100  0.0005  0.0467  0.2005  0.6490  0.8702  1.1936
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4328 -0.0995 -0.0481 -0.0481  0.0391  0.0391  0.7797  1.7311  1.7311

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.144577;  11.000000 electrons
         Sum occ. bands:   -0.853241, incl. Bloechl correction: -0.006586

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1    9.927753    3.113493    6.814260

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.509191   -0.293352    4.650000    4.686057    4.500000    4.686057
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.531005   -0.280671    4.340000    4.364506    4.250000    4.364506
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.679694   -0.050852    3.870000    3.857413    3.147584    3.857413
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.023532   -0.620420    4.110000    4.109925    4.102416    4.110000

 Harris energy:
 sumev=       -0.853241  val*vef=    -189.493806   sumtv=     188.640564
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -130.258097     utot=   -6534.971175    ehar=   -3304.832068

 srhov:     -6.360828   -168.222982   -174.583810 sumev=   -0.853241   sumtv=  173.730568

 Kohn-Sham energy:
 sumtv=      173.730568  sumtc=      3171.756639   ekin=     3345.487208
 rhoep=     -128.641260   utot=     -6521.364242   ehks=    -3304.518293
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 0.  RMS DQ=3.93e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.646654      4.185740      4.185740      0.038395      4.185740
 site    1    6.353346      6.814260      6.814260      0.019256      6.814260
 AMIX: nmix=0 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=3.93e-2
 unscreened rms difference:  smooth  0.045849   local  0.019256   tot
   screened rms difference:  smooth  0.038395   local  0.019256   tot  0.039302

 iors  : write restart file (binary, mesh density) 

   it  1  of 12    ehf=      -0.397568   ehk=      -0.083793
h nk=8 bigbas=0 ehf=-.397568 ehk=-.0837934

 --- BNDFP:  begin iteration 2 of 12 ---

 avg es pot at rmt= 0.633049  avg sphere pot= 0.653661  vconst=-0.633049

 smooth rhoves     12.286140   charge     4.185740
 smooth rhoeps =   -3.109206   rhomu =   -4.047533  avg vxc =   -0.858397 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000645

 Energy terms:             smooth           local           total
   rhoval*vef             -7.713657      -175.301725      -183.015382
   rhoval*ves            -48.957947      -107.937688      -156.895635
   psnuc*ves              73.530227    -12964.307484    -12890.777257
   utot                   12.286140     -6536.122586     -6523.836446
   rho*exc                -3.109206      -125.879113      -128.988318
   rho*vxc                -4.047533      -166.689186      -170.736719
   valence chg             4.185740         6.814260        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=0.144577

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7748 -0.6825 -0.6802 -0.6802 -0.6430 -0.6430  1.6026  1.7558  1.7558
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.7179 -0.6847 -0.6654 -0.6541 -0.6178 -0.3403  0.4593  1.0915  1.2747
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.7184 -0.6784 -0.6695 -0.6492 -0.6265 -0.4571  0.5708  1.2078  1.5794
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.7146 -0.6988 -0.6593 -0.6431 -0.6231 -0.4289  0.7940  1.0031  1.2312
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.6997 -0.6980 -0.6768 -0.6419 -0.6231 -0.1362  0.4145  0.6649  0.9900
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.7269 -0.6758 -0.6758 -0.6287 -0.6287 -0.4938  0.5479  1.5355  1.5355

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.191164;  11.000000 electrons
         Sum occ. bands:   -7.093537, incl. Bloechl correction: -0.013294

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.391065    1.889160    8.501905

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.417713   -0.500906    4.686057    4.669917    4.500000    4.669917
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.253604   -0.580929    4.364506    4.321916    4.250000    4.321916
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.859413   -0.665867    3.857413    3.897507    3.147584    3.897507
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.010349   -0.855436    4.110000    4.106422    4.102416    4.110000

 Harris energy:
 sumev=       -7.093537  val*vef=    -183.015382   sumtv=     175.921845
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -128.988318     utot=   -6523.836446    ehar=   -3305.146280

 srhov:     -4.200311   -222.071601   -226.271912 sumev=   -7.093537   sumtv=  219.178375

 Kohn-Sham energy:
 sumtv=      219.178375  sumtc=      3171.756639   ekin=     3390.935015
 rhoep=     -132.663784   utot=     -6562.033736   ehks=    -3303.762505
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 1.  RMS DQ=9.93e-2  last it=3.93e-2
 mixrho: (warning) scr. and lin-mixed densities had 43 and 43 negative points
 charges:       old           new         screened      rms diff       lin mix
 smooth       4.185740      2.498095      2.498095      0.032750      2.498095
 site    1    6.814260      8.501905      8.501905      0.064870      8.501905
 AMIX: nmix=1 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=9.93e-2
   tj: 0.82102
 unscreened rms difference:  smooth  0.024244   local  0.064870   tot
   screened rms difference:  smooth  0.032750   local  0.064870   tot  0.099256

 iors  : write restart file (binary, mesh density) 

   it  2  of 12    ehf=      -0.711780   ehk=       0.671995
 From last iter    ehf=      -0.397568   ehk=      -0.083793
 diffe(q)= -0.314212 (0.099256)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.7117796 ehk=.6719954

 --- BNDFP:  begin iteration 3 of 12 ---

 avg es pot at rmt= 0.604916  avg sphere pot= 0.655635  vconst=-0.604916

 smooth rhoves     10.956051   charge     3.883688
 smooth rhoeps =   -2.817571   rhomu =   -3.666846  avg vxc =   -0.843501 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000597

 Energy terms:             smooth           local           total
   rhoval*vef             -6.576790      -181.158615      -187.735405
   rhoval*ves            -47.654984      -113.403490      -161.058475
   psnuc*ves              69.567087    -12968.686260    -12899.119174
   utot                   10.956051     -6541.044875     -6530.088824
   rho*exc                -2.817571      -126.698711      -129.516282
   rho*vxc                -3.666846      -167.770342      -171.437189
   valence chg             3.883688         7.116312        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.191164

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7459 -0.4492 -0.4459 -0.4459 -0.3978 -0.3978  1.6618  1.8140  1.8140
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.5240 -0.4584 -0.4444 -0.4218 -0.3681 -0.2472  0.5246  1.1465  1.3362
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.5486 -0.4463 -0.4409 -0.4311 -0.3824 -0.3326  0.6354  1.2648  1.6406
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.5296 -0.4910 -0.4336 -0.4016 -0.3771 -0.3164  0.8534  1.0565  1.2958
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.4870 -0.4768 -0.4626 -0.4012 -0.3741 -0.0587  0.4793  0.7235  1.0493
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5673 -0.4409 -0.4409 -0.4023 -0.3806 -0.3806  0.6134  1.5939  1.5939

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.121671;  11.000000 electrons
         Sum occ. bands:   -4.729158, incl. Bloechl correction: -0.011779

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.274241    2.258615    8.015627

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.444930   -0.471542    4.669917    4.661876    4.500000    4.661876
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.298579   -0.559380    4.321916    4.315866    4.250000    4.315866
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.922377   -0.432983    3.897507    3.884967    3.147584    3.884967
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.013587   -0.809393    4.110000    4.107081    4.102416    4.110000

 Harris energy:
 sumev=       -4.729158  val*vef=    -187.735405   sumtv=     183.006247
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.516282     utot=   -6530.088824    ehar=   -3304.842220

 srhov:     -4.839716   -203.035058   -207.874774 sumev=   -4.729158   sumtv=  203.145616

 Kohn-Sham energy:
 sumtv=      203.145616  sumtc=      3171.756639   ekin=     3374.902255
 rhoep=     -131.316411   utot=     -6548.087137   ehks=    -3304.501293
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 2.  RMS DQ=4.81e-2  last it=9.93e-2
 mixrho: (warning) scr. and lin-mixed densities had 13 and 13 negative points
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.883688      2.984374      2.984374      0.016786      2.984374
 site    1    7.116312      8.015627      8.015627      0.032802      8.015627
 AMIX: nmix=2 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=4.81e-2
   tj:-1.03939  -0.10632
 add q= -0.000001 to preserve neutrality
 unscreened rms difference:  smooth  0.012621   local  0.032802   tot
   screened rms difference:  smooth  0.016786   local  0.032802   tot  0.048087

 iors  : write restart file (binary, mesh density) 

   it  3  of 12    ehf=      -0.407720   ehk=      -0.066793
 From last iter    ehf=      -0.711780   ehk=       0.671995
 diffe(q)=  0.304060 (0.048087)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.4077199 ehk=-.0667929

 --- BNDFP:  begin iteration 4 of 12 ---

 avg es pot at rmt= 0.545124  avg sphere pot= 0.668904  vconst=-0.545124

 smooth rhoves      8.670801   charge     3.362076
 smooth rhoeps =   -2.347228   rhomu =   -3.053301  avg vxc =   -0.813360 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000549

 Energy terms:             smooth           local           total
   rhoval*vef             -4.919950      -185.951567      -190.871517
   rhoval*ves            -44.677352      -118.835010      -163.512362
   psnuc*ves              62.018954    -12969.563968    -12907.545014
   utot                    8.670801     -6544.199489     -6535.528688
   rho*exc                -2.347228      -127.805291      -130.152519
   rho*vxc                -3.053301      -169.227280      -172.280580
   valence chg             3.362076         7.637924        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.121671

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.6943 -0.1479 -0.1429 -0.1429 -0.0767 -0.0767  1.7567  1.9062  1.9062
 Est Ef = -0.122 < evl(5)=-0.077 ... using qval=11.0, revise to -0.0767
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.3699 -0.2013 -0.1586 -0.1153 -0.0430  0.0028  0.6426  1.2386  1.4371
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4446 -0.1874 -0.1450 -0.1264 -0.0644 -0.0306  0.7502  1.3595  1.7391
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4219 -0.2145 -0.1469 -0.0883 -0.0631 -0.0275  0.9552  1.1468  1.4017
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.2764 -0.2096 -0.1913 -0.0899 -0.0480  0.1255  0.5981  0.8253  1.1496
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4683 -0.1753 -0.1373 -0.1373 -0.0554 -0.0554  0.7303  1.6886  1.6886

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:      0.066107;  11.000000 electrons
         Sum occ. bands:   -1.734343, incl. Bloechl correction: -0.007557

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.005947    2.935569    7.070378

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.497310   -0.352713    4.661876    4.677994    4.500000    4.677994
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.470940   -0.365528    4.315866    4.349210    4.250000    4.349210
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.758350   -0.137620    3.884967    3.863156    3.147584    3.863156
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021110   -0.677000    4.110000    4.109222    4.102416    4.110000

 Harris energy:
 sumev=       -1.734343  val*vef=    -190.871517   sumtv=     189.137174
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -130.152519     utot=   -6535.528688    ehar=   -3304.787394

 srhov:     -5.820552   -174.778370   -180.598922 sumev=   -1.734343   sumtv=  178.864579

 Kohn-Sham energy:
 sumtv=      178.864579  sumtc=      3171.756639   ekin=     3350.621218
 rhoep=     -129.135489   utot=     -6526.156007   ehks=    -3304.670278
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 3.  RMS DQ=2.66e-2  last it=4.81e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.362076      3.929623      3.929623      0.009672      3.929623
 site    1    7.637924      7.070378      7.070378      0.019610      7.070378
 AMIX: nmix=3 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=2.66e-2
   tj: 0.76324  -0.24309  -0.00278
 unscreened rms difference:  smooth  0.007487   local  0.019610   tot
   screened rms difference:  smooth  0.009672   local  0.019610   tot  0.026573

 iors  : write restart file (binary, mesh density) 

   it  4  of 12    ehf=      -0.352894   ehk=      -0.235778
 From last iter    ehf=      -0.407720   ehk=      -0.066793
 diffe(q)=  0.054826 (0.026573)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.3528937 ehk=-.2357778

 --- BNDFP:  begin iteration 5 of 12 ---

 avg es pot at rmt= 0.570014  avg sphere pot= 0.662710  vconst=-0.570014

 smooth rhoves      9.528839   charge     3.555443
 smooth rhoeps =   -2.515360   rhomu =   -3.272543  avg vxc =   -0.825511 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000560

 Energy terms:             smooth           local           total
   rhoval*vef             -5.478219      -184.922914      -190.401134
   rhoval*ves            -45.930034      -117.335734      -163.265768
   psnuc*ves              64.987712    -12970.136584    -12905.148872
   utot                    9.528839     -6543.736159     -6534.207320
   rho*exc                -2.515360      -127.430480      -129.945839
   rho*vxc                -3.272543      -168.734240      -172.006783
   valence chg             3.555443         7.444557        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=0.066107

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7153 -0.2526 -0.2483 -0.2483 -0.1888 -0.1888  1.7210  1.8717  1.8717
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4121 -0.2880 -0.2597 -0.2221 -0.1560 -0.0975  0.5961  1.2031  1.3985
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4748 -0.2770 -0.2491 -0.2327 -0.1755 -0.1419  0.7052  1.3231  1.7016
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4530 -0.3099 -0.2467 -0.1973 -0.1721 -0.1373  0.9157  1.1118  1.3611
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3389 -0.2998 -0.2860 -0.1980 -0.1618  0.0486  0.5511  0.7856  1.1108
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4977 -0.2604 -0.2428 -0.2428 -0.1689 -0.1689  0.6844  1.6526  1.6526

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014427;  11.000000 electrons
         Sum occ. bands:   -2.765423, incl. Bloechl correction: -0.009080

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.107186    2.683244    7.423942

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.473421   -0.417406    4.677994    4.664139    4.500000    4.664139
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.405007   -0.445721    4.349210    4.334398    4.250000    4.334398
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.839017   -0.241044    3.863156    3.869539    3.147584    3.869539
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018216   -0.727732    4.110000    4.108439    4.102416    4.110000

 Harris energy:
 sumev=       -2.765423  val*vef=    -190.401134   sumtv=     187.635711
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.945839     utot=   -6534.207320    ehar=   -3304.760809

 srhov:     -5.487236   -184.462988   -189.950223 sumev=   -2.765423   sumtv=  187.184801

 Kohn-Sham energy:
 sumtv=      187.184801  sumtc=      3171.756639   ekin=     3358.941440
 rhoep=     -129.896286   utot=     -6533.805664   ehks=    -3304.760509
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=1.06e-3  last it=2.66e-2
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.555443      3.576057      3.576057      0.000274      3.576057
 site    1    7.444557      7.423942      7.423942      0.000766      7.423942
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=1.06e-3
   tj:-0.04404  -0.00157
 unscreened rms difference:  smooth  0.000346   local  0.000766   tot
   screened rms difference:  smooth  0.000274   local  0.000766   tot  0.001055

 iors  : write restart file (binary, mesh density) 

   it  5  of 12    ehf=      -0.326309   ehk=      -0.326009
 From last iter    ehf=      -0.352894   ehk=      -0.235778
 diffe(q)=  0.026584 (0.001055)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.3263093 ehk=-.3260093

 --- BNDFP:  begin iteration 6 of 12 ---

 avg es pot at rmt= 0.571782  avg sphere pot= 0.662667  vconst=-0.571782

 smooth rhoves      9.571474   charge     3.561414
 smooth rhoeps =   -2.519240   rhomu =   -3.277586  avg vxc =   -0.826125 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.484081      -184.917589      -190.401670
   rhoval*ves            -45.999373      -117.283030      -163.282403
   psnuc*ves              65.142320    -12970.133252    -12904.990932
   utot                    9.571474     -6543.708141     -6534.136667
   rho*exc                -2.519240      -127.411874      -129.931113
   rho*vxc                -3.277586      -168.709707      -171.987293
   valence chg             3.561414         7.438586        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.014427

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7173 -0.2613 -0.2570 -0.2570 -0.1979 -0.1979  1.7177  1.8684  1.8684
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4163 -0.2953 -0.2680 -0.2309 -0.1652 -0.1053  0.5922  1.1999  1.3950
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4778 -0.2844 -0.2576 -0.2414 -0.1846 -0.1508  0.7015  1.3199  1.6981
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4560 -0.3179 -0.2549 -0.2062 -0.1810 -0.1460  0.9123  1.1087  1.3576
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3447 -0.3074 -0.2939 -0.2069 -0.1711  0.0428  0.5472  0.7822  1.1074
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5006 -0.2674 -0.2515 -0.2515 -0.1782 -0.1782  0.6806  1.6493  1.6493

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020483;  11.000000 electrons
         Sum occ. bands:   -2.851265, incl. Bloechl correction: -0.009203

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.116131    2.663451    7.452680

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.472518   -0.420342    4.664139    4.664126    4.500000    4.664126
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.397597   -0.454541    4.334398    4.332643    4.250000    4.332643
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.848578   -0.249221    3.869539    3.870439    3.147584    3.870439
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.017948   -0.733445    4.110000    4.108346    4.102416    4.110000

 Harris energy:
 sumev=       -2.851265  val*vef=    -190.401670   sumtv=     187.550405
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.931113     utot=   -6534.136667    ehar=   -3304.760736

 srhov:     -5.458915   -185.245576   -190.704491 sumev=   -2.851265   sumtv=  187.853226

 Kohn-Sham energy:
 sumtv=      187.853226  sumtc=      3171.756639   ekin=     3359.609865
 rhoep=     -129.958969   utot=     -6534.411548   ehks=    -3304.760652
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=7.40e-4  last it=1.06e-3
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.561414      3.547320      3.547320      0.000259      3.547320
 site    1    7.438586      7.452680      7.452680      0.000514      7.452680
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=7.4e-4
   tj: 0.32880   0.00545
 unscreened rms difference:  smooth  0.000200   local  0.000514   tot
   screened rms difference:  smooth  0.000259   local  0.000514   tot  0.000740

 iors  : write restart file (binary, mesh density) 

   it  6  of 12    ehf=      -0.326236   ehk=      -0.326152
 From last iter    ehf=      -0.326309   ehk=      -0.326009
 diffe(q)=  0.000073 (0.000740)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.3262364 ehk=-.3261525

 --- BNDFP:  begin iteration 7 of 12 ---

 avg es pot at rmt= 0.571479  avg sphere pot= 0.662692  vconst=-0.571479

 smooth rhoves      9.560599   charge     3.558853
 smooth rhoeps =   -2.516916   rhomu =   -3.274555  avg vxc =   -0.825983 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.475845      -184.951259      -190.427105
   rhoval*ves            -45.984798      -117.318680      -163.303478
   psnuc*ves              65.105996    -12970.154482    -12905.048486
   utot                    9.560599     -6543.736581     -6534.175982
   rho*exc                -2.516916      -127.418278      -129.935195
   rho*vxc                -3.274555      -168.718150      -171.992705
   valence chg             3.558853         7.441147        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.020483

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7170 -0.2595 -0.2552 -0.2552 -0.1960 -0.1960  1.7182  1.8689  1.8689
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4155 -0.2938 -0.2663 -0.2290 -0.1633 -0.1037  0.5930  1.2005  1.3956
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4773 -0.2829 -0.2558 -0.2396 -0.1827 -0.1489  0.7022  1.3205  1.6987
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4555 -0.3162 -0.2532 -0.2043 -0.1791 -0.1442  0.9130  1.1092  1.3582
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3436 -0.3058 -0.2923 -0.2050 -0.1691  0.0440  0.5479  0.7828  1.1080
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5001 -0.2660 -0.2497 -0.2497 -0.1762 -0.1762  0.6814  1.6498  1.6498

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.019243;  11.000000 electrons
         Sum occ. bands:   -2.833300, incl. Bloechl correction: -0.009175

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.114511    2.667762    7.446749

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.472962   -0.419362    4.664126    4.664349    4.500000    4.664349
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.398510   -0.453484    4.332643    4.332816    4.250000    4.332816
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.847705   -0.247414    3.870439    3.870340    3.147584    3.870340
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.017993   -0.732733    4.110000    4.108357    4.102416    4.110000

 Harris energy:
 sumev=       -2.833300  val*vef=    -190.427105   sumtv=     187.593804
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.935195     utot=   -6534.175982    ehar=   -3304.760733

 srhov:     -5.465713   -185.069263   -190.534976 sumev=   -2.833300   sumtv=  187.701676

 Kohn-Sham energy:
 sumtv=      187.701676  sumtc=      3171.756639   ekin=     3359.458315
 rhoep=     -129.945456   utot=     -6534.273580   ehks=    -3304.760721
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=2.75e-4  last it=7.40e-4
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.558853      3.553251      3.553251      0.000103      3.553251
 site    1    7.441147      7.446749      7.446749      0.000198      7.446749
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: nmix=2 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=2.75e-4
   tj:-0.27367   0.11136
 unscreened rms difference:  smooth  0.000081   local  0.000198   tot
   screened rms difference:  smooth  0.000103   local  0.000198   tot  0.000275

 iors  : write restart file (binary, mesh density) 

   it  7  of 12    ehf=      -0.326233   ehk=      -0.326221
 From last iter    ehf=      -0.326236   ehk=      -0.326152
 diffe(q)=  0.000004 (0.000275)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.3262327 ehk=-.3262211

 --- BNDFP:  begin iteration 8 of 12 ---

 avg es pot at rmt= 0.571303  avg sphere pot= 0.662733  vconst=-0.571303

 smooth rhoves      9.554396   charge     3.557414
 smooth rhoeps =   -2.515610   rhomu =   -3.272851  avg vxc =   -0.825903 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.471294      -184.960161      -190.431455
   rhoval*ves            -45.976455      -117.329765      -163.306220
   psnuc*ves              65.085247    -12970.151434    -12905.066187
   utot                    9.554396     -6543.740599     -6534.186203
   rho*exc                -2.515610      -127.421078      -129.936688
   rho*vxc                -3.272851      -168.721833      -171.994684
   valence chg             3.557414         7.442586        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.019243

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7168 -0.2587 -0.2544 -0.2544 -0.1952 -0.1952  1.7185  1.8692  1.8692
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4151 -0.2931 -0.2655 -0.2282 -0.1625 -0.1030  0.5933  1.2007  1.3959
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4770 -0.2822 -0.2551 -0.2388 -0.1819 -0.1481  0.7025  1.3207  1.6990
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4552 -0.3156 -0.2525 -0.2036 -0.1783 -0.1434  0.9132  1.1095  1.3585
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3431 -0.3052 -0.2916 -0.2042 -0.1683  0.0444  0.5483  0.7831  1.1083
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4998 -0.2654 -0.2489 -0.2489 -0.1754 -0.1754  0.6817  1.6501  1.6501

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018730;  11.000000 electrons
         Sum occ. bands:   -2.825938, incl. Bloechl correction: -0.009164

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.113797    2.669497    7.444300

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.473097   -0.419036    4.664349    4.664396    4.500000    4.664396
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.399003   -0.452916    4.332816    4.332919    4.250000    4.332919
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.847155   -0.246692    3.870340    3.870282    3.147584    3.870282
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018013   -0.732361    4.110000    4.108363    4.102416    4.110000

 Harris energy:
 sumev=       -2.825938  val*vef=    -190.431455   sumtv=     187.605517
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.936688     utot=   -6534.186203    ehar=   -3304.760735

 srhov:     -5.468090   -184.999129   -190.467219 sumev=   -2.825938   sumtv=  187.641281

 Kohn-Sham energy:
 sumtv=      187.641281  sumtc=      3171.756639   ekin=     3359.397920
 rhoep=     -129.939954   utot=     -6534.218700   ehks=    -3304.760734
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=8.82e-5  last it=2.75e-4
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.557414      3.555700      3.555700      0.000033      3.555700
 site    1    7.442586      7.444300      7.444300      0.000062      7.444300
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=8.82e-5
   tj:-0.46979
 unscreened rms difference:  smooth  0.000026   local  0.000062   tot
   screened rms difference:  smooth  0.000033   local  0.000062   tot  0.000088

 iors  : write restart file (binary, mesh density) 

   it  8  of 12    ehf=      -0.326235   ehk=      -0.326234
 From last iter    ehf=      -0.326233   ehk=      -0.326221
 diffe(q)= -0.000002 (0.000088)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=0 ehf=-.3262352 ehk=-.3262339

 --- BNDFP:  begin iteration 9 of 12 ---

 avg es pot at rmt= 0.571240  avg sphere pot= 0.662744  vconst=-0.571240

 smooth rhoves      9.552091   charge     3.556850
 smooth rhoeps =   -2.515086   rhomu =   -3.272167  avg vxc =   -0.825874 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=16  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.469413      -184.966553      -190.435966
   rhoval*ves            -45.973434      -117.336533      -163.309966
   psnuc*ves              65.077616    -12970.153887    -12905.076271
   utot                    9.552091     -6543.745210     -6534.193119
   rho*exc                -2.515086      -127.422320      -129.937406
   rho*vxc                -3.272167      -168.723469      -171.995637
   valence chg             3.556850         7.443150        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read qp weights ...  ef=-0.01873

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7168 -0.2584 -0.2541 -0.2541 -0.1948 -0.1948  1.7186  1.8693  1.8693
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4150 -0.2929 -0.2652 -0.2279 -0.1621 -0.1027  0.5934  1.2008  1.3960
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4769 -0.2820 -0.2548 -0.2385 -0.1815 -0.1478  0.7026  1.3208  1.6991
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4551 -0.3153 -0.2522 -0.2032 -0.1780 -0.1431  0.9134  1.1096  1.3586
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3429 -0.3049 -0.2913 -0.2039 -0.1680  0.0447  0.5484  0.7832  1.1084
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4997 -0.2651 -0.2486 -0.2486 -0.1751 -0.1751  0.6818  1.6502  1.6502

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018513;  11.000000 electrons
         Sum occ. bands:   -2.822819, incl. Bloechl correction: -0.009160

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.113484    2.670273    7.443211

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.473158   -0.418903    4.664396    4.664418    4.500000    4.664418
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.399217   -0.452680    4.332919    4.332963    4.250000    4.332963
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.846904   -0.246385    3.870282    3.870257    3.147584    3.870257
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.018022   -0.732210    4.110000    4.108365    4.102416    4.110000

 Harris energy:
 sumev=       -2.822819  val*vef=    -190.435966   sumtv=     187.613148
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.937406     utot=   -6534.193119    ehar=   -3304.760737

 srhov:     -5.469249   -184.966960   -190.436209 sumev=   -2.822819   sumtv=  187.613391

 Kohn-Sham energy:
 sumtv=      187.613391  sumtc=      3171.756639   ekin=     3359.370030
 rhoep=     -129.937449   utot=     -6534.193318   ehks=    -3304.760737
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=2.34e-6  last it=8.82e-5
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.556850      3.556789      3.556789      0.000006      3.556789
 site    1    7.443150      7.443211      7.443211      0.000002      7.443211
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2016  beta=1  tm=5  rmsdel=2.34e-6
   tj:-0.01976
 unscreened rms difference:  smooth  0.000006   local  0.000002   tot
   screened rms difference:  smooth  0.000006   local  0.000002   tot  0.000002

 iors  : write restart file (binary, mesh density) 

   it  9  of 12    ehf=      -0.326237   ehk=      -0.326237
 From last iter    ehf=      -0.326235   ehk=      -0.326234
 diffe(q)= -0.000002 (0.000002)    tol= 0.000010 (0.000010)   more=F
c nk=8 bigbas=0 ehf=-.3262374 ehk=-.3262372
 Exit 0 LMF 
 wkinfo:  used   406 K  workspace of 80000 K   in  74 K calls
rdcmd:  rm mixm.cu
rdcmd:  lmf  --no-iactiv cu -vnk=8 -vbigbas=t -vpwmode=0 -voveps=0d-7
 -----------------------  START LMF (80000K)  -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.

 LMF:      alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMF 7.0)  verb 31,30
 pot:      XC:BH
 bz:       metal(3), tetra, invit 

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:  60 irreducible QP from 512 ( 8 8 8 )  shift= T T T
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    4    4         4  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)

 gvlist: cutoff radius   9.000 gives    941   recips of max   1331
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors
 

 Makidx:  hamiltonian dimensions Low, Int, High, Negl: 31 0 28 16
 kappa   Low   Int   High  L+I  L+I+H  Neglected
   1       9     0    16     9    25       0
   2      13     0    12    13    25       0
   3       9     0     0     9     9      16
  all     31     0    28    31    59      16
 suham :  25 augmentation channels, 25 local potential channels  Maximum lmxa=4

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     869 
  A        3    1.30  -0.01   6.806    2.09E-06     411 

 iors  : read restart file (binary, mesh density) 
         use from  restart file: ef window, positions, pnu 
         ignore in restart file: *
         site   1, species A       : augmentation lmax changed from 3 to 4
         site   1, species A       : inflate local density from nlm= 16 to 25

 --- BNDFP:  begin iteration 1 of 12 ---

 avg es pot at rmt= 0.571237  avg sphere pot= 0.662746  vconst=-0.571237

 smooth rhoves      9.551947   charge     3.556810
 smooth rhoeps =   -2.515047   rhomu =   -3.272117  avg vxc =   -0.825872 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000558

 Energy terms:             smooth           local           total
   rhoval*vef             -5.469267      -184.966399      -190.435666
   rhoval*ves            -45.973260      -117.336408      -163.309668
   psnuc*ves              65.077154    -12970.153157    -12905.076004
   utot                    9.551947     -6543.744783     -6534.192836
   rho*exc                -2.515047      -127.422356      -129.937403
   rho*vxc                -3.272117      -168.723515      -171.995632
   valence chg             3.556810         7.443190        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7176 -0.2587 -0.2544 -0.2544 -0.1946 -0.1946  1.5859  1.7373  1.8052
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4154 -0.2932 -0.2656 -0.2280 -0.1622 -0.1029  0.5909  1.1772  1.3092
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4775 -0.2825 -0.2549 -0.2386 -0.1817 -0.1479  0.6977  1.2862  1.5948
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4557 -0.3158 -0.2525 -0.2034 -0.1781 -0.1433  0.9069  1.0961  1.2465
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3433 -0.3051 -0.2917 -0.2038 -0.1683  0.0443  0.5466  0.7740  1.0777
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5003 -0.2656 -0.2487 -0.2487 -0.1751 -0.1751  0.6763  1.6276  1.6276

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018792;  11.000000 electrons
         Sum occ. bands:   -2.825304, incl. Bloechl correction: -0.009153

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7176 -0.2587 -0.2544 -0.2544 -0.1946 -0.1946  1.5859  1.7373  1.8052
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4154 -0.2932 -0.2656 -0.2280 -0.1622 -0.1029  0.5909  1.1772  1.3092
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4775 -0.2825 -0.2549 -0.2386 -0.1817 -0.1479  0.6977  1.2862  1.5948
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4557 -0.3158 -0.2525 -0.2034 -0.1781 -0.1433  0.9069  1.0961  1.2465
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3433 -0.3051 -0.2917 -0.2038 -0.1683  0.0443  0.5466  0.7740  1.0777
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5003 -0.2656 -0.2487 -0.2487 -0.1751 -0.1751  0.6763  1.6276  1.6276

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018792;  11.000000 electrons
         Sum occ. bands:   -2.825304, incl. Bloechl correction: -0.009153

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.128160    2.833312    7.294848

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.480584   -0.533465    4.664418    4.612337    4.500000    4.612337
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.407336   -0.469584    4.332963    4.328212    4.250000    4.328212
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.971687   -0.255973    3.870257    3.858616    3.147584    3.858616
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021180   -0.806341    4.110000    4.106719    4.102416    4.110000
 4     1    0.004395   -0.781220    5.100000    5.079490    5.077979    5.100000

 Harris energy:
 sumev=       -2.825304  val*vef=    -190.435666   sumtv=     187.610362
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.937403     utot=   -6534.192836    ehar=   -3304.763238

 srhov:     -5.929501   -184.355314   -190.284815 sumev=   -2.825304   sumtv=  187.459511

 Kohn-Sham energy:
 sumtv=      187.459511  sumtc=      3171.756639   ekin=     3359.216151
 rhoep=     -129.927456   utot=     -6534.051861   ehks=    -3304.763166
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 0.  RMS DQ=4.37e-3
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.556810      3.705152      3.705152      0.003469      3.705152
 site    1    7.443190      7.294848      7.294848      0.004825      7.294848
 AMIX: nmix=0 mmix=8  nelts=2178  beta=1  tm=5  rmsdel=4.37e-3
 unscreened rms difference:  smooth  0.003486   local  0.004825   tot
   screened rms difference:  smooth  0.003469   local  0.004825   tot  0.004371

 iors  : write restart file (binary, mesh density) 

   it  1  of 12    ehf=      -0.328738   ehk=      -0.328666
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf=-.3287376 ehk=-.3286662

 --- BNDFP:  begin iteration 2 of 12 ---

 avg es pot at rmt= 0.581158  avg sphere pot= 0.653091  vconst=-0.581158

 smooth rhoves     10.108795   charge     3.705152
 smooth rhoeps =   -2.657496   rhomu =   -3.458045  avg vxc =   -0.833024 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000127

 Energy terms:             smooth           local           total
   rhoval*vef             -6.009415      -184.304839      -190.314253
   rhoval*ves            -46.615710      -116.582740      -163.198450
   psnuc*ves              66.833301    -12971.740403    -12904.907102
   utot                   10.108795     -6544.161572     -6534.052776
   rho*exc                -2.657496      -127.270006      -129.927501
   rho*vxc                -3.458045      -168.524429      -171.982474
   valence chg             3.705152         7.294848        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7177 -0.2622 -0.2578 -0.2578 -0.1970 -0.1970  1.5859  1.7372  1.8045
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4165 -0.2957 -0.2683 -0.2308 -0.1654 -0.1056  0.5898  1.1765  1.3087
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4780 -0.2853 -0.2576 -0.2415 -0.1847 -0.1509  0.6967  1.2855  1.5944
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4562 -0.3188 -0.2550 -0.2066 -0.1810 -0.1463  0.9061  1.0955  1.2460
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3450 -0.3076 -0.2945 -0.2062 -0.1720  0.0424  0.5456  0.7733  1.0771
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5008 -0.2682 -0.2516 -0.2516 -0.1782 -0.1782  0.6752  1.6269  1.6269

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020714;  11.000000 electrons
         Sum occ. bands:   -2.853789, incl. Bloechl correction: -0.009204

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7177 -0.2622 -0.2578 -0.2578 -0.1970 -0.1970  1.5859  1.7372  1.8045
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4165 -0.2957 -0.2683 -0.2308 -0.1654 -0.1056  0.5898  1.1765  1.3087
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4780 -0.2853 -0.2576 -0.2415 -0.1847 -0.1509  0.6967  1.2855  1.5944
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4562 -0.3188 -0.2550 -0.2066 -0.1810 -0.1463  0.9061  1.0955  1.2460
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3450 -0.3076 -0.2945 -0.2062 -0.1720  0.0424  0.5456  0.7733  1.0771
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5008 -0.2682 -0.2516 -0.2516 -0.1782 -0.1782  0.6752  1.6269  1.6269

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.020714;  11.000000 electrons
         Sum occ. bands:   -2.853789, incl. Bloechl correction: -0.009204

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.131732    2.825132    7.306600

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.479387   -0.535597    4.612337    4.611672    4.500000    4.611672
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.404856   -0.472668    4.328212    4.327646    4.250000    4.327646
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.969521   -0.258624    3.858616    3.858674    3.147584    3.858674
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021024   -0.810002    4.110000    4.106662    4.102416    4.110000
 4     1    0.004375   -0.784068    5.100000    5.079474    5.077979    5.100000

 Harris energy:
 sumev=       -2.853789  val*vef=    -190.314253   sumtv=     187.460464
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.927501     utot=   -6534.052776    ehar=   -3304.763174

 srhov:     -5.994048   -184.684580   -190.678628 sumev=   -2.853789   sumtv=  187.824839

 Kohn-Sham energy:
 sumtv=      187.824839  sumtc=      3171.756639   ekin=     3359.581478
 rhoep=     -129.959113   utot=     -6534.385433   ehks=    -3304.763067
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 1.  RMS DQ=7.67e-4  last it=4.37e-3
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.705152      3.693400      3.693400      0.000208      3.693400
 site    1    7.294848      7.306600      7.306600      0.000500      7.306600
 AMIX: nmix=1 mmix=8  nelts=2178  beta=1  tm=5  rmsdel=7.67e-4
   tj: 0.08056
 unscreened rms difference:  smooth  0.000174   local  0.000500   tot
   screened rms difference:  smooth  0.000208   local  0.000500   tot  0.000767

 iors  : write restart file (binary, mesh density) 

   it  2  of 12    ehf=      -0.328674   ehk=      -0.328567
 From last iter    ehf=      -0.328738   ehk=      -0.328666
 diffe(q)=  0.000064 (0.000767)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf=-.328674 ehk=-.3285671

 --- BNDFP:  begin iteration 3 of 12 ---

 avg es pot at rmt= 0.579980  avg sphere pot= 0.653052  vconst=-0.579980

 smooth rhoves     10.061393   charge     3.694347
 smooth rhoeps =   -2.647850   rhomu =   -3.445462  avg vxc =   -0.832366 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000134

 Energy terms:             smooth           local           total
   rhoval*vef             -5.973948      -184.535193      -190.509140
   rhoval*ves            -46.555926      -116.812277      -163.368203
   psnuc*ves              66.678712    -12971.953009    -12905.274297
   utot                   10.061393     -6544.382643     -6534.321250
   rho*exc                -2.647850      -127.303260      -129.951109
   rho*vxc                -3.445462      -168.568326      -172.013789
   valence chg             3.694347         7.305653        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7163 -0.2523 -0.2479 -0.2479 -0.1865 -0.1865  1.5877  1.7387  1.8068
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4124 -0.2876 -0.2588 -0.2208 -0.1547 -0.0967  0.5937  1.1793  1.3115
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4754 -0.2769 -0.2478 -0.2315 -0.1743 -0.1406  0.7004  1.2883  1.5968
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4535 -0.3098 -0.2457 -0.1963 -0.1708 -0.1362  0.9093  1.0982  1.2490
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3388 -0.2991 -0.2856 -0.1960 -0.1612  0.0488  0.5494  0.7764  1.0801
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4983 -0.2604 -0.2416 -0.2416 -0.1675 -0.1675  0.6790  1.6297  1.6297

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014006;  11.000000 electrons
         Sum occ. bands:   -2.756738, incl. Bloechl correction: -0.009049

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7163 -0.2523 -0.2479 -0.2479 -0.1865 -0.1865  1.5877  1.7387  1.8068
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4124 -0.2876 -0.2588 -0.2208 -0.1547 -0.0967  0.5937  1.1793  1.3115
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4754 -0.2769 -0.2478 -0.2315 -0.1743 -0.1406  0.7004  1.2883  1.5968
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4535 -0.3098 -0.2457 -0.1963 -0.1708 -0.1362  0.9093  1.0982  1.2490
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3388 -0.2991 -0.2856 -0.1960 -0.1612  0.0488  0.5494  0.7764  1.0801
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.4983 -0.2604 -0.2416 -0.2416 -0.1675 -0.1675  0.6790  1.6297  1.6297

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.014006;  11.000000 electrons
         Sum occ. bands:   -2.756738, incl. Bloechl correction: -0.009049

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.121437    2.845312    7.276124

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.481245   -0.529907    4.611672    4.613320    4.500000    4.613320
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.411022   -0.464864    4.327646    4.329215    4.250000    4.329215
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.950064   -0.248938    3.858674    3.858073    3.147584    3.858073
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021443   -0.799134    4.110000    4.106882    4.102416    4.110000
 4     1    0.004440   -0.772503    5.100000    5.079581    5.077979    5.100000

 Harris energy:
 sumev=       -2.756738  val*vef=    -190.509140   sumtv=     187.752402
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.951109     utot=   -6534.321250    ehar=   -3304.763318

 srhov:     -6.021913   -183.666474   -189.688387 sumev=   -2.756738   sumtv=  186.931649

 Kohn-Sham energy:
 sumtv=      186.931649  sumtc=      3171.756639   ekin=     3358.688288
 rhoep=     -129.879167   utot=     -6533.571887   ehks=    -3304.762766
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 2.  RMS DQ=1.80e-3  last it=7.67e-4
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.694347      3.723876      3.723876      0.000529      3.723876
 site    1    7.305653      7.276124      7.276124      0.001211      7.276124
 AMIX: nmix=2 mmix=8  nelts=2178  beta=1  tm=5  rmsdel=1.8e-3
   tj: 0.70189  -0.00506
 unscreened rms difference:  smooth  0.000390   local  0.001211   tot
   screened rms difference:  smooth  0.000529   local  0.001211   tot  0.001801

 iors  : write restart file (binary, mesh density) 

   it  3  of 12    ehf=      -0.328818   ehk=      -0.328266
 From last iter    ehf=      -0.328674   ehk=      -0.328567
 diffe(q)= -0.000143 (0.001801)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf=-.3288175 ehk=-.3282662

 --- BNDFP:  begin iteration 4 of 12 ---

 avg es pot at rmt= 0.580860  avg sphere pot= 0.653051  vconst=-0.580860

 smooth rhoves     10.096266   charge     3.702580
 smooth rhoeps =   -2.655417   rhomu =   -3.455336  avg vxc =   -0.832820 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.002045      -184.376183      -190.378228
   rhoval*ves            -46.599143      -116.655488      -163.254631
   psnuc*ves              66.791674    -12971.816331    -12905.024657
   utot                   10.096266     -6544.235910     -6534.139644
   rho*exc                -2.655417      -127.279393      -129.934810
   rho*vxc                -3.455336      -168.536831      -171.992167
   valence chg             3.702580         7.297420        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7173 -0.2591 -0.2547 -0.2547 -0.1937 -0.1937  1.5865  1.7376  1.8052
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4152 -0.2932 -0.2653 -0.2277 -0.1621 -0.1029  0.5910  1.1774  1.3096
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4772 -0.2826 -0.2546 -0.2384 -0.1815 -0.1477  0.6978  1.2864  1.5951
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4553 -0.3160 -0.2521 -0.2034 -0.1778 -0.1431  0.9070  1.0963  1.2470
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3431 -0.3049 -0.2917 -0.2031 -0.1686  0.0444  0.5468  0.7742  1.0780
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5000 -0.2658 -0.2485 -0.2485 -0.1749 -0.1749  0.6764  1.6277  1.6277

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018651;  11.000000 electrons
         Sum occ. bands:   -2.823576, incl. Bloechl correction: -0.009156

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7173 -0.2591 -0.2547 -0.2547 -0.1937 -0.1937  1.5865  1.7376  1.8052
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4152 -0.2932 -0.2653 -0.2277 -0.1621 -0.1029  0.5910  1.1774  1.3096
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4772 -0.2826 -0.2546 -0.2384 -0.1815 -0.1477  0.6978  1.2864  1.5951
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4553 -0.3160 -0.2521 -0.2034 -0.1778 -0.1431  0.9070  1.0963  1.2470
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3431 -0.3049 -0.2917 -0.2031 -0.1686  0.0444  0.5468  0.7742  1.0780
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5000 -0.2658 -0.2485 -0.2485 -0.1749 -0.1749  0.6764  1.6277  1.6277

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018651;  11.000000 electrons
         Sum occ. bands:   -2.823576, incl. Bloechl correction: -0.009156

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.128443    2.831457    7.296986

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.479978   -0.533700    4.613320    4.612251    4.500000    4.612251
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.406875   -0.470206    4.329215    4.328151    4.250000    4.328151
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.963081   -0.255629    3.858073    3.858466    3.147584    3.858466
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021154   -0.806683    4.110000    4.106730    4.102416    4.110000
 4     1    0.004395   -0.780578    5.100000    5.079507    5.077979    5.100000

 Harris energy:
 sumev=       -2.823576  val*vef=    -190.378228   sumtv=     187.554652
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.934810     utot=   -6534.139644    ehar=   -3304.763162

 srhov:     -6.003000   -184.365805   -190.368804 sumev=   -2.823576   sumtv=  187.545228

 Kohn-Sham energy:
 sumtv=      187.545228  sumtc=      3171.756639   ekin=     3359.301868
 rhoep=     -129.934006   utot=     -6534.131024   ehks=    -3304.763162
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 3.  RMS DQ=2.21e-5  last it=1.80e-3
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.702580      3.703014      3.703014      0.000017      3.703014
 site    1    7.297420      7.296986      7.296986      0.000016      7.296986
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2178  beta=1  tm=5  rmsdel=2.21e-5
   tj:-0.01228
 unscreened rms difference:  smooth  0.000021   local  0.000016   tot
   screened rms difference:  smooth  0.000017   local  0.000016   tot  0.000022

 iors  : write restart file (binary, mesh density) 

   it  4  of 12    ehf=      -0.328662   ehk=      -0.328662
 From last iter    ehf=      -0.328818   ehk=      -0.328266
 diffe(q)=  0.000155 (0.000022)    tol= 0.000010 (0.000010)   more=T
i nk=8 bigbas=1 pwmode=0 oveps=0 ehf=-.3286622 ehk=-.328662

 --- BNDFP:  begin iteration 5 of 12 ---

 avg es pot at rmt= 0.580888  avg sphere pot= 0.653045  vconst=-0.580888

 smooth rhoves     10.096995   charge     3.702758
 smooth rhoeps =   -2.655612   rhomu =   -3.455591  avg vxc =   -0.832823 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.002699      -184.374100      -190.376800
   rhoval*ves            -46.599988      -116.653393      -163.253381
   psnuc*ves              66.793979    -12971.815985    -12905.022006
   utot                   10.096995     -6544.234689     -6534.137694
   rho*exc                -2.655612      -127.279025      -129.934637
   rho*vxc                -3.455591      -168.536346      -171.991937
   valence chg             3.702758         7.297242        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 subzi: tetrahedron integration of bands; tetrahedron integration of density

 Start first of two band passes ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7173 -0.2591 -0.2548 -0.2548 -0.1938 -0.1938  1.5865  1.7376  1.8051
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4153 -0.2932 -0.2654 -0.2278 -0.1621 -0.1029  0.5910  1.1774  1.3095
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4772 -0.2827 -0.2546 -0.2385 -0.1815 -0.1478  0.6978  1.2863  1.5951
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4554 -0.3160 -0.2522 -0.2035 -0.1779 -0.1432  0.9070  1.0963  1.2469
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3431 -0.3050 -0.2918 -0.2031 -0.1687  0.0443  0.5467  0.7742  1.0780
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5000 -0.2659 -0.2486 -0.2486 -0.1749 -0.1749  0.6763  1.6277  1.6277

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018702;  11.000000 electrons
         Sum occ. bands:   -2.824270, incl. Bloechl correction: -0.009157

 Saved qp weights ...
 Start second band pass ...
 bndfp:  kpt 1 of 60, k=  0.06250  0.06250  0.06250
 -0.7173 -0.2591 -0.2548 -0.2548 -0.1938 -0.1938  1.5865  1.7376  1.8051
 bndfp:  kpt 11 of 60, k=  -0.18750  0.31250  0.56250
 -0.4153 -0.2932 -0.2654 -0.2278 -0.1621 -0.1029  0.5910  1.1774  1.3095
 bndfp:  kpt 21 of 60, k=  0.43750  -0.31250  0.18750
 -0.4772 -0.2827 -0.2546 -0.2385 -0.1815 -0.1478  0.6978  1.2863  1.5951
 bndfp:  kpt 31 of 60, k=  -0.06250  0.18750  -0.56250
 -0.4554 -0.3160 -0.2522 -0.2035 -0.1779 -0.1432  0.9070  1.0963  1.2469
 bndfp:  kpt 41 of 60, k=  -0.06250  0.43750  0.68750
 -0.3431 -0.3050 -0.2918 -0.2031 -0.1687  0.0443  0.5467  0.7742  1.0780
 bndfp:  kpt 51 of 60, k=  0.31250  0.31250  0.31250
 -0.5000 -0.2659 -0.2486 -0.2486 -0.1749 -0.1749  0.6763  1.6277  1.6277

 BZWTS : --- Tetrahedron Integration ---
 BZINTS: Fermi energy:     -0.018702;  11.000000 electrons
         Sum occ. bands:   -2.824270, incl. Bloechl correction: -0.009157

 Saved qp weights ...

 ... Generating total DOS

 mkrout:  Qtrue      sm,loc       local
   1   10.128504    2.831326    7.297178

 Symmetrize density..

 Make new boundary conditions for phi,phidot..

 site    1   species   1:A       
 l  idmod     ql         ebar        pold        ptry        pfree        pnew
 0     0    0.479955   -0.533756    4.612251    4.612234    4.500000    4.612234
 0     0    sc          0.000000    5.500000    5.500000    5.500000    5.500000
 1     0    0.406826   -0.470263    4.328151    4.328141    4.250000    4.328141
 1     0    sc          0.000000    5.500000    5.250000    5.250000    5.500000
 2     0    8.963382   -0.255690    3.858466    3.858479    3.147584    3.858479
 2     0    sc          0.000000    4.500000    4.147584    4.147584    4.500000
 3     1    0.021151   -0.806765    4.110000    4.106729    4.102416    4.110000
 4     1    0.004395   -0.780659    5.100000    5.079506    5.077979    5.100000

 Harris energy:
 sumev=       -2.824270  val*vef=    -190.376800   sumtv=     187.552529
 sumec=        0.000000  cor*vef=       0.000000   ttcor=    3171.756639
 rhoeps=    -129.934637     utot=   -6534.137694    ehar=   -3304.763162

 srhov:     -6.002879   -184.372691   -190.375570 sumev=   -2.824270   sumtv=  187.551300

 Kohn-Sham energy:
 sumtv=      187.551300  sumtc=      3171.756639   ekin=     3359.307939
 rhoep=     -129.934540   utot=     -6534.136561   ehks=    -3304.763162
  
 mixing: mode=A  nmix=3  beta=1  elind=1.291
 mixrho:  sought 3 iter from file mixm; read 4.  RMS DQ=3.06e-6  last it=2.21e-5
 charges:       old           new         screened      rms diff       lin mix
 smooth       3.702758      3.702822      3.702822      0.000008      3.702822
 site    1    7.297242      7.297178      7.297178      0.000002      7.297178
 AMIX: condition of normal eqns >100000. Reducing nmix to 2
 AMIX: condition of normal eqns >100000. Reducing nmix to 1
 AMIX: nmix=1 mmix=8  nelts=2178  beta=1  tm=5  rmsdel=3.06e-6
   tj:-0.15999
 unscreened rms difference:  smooth  0.000009   local  0.000002   tot
   screened rms difference:  smooth  0.000008   local  0.000002   tot  0.000003

 iors  : write restart file (binary, mesh density) 

   it  5  of 12    ehf=      -0.328662   ehk=      -0.328662
 From last iter    ehf=      -0.328662   ehk=      -0.328662
 diffe(q)=  0.000000 (0.000003)    tol= 0.000010 (0.000010)   more=F
c nk=8 bigbas=1 pwmode=0 oveps=0 ehf=-.3286618 ehk=-.3286617
 Exit 0 LMF 
 wkinfo:  used   626 K  workspace of 80000 K   in 242 K calls
rdcmd:  lmf  --no-iactiv cu -vnk=8 -vbigbas=t -vpwmode=0 -voveps=0d-7 --band:fn=syml
 -----------------------  START LMF (80000K)  -----------------------
 ptenv() is called with EXT=cu
 ptenv() not supported, but continue.

 LMF:      alat = 6.798  nbas = 1  nspec = 1  vn 7.00(LMF 7.0)  verb 31,30
 pot:      XC:BH
 bz:       metal(3), tetra, invit 

                Plat                                  Qlat
   0.000000   0.500000   0.500000       -1.000000   1.000000   1.000000
   0.500000   0.000000   0.500000        1.000000  -1.000000   1.000000
   0.500000   0.500000   0.000000        1.000000   1.000000  -1.000000
  Cell vol= 78.538660

 LATTC:  as= 2.000   tol=1.00E-12   alat= 6.79800   awald= 0.467
         r1=  2.252   nkd= 201      q1=  6.871   nkg= 331

 SGROUP: 1 symmetry operations from 0 generators
 SYMLAT: Bravais system is cubic with 48 symmetry operations.
 SYMCRY: crystal invariant under 48 symmetry operations for tol=1e-5
 GROUPG: the following are sufficient to generate the space group:
         i*r3(1,1,-1) r4x
         i*r3(1,1,-1) r4x
 MKSYM:  found 48 space group operations ... includes inversion
 
 BZMESH:  60 irreducible QP from 512 ( 8 8 8 )  shift= T T T
 TETIRR: sorting 3072 tetrahedra ...
 264 inequivalent ones found

 species data:  augmentation                           density
 spec       rmt   rsma lmxa kmxa      lmxl     rg   rsmv  kmxv foca   rfoca
 A        2.311  0.925    4    4         4  0.578  1.156    15    1   0.925

 MSHSIZ: mesh has 11 x 11 x 11 divisions; length 0.437, 0.437, 0.437
         generated from gmax = 9.0 a.u. : 941 vectors of 1331 (70%)

 gvlist: cutoff radius   9.000 gives    941   recips of max   1331
 SGVSYM: 41 symmetry stars found for 941 reciprocal lattice vectors
 

 Makidx:  hamiltonian dimensions Low, Int, High, Negl: 31 0 28 16
 kappa   Low   Int   High  L+I  L+I+H  Neglected
   1       9     0    16     9    25       0
   2      13     0    12    13    25       0
   3       9     0     0     9     9      16
  all     31     0    28    31    59      16
 suham :  25 augmentation channels, 25 local potential channels  Maximum lmxa=4

 sugcut:  make orbital-dependent reciprocal vector cutoffs for tol= 1.00E-06
 spec      l    rsm    eh     gmax    last term    cutoff
  A        0*   2.50  -0.01   2.974    2.30E-05      27 
  A        1    2.50  -0.01   3.093    1.29E-06      51 
  A        2    1.00  -0.01   8.508    1.16E-06     869 
  A        0    1.30  -1.00   5.718    2.28E-06     259 
  A        2    1.00  -1.00   8.508    1.16E-06     869 
  A        3    1.30  -0.01   6.806    2.09E-06     411 

 iors  : read restart file (binary, mesh density) 
         use from  restart file: ef window, positions, pnu 
         ignore in restart file: *

 --- BNDFP:  begin iteration 1 of 12 ---

 avg es pot at rmt= 0.580894  avg sphere pot= 0.653044  vconst=-0.580894

 smooth rhoves     10.097116   charge     3.702792
 smooth rhoeps =   -2.655652   rhomu =   -3.455643  avg vxc =   -0.832822 

 locpot:

 site  1  z= 29.0  rmt= 2.31127  nr=393   a=0.025  nlml=25  rg=0.578  Vfloat=T
 sm core charge = 0.263001 (sphere) + 0.004646 (spillout) = 0.267647
 potential shift to crystal energy zero:    0.000135

 Energy terms:             smooth           local           total
   rhoval*vef             -6.002841      -184.373715      -190.376556
   rhoval*ves            -46.600123      -116.653034      -163.253156
   psnuc*ves              66.794355    -12971.816011    -12905.021656
   utot                   10.097116     -6544.234522     -6534.137406
   rho*exc                -2.655652      -127.278965      -129.934618
   rho*vxc                -3.455643      -168.536268      -171.991911
   valence chg             3.702792         7.297208        11.000000
   core charge            18.000000         0.000000        18.000000

 Charges:  valence    11.00000   cores    18.00000   nucleii   -29.00000
    hom background     0.00000   deviation from neutrality:      0.00000

 Read efermi from weights file : ef = -0.018702
 suqlst:  generate bands, mode 1

 suqlst:  nq= 41   q1= 0.5000 0.5000 0.5000   q2= 0.0000 0.0000 0.0000
 bndfp:  kpt 1 of 41, k=  -0.50000  -0.50000  -0.50000
 -0.4126 -0.2588 -0.2588 -0.1500 -0.1500 -0.1050  0.2487  1.5877  1.5973
 bndfp:  kpt 11 of 41, k=  0.37500  0.37500  0.37500
 -0.4449 -0.2536 -0.2536 -0.2248 -0.1624 -0.1624  0.4812  1.6094  1.6094
 bndfp:  kpt 21 of 41, k=  0.25000  0.25000  0.25000
 -0.5703 -0.2760 -0.2447 -0.2447 -0.1870 -0.1870  0.8928  1.6566  1.6566
 bndfp:  kpt 31 of 41, k=  0.12500  0.12500  0.12500
 -0.6860 -0.2652 -0.2497 -0.2497 -0.1953 -0.1953  1.3655  1.7127  1.7482
 bndfp:  kpt 41 of 41, k=  0.00000  0.00000  0.00000
 -0.7279 -0.2568 -0.2568 -0.2568 -0.1930 -0.1930  1.6567  1.8387  1.8387

 Read efermi from weights file : ef = -0.018702

 suqlst:  nq= 41   q1= 0.0000 0.0000 0.0000   q2= 1.0000 0.0000 0.0000
 bndfp:  kpt 1 of 41, k=  0.00000  0.00000  0.00000
 -0.7279 -0.2568 -0.2568 -0.2568 -0.1930 -0.1930  1.6567  1.8387  1.8387
 bndfp:  kpt 11 of 41, k=  0.25000  0.00000  0.00000
 -0.6722 -0.2732 -0.2433 -0.2433 -0.2075 -0.1870  1.5242  1.5425  1.5425
 bndfp:  kpt 21 of 41, k=  0.50000  0.00000  0.00000
 -0.5227 -0.3114 -0.2283 -0.2064 -0.2064 -0.1723  1.2328  1.2328  1.3539
 bndfp:  kpt 31 of 41, k=  0.75000  0.00000  0.00000
 -0.4048 -0.3480 -0.1614 -0.1614 -0.1572 -0.1230  0.8184  1.0147  1.0147
 bndfp:  kpt 41 of 41, k=  -1.00000  0.00000  0.00000
 -0.3961 -0.3627 -0.1509 -0.1394 -0.1394  0.0772  0.4995  0.9336  0.9336

 Read efermi from weights file : ef = -0.018702

 suqlst:  nq= 21   q1= 1.0000 0.0000 0.0000   q2= 1.0000 0.5000 0.0000
 bndfp:  kpt 1 of 21, k=  -1.00000  0.00000  0.00000
 -0.3961 -0.3627 -0.1509 -0.1394 -0.1394  0.0772  0.4995  0.9336  0.9336
 bndfp:  kpt 11 of 21, k=  -1.00000  0.25000  0.00000
 -0.3718 -0.3463 -0.2102 -0.1698 -0.1394  0.1977  0.5208  0.7107  0.7756
 bndfp:  kpt 21 of 21, k=  -1.00000  0.50000  0.00000
 -0.3401 -0.2953 -0.2953 -0.1964 -0.1394  0.4291  0.4291  0.5910  0.6156

 Read efermi from weights file : ef = -0.018702

 suqlst:  nq= 41   q1= 1.0000 0.5000 0.0000   q2= 0.0000 0.0000 0.0000
 bndfp:  kpt 1 of 41, k=  -1.00000  0.50000  0.00000
 -0.3401 -0.2953 -0.2953 -0.1964 -0.1394  0.4291  0.4291  0.5910  0.6156
 bndfp:  kpt 11 of 41, k=  0.75000  0.37500  0.00000
 -0.3461 -0.3172 -0.2822 -0.1886 -0.1617  0.0625  0.6342  0.7293  0.9129
 bndfp:  kpt 21 of 41, k=  0.50000  0.25000  0.00000
 -0.4790 -0.3053 -0.2569 -0.2057 -0.2018 -0.1412  0.9570  1.0059  1.4114
 bndfp:  kpt 31 of 41, k=  0.25000  0.12500  0.00000
 -0.6587 -0.2738 -0.2436 -0.2428 -0.2061 -0.1870  1.3635  1.4235  1.7178
 bndfp:  kpt 41 of 41, k=  0.00000  0.00000  0.00000
 -0.7279 -0.2568 -0.2568 -0.2568 -0.1930 -0.1930  1.6567  1.8387  1.8387

 Read efermi from weights file : ef = -0.018702
 Exit 0 bndfp 
 wkinfo:  used   555 K  workspace of 80000 K   in  53 K calls
